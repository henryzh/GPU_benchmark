#include "hip/hip_runtime.h"
#include "config.h"
#ifdef LOCKFREE
__device__ inline void __barrier_lockfree(int goalVal, volatile int *Arrayin, volatile int *Arrayout) {
	int tx = threadIdx.x;// * blockDim.y + threadIdx.y;
	int numBlocks = gridDim.x;// * gridDim.y;
	int bid = blockIdx.x;// * gridDim.y + blockIdx.y;
	if(tx == 0) {
		Arrayin[bid] = goalVal;
	}
	if(bid == 1) {
		if(tx < numBlocks) {
			while (Arrayin[tx] != goalVal) {}
		}
		__syncthreads();
		if(tx < numBlocks) {
			Arrayout[tx] = goalVal;
		}
	}
	if(tx == 0) {
		while(Arrayout[bid] != goalVal) {}
	}
	__syncthreads();
}
#endif

#ifdef ATOMIC
__device__ volatile int mutex = 0;
__device__ inline void __barrier_atomic(int goal) {
	__syncthreads();
//	int bx = blockIdx.x;
	uint tx = threadIdx.x;// * blockDim.y + threadIdx.y;
	if (tx == 0) {
		atomicAdd((int *)&mutex, 1);
		while(mutex != goal) {}
	}
	__syncthreads();
}
#endif
