#include "hip/hip_runtime.h"
// (C) Copyright 2013, University of Illinois. All Rights Reserved
// Author: Lijiuan Luo (lluo3@uiuc.edu), Geng Daniel Liu (gengliu2@illinois.edu)

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <parboil.h>
#include <deque>
#include <iostream>
#include "config.h"
FILE *fp;

typedef int2 Node;
typedef int2 Edge;

#include "kernel.cu"
const int h_top = 1;
const int zero = 0;

int main(int argc, char** argv) {
  printf("[BENCH] BFS from Parboil\n");
#ifdef LOCKFREE
  printf("[BENCH] Lock-free Barrier\n");
#endif
#ifdef ATOMIC
  printf("[BENCH] Atomic Barrier\n");
#endif
#ifdef HW_SYNC
  printf("[BENCH] Hardware Barrier\n");
#endif
  int num_of_nodes = 0;
  int num_of_edges = 0;
  struct pb_Parameters *params;
  struct pb_TimerSet timers;

  pb_InitializeTimerSet(&timers);
  params = pb_ReadParameters(&argc, argv);
  if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL)) {
    fprintf(stderr, "Expecting one input filename\n");
    exit(-1);
  }

  pb_SwitchToTimer(&timers, pb_TimerID_IO);

//  printf("Read in Graph from a file\n");
  fp = fopen(params->inpFiles[0],"r");
  if(!fp) {
    printf("Error Reading graph file\n");
    return 0;
  }
  int source;
  int res;
  res = fscanf(fp,"%d",&num_of_nodes);

//   printf("Allocate host memory\n");
  Node* h_graph_nodes = (Node*) malloc(sizeof(Node)*num_of_nodes);
  int *color = (int*) malloc(sizeof(int)*num_of_nodes);
  int start, edgeno;   

//  printf("Initalize the memory\n");
  for( unsigned int i = 0; i < num_of_nodes; i++) {
    res = fscanf(fp,"%d %d",&start,&edgeno);
    h_graph_nodes[i].x = start;
    h_graph_nodes[i].y = edgeno;
    color[i]=WHITE;
  }
//  printf("Read the source node from the file\n");
  res = fscanf(fp,"%d",&source);
  res = fscanf(fp,"%d",&num_of_edges);
  int id,cost;
  Edge* h_graph_edges = (Edge*) malloc(sizeof(Edge)*num_of_edges);
  for(int i=0; i < num_of_edges ; i++) {
    res = fscanf(fp,"%d",&id);
    res = fscanf(fp,"%d",&cost);
    h_graph_edges[i].x = id;
    h_graph_edges[i].y = cost;
  }
  if(res!=1)
    printf("Reading input failed\n");
  if(fp)
    fclose(fp);

//  printf("Allocate mem for the result on host side\n");
  int* h_cost = (int*) malloc( sizeof(int)*num_of_nodes);
  for(int i = 0; i < num_of_nodes; i++){
    h_cost[i] = INF;
  }
  h_cost[source] = 0;

  pb_SwitchToTimer(&timers, pb_TimerID_COPY);

//  printf("Copy the Node List to device memory\n");
  Node* d_graph_nodes;
  hipMalloc((void**) &d_graph_nodes, sizeof(Node)*num_of_nodes);
  hipMemcpy(d_graph_nodes, h_graph_nodes, sizeof(Node)*num_of_nodes, hipMemcpyHostToDevice);

//  printf("Copy the Edge List to device Memory\n");
  Edge* d_graph_edges;
  hipMalloc((void**) &d_graph_edges, sizeof(Edge)*num_of_edges);
  hipMemcpy(d_graph_edges, h_graph_edges, sizeof(Edge)*num_of_edges, hipMemcpyHostToDevice);

  int* d_color;
  hipMalloc((void**) &d_color, sizeof(int)*num_of_nodes);
  int* d_cost;
  hipMalloc((void**) &d_cost, sizeof(int)*num_of_nodes);
  int * d_q1;
  int * d_q2;
  hipMalloc((void**) &d_q1, sizeof(int)*num_of_nodes);
  hipMalloc((void**) &d_q2, sizeof(int)*num_of_nodes);
  int * tail;
  hipMalloc((void**) &tail, sizeof(int));
  int *front_cost_d;
  hipMalloc((void**) &front_cost_d, sizeof(int));
  hipMemcpy(d_color, color, sizeof(int)*num_of_nodes, hipMemcpyHostToDevice);
  hipMemcpy(d_cost, h_cost, sizeof(int)*num_of_nodes, hipMemcpyHostToDevice);

  //bind the texture memory with global memory
  hipBindTexture(0,g_graph_node_ref,d_graph_nodes, sizeof(Node)*num_of_nodes);
  hipBindTexture(0,g_graph_edge_ref,d_graph_edges,sizeof(Edge)*num_of_edges);

  printf("[BENCH] Starting GPU kernel\n");
  hipDeviceSynchronize();
  pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);

  int num_of_blocks; 
  int num_of_threads_per_block;

  hipMemcpy(tail,&h_top,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(&d_cost[source],&zero,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy( &d_q1[0], &source, sizeof(int), hipMemcpyHostToDevice);

  int num_t;//number of threads
  int k=0;//BFS level index

  //whether or not to adjust "k", see comment on "BFS_kernel_multi_blk_inGPU" for more details 
  int * switch_kd;
  hipMalloc((void**) &switch_kd, sizeof(int));
  int * num_td;//number of threads
  hipMalloc((void**) &num_td, sizeof(int));

  //whether to stay within a kernel, used in "BFS_kernel_multi_blk_inGPU"
  bool *stay;
  hipMalloc( (void**) &stay, sizeof(bool));
  int switch_k;

  //max number of frontier nodes assigned to a block
  int * max_nodes_per_block_d;
  hipMalloc( (void**) &max_nodes_per_block_d, sizeof(int));
  int *global_kt_d;
  hipMalloc( (void**) &global_kt_d, sizeof(int));
  hipMemcpy(global_kt_d,&zero, sizeof(int),hipMemcpyHostToDevice);

  int h_overflow = 0;
  int *d_overflow;
  hipMalloc((void**) &d_overflow, sizeof(int));
  hipMemcpy(d_overflow, &h_overflow, sizeof(int), hipMemcpyHostToDevice);
  int count1=0, count2=0, count3=0;

  int *in, *out;
#ifdef LOCKFREE
  int flag_size = NUM_SM*sizeof(int);
  hipMalloc((void **)&in, flag_size);
  hipMalloc((void **)&out, flag_size);
  hipMemset(in, 0, flag_size);
  hipMemset(out, 0, flag_size);
#endif
  hipDeviceSynchronize();

  do {
    hipMemcpy( &num_t, tail, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(tail,&zero,sizeof(int),hipMemcpyHostToDevice);

    if(num_t == 0) {//frontier is empty
      hipFree(stay);
      hipFree(switch_kd);
      hipFree(num_td);
      break;
    }

    num_of_blocks = 1;
    num_of_threads_per_block = num_t;
    if(num_of_threads_per_block <NUM_BIN)
      num_of_threads_per_block = NUM_BIN;
    if(num_t>MAX_THREADS_PER_BLOCK) {
      num_of_blocks = (int)ceil(num_t/(double)MAX_THREADS_PER_BLOCK); 
      num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
    }
    if(num_of_blocks == 1)//will call "BFS_in_GPU_kernel" 
      num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
    if(num_of_blocks >1 && num_of_blocks <= NUM_SM)// will call "BFS_kernel_multi_blk_inGPU"
      num_of_blocks = NUM_SM;

    //assume "num_of_blocks" can not be very large
    dim3  grid( num_of_blocks, 1, 1);
    dim3  threads( num_of_threads_per_block, 1, 1);

    if(k%2 == 0) {
      if(num_of_blocks == 1) {
        count1 ++;
        BFS_in_GPU_kernel<<< grid, threads >>>(d_q1,d_q2, d_graph_nodes, 
            d_graph_edges, d_color, d_cost,num_t , tail,GRAY0,k,d_overflow);
      }
      else if(num_of_blocks <= NUM_SM) {
        count2 ++;
        (hipMemcpy(num_td,&num_t,sizeof(int),
                    hipMemcpyHostToDevice));
        BFS_kernel_multi_blk_inGPU
          <<< grid, threads >>>(d_q1,d_q2, d_graph_nodes, 
              d_graph_edges, d_color, d_cost, num_td, tail,GRAY0,k,
              switch_kd, max_nodes_per_block_d, global_kt_d,d_overflow, in, out);
        (hipMemcpy(&switch_k,switch_kd, sizeof(int),
                    hipMemcpyDeviceToHost));
        if(!switch_k) {
          k--;
        }
      }
      else {
        count3 ++;
        BFS_kernel<<< grid, threads >>>(d_q1,d_q2, d_graph_nodes, 
            d_graph_edges, d_color, d_cost, num_t, tail,GRAY0,k,d_overflow);
      }
    }
    else {
      if(num_of_blocks == 1) {
        count1 ++;
        BFS_in_GPU_kernel<<< grid, threads >>>(d_q2,d_q1, d_graph_nodes, 
            d_graph_edges, d_color, d_cost, num_t, tail,GRAY1,k,d_overflow);
      }
      else if(num_of_blocks <= NUM_SM) {
        count2 ++;
        (hipMemcpy(num_td,&num_t,sizeof(int),
                    hipMemcpyHostToDevice));
        BFS_kernel_multi_blk_inGPU
          <<< grid, threads >>>(d_q2,d_q1, d_graph_nodes, 
              d_graph_edges, d_color, d_cost, num_td, tail,GRAY1,k,
              switch_kd, max_nodes_per_block_d, global_kt_d,d_overflow, in, out);
        (hipMemcpy(&switch_k,switch_kd, sizeof(int),
                    hipMemcpyDeviceToHost));
        if(!switch_k) {
          k--;
        }
      }
      else {
        count3 ++;
        BFS_kernel<<< grid, threads >>>(d_q2,d_q1, d_graph_nodes, 
            d_graph_edges, d_color, d_cost, num_t, tail, GRAY1,k,d_overflow);
      }
    }
    k++;
    hipMemcpy(&h_overflow, d_overflow, sizeof(int), hipMemcpyDeviceToHost);
    if(h_overflow) {
      printf("Error: local queue was overflown. Need to increase W_LOCAL_QUEUE\n");
      return 0;
    }
  } while(1);

  hipDeviceSynchronize();
#ifdef LOCKFREE
  hipFree(in);
  hipFree(out);
#endif
  pb_SwitchToTimer(&timers, pb_TimerID_COPY);
  printf("[BENCH] GPU kernel done\n");
  printf("[BENCH] Kernel called %d times (%d, %d, %d)\n", k, count1, count2, count3);

  // Copy result from device to host
  hipMemcpy(h_cost, d_cost, sizeof(int)*num_of_nodes, hipMemcpyDeviceToHost);
  hipMemcpy(color, d_color, sizeof(int)*num_of_nodes, hipMemcpyDeviceToHost);
  hipUnbindTexture(g_graph_node_ref);
  hipUnbindTexture(g_graph_edge_ref);

  hipFree(d_graph_nodes);
  hipFree(d_graph_edges);
  hipFree(d_color);
  hipFree(d_cost);
  hipFree(tail);
  hipFree(front_cost_d);

  // Store the result into a file
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
  FILE *fp = fopen(params->outFile,"w");
  fprintf(fp, "%d\n", num_of_nodes);
  for(int i=0;i<num_of_nodes;i++)
    fprintf(fp,"%d %d\n",i,h_cost[i]);
  fclose(fp);

  // Cleanup memory
  free( h_graph_nodes);
  free( h_graph_edges);
  free( color);
  free( h_cost);
  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(params);
  return 0;
}
