#include "hip/hip_runtime.h"

/***********************************************************************************************
 * * Implementing Graph Cuts on CUDA using algorithm given in CVGPU '08                       **
 * * paper "CUDA Cuts: Fast Graph Cuts on GPUs"                                               **
 * * Copyright (c) 2008 International Institute of Information Technology.                    **
 * * All rights reserved.                                                                     **
 * * Created By Vibhav Vineet.                                                                **
 * ********************************************************************************************/

#ifndef _PUSHRELABEL_KERNEL_CU_
#define _PUSHRELABEL_KERNEL_CU_

#include "CudaCuts.h"

#define LOCAL_INDEX(localX,localY) __umul24(localY+1 , 34 ) + localX + 1;
#define LOCAL_LEFT(index) (index-1)
#define LOCAL_RIGHT(index) (index+1)
#define LOCAL_TOP(index) (index-34)
#define LOCAL_BOTTOM(index) (index+34)

#define GLOBAL_LEFT(gid) (gid-1)
#define GLOBAL_RIGHT(gid) (gid+1)
#define GLOBAL_TOP(gid) (gid-gSizeX)
#define GLOBAL_BOTTOM(gid) (gid+gSizeX)

__device__ void
load_shared_mem(int* height_fn, int *g_graph_height, const int gSizeX, const int gSizeY) {
    const int gX  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
    const int gY  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
    const int gid = __umul24( gY , gSizeX ) + gX ;

    const int lX = threadIdx.x;
    const int lY = threadIdx.y;
    const int lid = LOCAL_INDEX(lX, lY);

    height_fn[lid] = g_graph_height[gid] ;
    (lX == 31 && gX < gSizeX - 1 ) ? height_fn[LOCAL_RIGHT(lid)] =  (g_graph_height[GLOBAL_RIGHT(gid)]) : 0;
    (lX == 0 && gX > 0 ) ? height_fn[LOCAL_LEFT(lid)] = (g_graph_height[GLOBAL_LEFT(gid)]) : 0;
    (lY == 7 && gY < gSizeY - 1 ) ? height_fn[LOCAL_BOTTOM(lid)] = (g_graph_height[GLOBAL_BOTTOM(gid)]) : 0;
    (lY == 0 && gY > 0 ) ? height_fn[LOCAL_TOP(lid)] = (g_graph_height[GLOBAL_TOP(gid)]) : 0;
}


__device__ void
set_state(int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
        int *g_sink_weight, int *g_push_reser,
        int *g_relabel_mask, int *g_graph_height,
        int gid, int gSizeX, int gSizeY)
{
    int flow_push = g_push_reser[gid] ;

    if(flow_push <= 0 || (g_left_weight[gid] == 0 && g_right_weight[gid] == 0 && g_down_weight[gid] == 0 && g_up_weight[gid] == 0 && g_sink_weight[gid] == 0))
        g_relabel_mask[gid] = 2 ;
    else
    {
        ( flow_push > 0 &&
          (
            ( (g_graph_height[gid] == g_graph_height[GLOBAL_LEFT(gid)] + 1 ) && g_left_weight[gid] > 0  ) ||
            ( (g_graph_height[gid] == g_graph_height[GLOBAL_RIGHT(gid)]+1 ) && g_right_weight[gid] > 0) ||
            ( (g_graph_height[gid] == g_graph_height[GLOBAL_BOTTOM(gid)]+1 ) && g_down_weight[gid] > 0) ||
            ( (g_graph_height[gid] == g_graph_height[GLOBAL_TOP(gid)]+1 ) && g_up_weight[gid] > 0 ) ||
            ( g_graph_height[gid] == 1 && g_sink_weight[gid] > 0 )
          )
        ) ? g_relabel_mask[gid] = 1 : g_relabel_mask[gid] = 0 ;
    }
}

/************************************************
 * Relabel operation                           **
 * *********************************************/

__device__ void
relabel( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
        int *g_sink_weight, int *g_push_reser,
        int *g_relabel_mask, int *g_graph_height,
        int gRealSizeTotal, int gRealSizeX, int gRealSizeY, int gSizeTotal, int gSizeX, int gSizeY,
        int gid, int gX, int gY)
{
    set_state(g_left_weight, g_right_weight, g_down_weight, g_up_weight,
                g_sink_weight, g_push_reser,
                g_relabel_mask, g_graph_height,
                gid, gSizeX, gSizeY);


    __syncthreads();

    if(gid < gSizeTotal && gX < gRealSizeX - 1  && gX > 0 && gY < gRealSizeY - 1  && gY > 0  )
    {
        if(g_sink_weight[gid] > 0)
        {
            g_graph_height[gid] = 1 ;
        }
        else
        {
            int min_height = gRealSizeTotal ;
            (g_left_weight[gid] > 0 && min_height > g_graph_height[GLOBAL_LEFT(gid)] ) ? min_height = g_graph_height[GLOBAL_LEFT(gid)] : 0 ;
            (g_right_weight[gid] > 0 && min_height > g_graph_height[GLOBAL_RIGHT(gid)]) ? min_height = g_graph_height[GLOBAL_RIGHT(gid)] : 0 ;
            (g_down_weight[gid] > 0 && min_height > g_graph_height[GLOBAL_BOTTOM(gid)] ) ? min_height = g_graph_height[GLOBAL_BOTTOM(gid)] : 0 ;
            (g_up_weight[gid] > 0 && min_height > g_graph_height[GLOBAL_TOP(gid)] ) ? min_height = g_graph_height[GLOBAL_TOP(gid)] : 0 ;
            g_graph_height[gid] = min_height + 1 ;
        }
    }
}

__global__ void
kernel_relabel( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
        int *g_sink_weight, int *g_push_reser,
        int *g_relabel_mask, int *g_graph_height,
        int gRealSizeTotal, int gRealSizeX, int gRealSizeY, int gSizeTotal, int gSizeX, int gSizeY )
{
    const int gX  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
    const int gY  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
    const int gid = __umul24( gY , gSizeX ) + gX ;

    relabel( g_left_weight, g_right_weight, g_down_weight,g_up_weight,
            g_sink_weight, g_push_reser,
            g_relabel_mask, g_graph_height,
            gRealSizeTotal, gRealSizeX, gRealSizeY, gSizeTotal, gSizeX, gSizeY,
            gid, gX, gY);
}


/************************************************
 * Push operation                              **
 * *********************************************/

__device__ void
push_sink(int *g_sink_weight, int *g_push_reser,
        int *g_graph_height,
        int gid)
{
    int temp_weight = g_sink_weight[gid];
    int flow_push = g_push_reser[gid];

    if(temp_weight > 0 && flow_push > 0 && g_graph_height[gid] == 1 )
    {
        int min_flow_pushed = flow_push ;
        (temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;

        g_sink_weight[gid] -= min_flow_pushed;
        g_push_reser[gid] -= min_flow_pushed;
    }
}

__device__ void
push_neighbour(int *g_sink_weight, int *g_push_reser, int *g_graph_height,
        int* my_weight, int* neighbour_pull_weight,
        int gid, int neighbour_id)
{
    int flow_push = g_push_reser[gid];
    int temp_weight = *my_weight;

    if(temp_weight > 0 && flow_push > 0 && g_graph_height[gid] == g_graph_height[neighbour_id] + 1 && *neighbour_pull_weight==0 )
    {
        int min_flow_pushed = flow_push;
        (temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0;

        *my_weight -= min_flow_pushed;
        g_push_reser[gid] -= min_flow_pushed;
        *neighbour_pull_weight = min_flow_pushed;

    }
}

__device__ void pull(int* my_pull_weight, int* my_weight, int *g_push_reser, int gid) {
    if(*my_pull_weight > 0) {
        int pull_weight = *my_pull_weight;
        *my_weight += pull_weight;
        g_push_reser[gid] += pull_weight;
        __threadfence();
        *my_pull_weight = 0;
    }
}

__device__ void
push_all(int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
        int *g_pull_left, int *g_pull_right, int *g_pull_down, int *g_pull_up,
        int *g_sink_weight, int *g_push_reser,
        int *g_relabel_mask, int *g_graph_height,
        int gRealSizeTotal, int gRealSizeX, int gRealSizeY, int gSizeTotal, int gSizeX, int gSizeY,
        int gid, int gX, int gY) {


    if( gid < gSizeTotal && g_relabel_mask[gid] == 1 && gX < gRealSizeX-1 && gX > 0 && gY < gRealSizeY-1 && gY > 0 )
    {
        push_sink(g_sink_weight, g_push_reser, g_graph_height, gid);

        push_neighbour(g_sink_weight, g_push_reser, g_graph_height,
                &g_left_weight[gid], &g_pull_right[GLOBAL_LEFT(gid)],
                gid, GLOBAL_LEFT(gid));


        push_neighbour(g_sink_weight, g_push_reser, g_graph_height,
                &g_up_weight[gid], &g_pull_down[GLOBAL_TOP(gid)],
                gid, GLOBAL_TOP(gid));


        push_neighbour(g_sink_weight, g_push_reser, g_graph_height,
                &g_right_weight[gid], &g_pull_left[GLOBAL_RIGHT(gid)],
                gid, GLOBAL_RIGHT(gid));


        push_neighbour(g_sink_weight, g_push_reser, g_graph_height,
                &g_down_weight[gid], &g_pull_up[GLOBAL_BOTTOM(gid)],
                gid, GLOBAL_BOTTOM(gid));
    }
}

__device__ void
pull_all(int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
        int *g_pull_left, int *g_pull_right, int *g_pull_down, int *g_pull_up,
        int *g_push_reser,
        int gRealSizeTotal, int gRealSizeX, int gRealSizeY, int gSizeTotal, int gSizeX, int gSizeY,
        int gid, int gX, int gY)
{

    pull(&g_pull_right[gid], &g_right_weight[gid], g_push_reser, gid);
    pull(&g_pull_down[gid], &g_down_weight[gid], g_push_reser, gid);
    pull(&g_pull_left[gid], &g_left_weight[gid], g_push_reser, gid);
    pull(&g_pull_up[gid], &g_up_weight[gid], g_push_reser, gid);
}


__global__ void
kernel_push( const int k, const int total_blocks_x, const int total_blocks_y,
        int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
        int *g_pull_left, int *g_pull_right, int *g_pull_down, int *g_pull_up,
        int *g_sink_weight, int *g_push_reser,
        int *g_relabel_mask, int *g_graph_height,
        int gRealSizeTotal, int gRealSizeX, int gRealSizeY, int gSizeTotal, int gSizeX, int gSizeY)
{
    const int gridSizeX = gridDim.x;
    const int gridSizeY = gridDim.y;

    for(int j=0; j<k; j++) {    // Number of iterations for whole kernel

        for( int bidY=blockIdx.y; bidY<total_blocks_y; bidY+=gridSizeY){
        for( int bidX=blockIdx.x; bidX<total_blocks_x; bidX+=gridSizeX){ // Loop over all tiles for this block (persistent threads style)

            int gX  = bidX*blockDim.x + threadIdx.x ;
            int gY  = bidY*blockDim.y + threadIdx.y ;
            int gid = gY*gSizeX + gX ;

            const int m = 1;
            for(int i=0; i<m; i++) {
                push_all(g_left_weight, g_right_weight, g_down_weight, g_up_weight,
                                g_pull_left, g_pull_right, g_pull_down, g_pull_up,
                                g_sink_weight, g_push_reser,
                                g_relabel_mask, g_graph_height,
                                gRealSizeTotal, gRealSizeX, gRealSizeY, gSizeTotal, gSizeX, gSizeY,
                                gid, gX, gY);

                __syncthreads();

                pull_all(g_left_weight, g_right_weight, g_down_weight, g_up_weight,
                        g_pull_left, g_pull_right, g_pull_down, g_pull_up,
                        g_push_reser,
                        gRealSizeTotal, gRealSizeX, gRealSizeY, gSizeTotal, gSizeX, gSizeY,
                        gid, gX, gY);

                __syncthreads();

                if(i<m-1) { // Don't run this in last iteration as relabel will run it
                    set_state(g_left_weight, g_right_weight, g_down_weight, g_up_weight,
                            g_sink_weight, g_push_reser,
                            g_relabel_mask, g_graph_height,
                            gid, gSizeX, gSizeY);

                    __syncthreads();
                }
            }

            relabel( g_left_weight, g_right_weight, g_down_weight,g_up_weight,
                        g_sink_weight, g_push_reser,
                        g_relabel_mask, g_graph_height,
                        gRealSizeTotal, gRealSizeX, gRealSizeY, gSizeTotal, gSizeX, gSizeY,
                        gid, gX, gY);

            __syncthreads();

        } // Loop over all tiles for this block
        }

    } // Number of iterations for whole kernel
}

__global__ void
kernel_pull_end(int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
        int *g_pull_left, int *g_pull_right, int *g_pull_down, int *g_pull_up,
        int *g_push_reser,
        int gRealSizeTotal, int gRealSizeX, int gRealSizeY, int gSizeTotal, int gSizeX, int gSizeY)
{
    const int gX  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
    const int gY  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
    const int gid = __umul24( gY , gSizeX ) + gX ;

    pull_all(g_left_weight, g_right_weight, g_down_weight, g_up_weight,
            g_pull_left, g_pull_right, g_pull_down, g_pull_up,
            g_push_reser,
            gRealSizeTotal, gRealSizeX, gRealSizeY, gSizeTotal, gSizeX, gSizeY,
            gid, gX, gY);
}

__global__ void
kernel_start_sink( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
        int *g_sink_weight, int *g_push_reser,
        int *g_relabel_mask, int *g_graph_height,
        int gRealSizeTotal, int gRealSizeX, int gRealSizeY, int gSizeTotal, int gSizeX, int gSizeY, int *d_relabel, int *d_stochastic, int *d_counter, bool *d_finish)
{
    const int gX  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
    const int gY  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
    const int gid = __umul24( gY , gSizeX ) + gX ;

    if( gid < gSizeTotal && g_relabel_mask[gid] == 1 && gX < gRealSizeX-1 && gX > 0 && gY < gRealSizeY-1 && gY > 0 )
    {
        push_sink(g_sink_weight, g_push_reser, g_graph_height, gid);
    }
}

__global__ void
kernel_relabel_stochastic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
		int *g_sink_weight, int *g_push_reser,
		int *g_relabel_mask, int *g_graph_height, int *g_height_write,
		int gRealSizeTotal, int gRealSizeX, int gRealSizeY, int gSizeTotal, int gSizeX, int gSizeY, int *d_stochastic, int *g_block_num )
{
	if(d_stochastic[blockIdx.y * (*g_block_num) + blockIdx.x] == 1 )
	{
        const int gX  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
        const int gY  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
        const int gid = __umul24( gY , gSizeX ) + gX ;
        const int lX = threadIdx.x;
        const int lY = threadIdx.y;
        const int lid = LOCAL_INDEX(lX, lY);

        __shared__ int height_fn[356];
        load_shared_mem(height_fn, g_graph_height, gSizeX, gSizeY);

		__syncthreads();


		int min_flow_pushed = g_left_weight[gid] ;
		int flow_push = g_push_reser[gid] ;

		if(flow_push <= 0 || (g_left_weight[gid] == 0 && g_right_weight[gid] == 0 && g_down_weight[gid] == 0 && g_up_weight[gid] == 0 && g_sink_weight[gid] == 0))
			g_relabel_mask[gid] = 2 ;
		else
		{
			( flow_push > 0 && ( ( (height_fn[lid] == height_fn[LOCAL_LEFT(lid)] + 1 ) && g_left_weight[gid] > 0  ) ||( (height_fn[lid] == height_fn[LOCAL_RIGHT(lid)]+1 ) && g_right_weight[gid] > 0) || ( ( height_fn[lid] == height_fn[LOCAL_BOTTOM(lid)]+1 ) && g_down_weight[gid] > 0) || ( (height_fn[lid] == height_fn[LOCAL_TOP(lid)]+1 ) && g_up_weight[gid] > 0 ) || ( height_fn[lid] == 1 && g_sink_weight[gid] > 0 )  ) ) ? g_relabel_mask[gid] = 1 : g_relabel_mask[gid] = 0 ;
		}


		__syncthreads();

		if(gid < gSizeTotal && gX < gRealSizeX - 1  && gX > 0 && gY < gRealSizeY - 1  && gY > 0  )
		{
			if(g_sink_weight[gid] > 0)
			{
				g_height_write[gid] = 1 ;
			}
			else
			{
				int min_height = gRealSizeTotal ;
				(min_flow_pushed > 0 && min_height > height_fn[LOCAL_LEFT(lid)] ) ? min_height = height_fn[LOCAL_LEFT(lid)] : 0 ;
				(g_right_weight[gid] > 0 && min_height > height_fn[LOCAL_RIGHT(lid)]) ? min_height = height_fn[LOCAL_RIGHT(lid)] : 0 ;
				(g_down_weight[gid] > 0 && min_height > height_fn[LOCAL_BOTTOM(lid)] ) ? min_height = height_fn[LOCAL_BOTTOM(lid)] : 0 ;
				(g_up_weight[gid] > 0 && min_height > height_fn[LOCAL_TOP(lid)] ) ? min_height = height_fn[LOCAL_TOP(lid)] : 0 ;
				g_height_write[gid] = min_height + 1 ;
			}
		}
	}
}

__global__ void
kernel_End( int *g_stochastic, int *g_count_blocks, int *g_counter)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( gid < ( *g_counter ) )
	{
		if( g_stochastic[gid] == 1 )
			atomicAdd(g_count_blocks,1);
			//(*g_count_blocks) = (*g_count_blocks) + 1 ; 
	}
}

__global__ void
kernel_push1_stochastic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
	int *g_sink_weight, int *g_push_reser, int *g_relabel_mask, int *g_graph_height, int *g_height_write,
	int gRealSizeTotal, int gRealSizeX, int gRealSizeY, int gSizeTotal, int gSizeX, int gSizeY,
	int *d_stochastic,int *g_block_num ) {
	if(d_stochastic[blockIdx.y * (*g_block_num) + blockIdx.x] == 1 ) {
		const int gX  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
		const int gY  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
		const int gid = __umul24( gY , gSizeX ) + gX ;
		const int lX = threadIdx.x;
		const int lY = threadIdx.y;
		const int lid = LOCAL_INDEX(lX, lY);

		__shared__ int height_fn[356];
		load_shared_mem(height_fn, g_graph_height, gSizeX, gSizeY);
		__syncthreads();
		int flow_push = 0, min_flow_pushed = 0 ;
		flow_push = g_push_reser[gid] ;
		if( gid < gSizeTotal && g_relabel_mask[gid] == 1 && gX < gRealSizeX-1 && gX > 0 && gY < gRealSizeY-1 && gY > 0 ) {
			int temp_weight = 0;
			temp_weight = g_sink_weight[gid] ;
			min_flow_pushed = flow_push ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == 1 ) {
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				g_sink_weight[gid] = temp_weight ;
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
			}
			__threadfence();
			flow_push = g_push_reser[gid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_left_weight[gid] ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == height_fn[LOCAL_LEFT(lid)] + 1 ) {
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_left_weight[gid] , min_flow_pushed);
				atomicAdd(&g_right_weight[GLOBAL_LEFT(gid)],min_flow_pushed);
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
				atomicAdd(&g_push_reser[GLOBAL_LEFT(gid)], min_flow_pushed);

			}
			__threadfence();
			flow_push = g_push_reser[gid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_up_weight[gid] ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == height_fn[LOCAL_TOP(lid)] + 1) {
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_up_weight[gid] , min_flow_pushed);
				atomicAdd(&g_down_weight[GLOBAL_TOP(gid)],min_flow_pushed);
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
				atomicAdd(&g_push_reser[GLOBAL_TOP(gid)], min_flow_pushed);
			}
			__threadfence();
			flow_push = g_push_reser[gid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_right_weight[gid] ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == height_fn[LOCAL_RIGHT(lid)] + 1 ) {
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_right_weight[gid] , min_flow_pushed);
				atomicAdd(&g_left_weight[GLOBAL_RIGHT(gid)],min_flow_pushed);
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
				atomicAdd(&g_push_reser[GLOBAL_RIGHT(gid)], min_flow_pushed);
			}
			__threadfence();
			flow_push = g_push_reser[gid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_down_weight[gid] ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == height_fn[LOCAL_BOTTOM(lid)] + 1 ) {
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_down_weight[gid] , min_flow_pushed);
				atomicAdd(&g_up_weight[GLOBAL_BOTTOM(gid)], min_flow_pushed);
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
				atomicAdd(&g_push_reser[GLOBAL_BOTTOM(gid)], min_flow_pushed);
			}
			__threadfence();
		}
	}
}

__global__ void
kernel_push2_stochastic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
	int *g_sink_weight, int *g_push_reser, int *g_relabel_mask, int *g_graph_height, int *g_height_write,
	int gRealSizeTotal, int gRealSizeX, int gRealSizeY, int gSizeTotal, int gSizeX, int gSizeY,
	int *d_relabel, int *d_stochastic, int *d_counter, bool *d_finish) {
	if(d_stochastic[blockIdx.y * 20 + blockIdx.x] == 1 ) {
		const int gX  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
		const int gY  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
		const int gid = __umul24( gY , gSizeX ) + gX ;
		const int lX = threadIdx.x;
		const int lY = threadIdx.y;
		const int lid = LOCAL_INDEX(lX, lY);
		__shared__ int height_fn[356];
		load_shared_mem(height_fn, g_graph_height, gSizeX, gSizeY);
		__syncthreads();
		int flow_push = 0, min_flow_pushed = 0 ;
		flow_push = g_push_reser[gid] ;
		if( gid < gSizeTotal && g_relabel_mask[gid] == 1 && gX < gRealSizeX-1 && gX > 0 && gY < gRealSizeY-1 && gY > 0 ) {
			int temp_weight = 0;
			temp_weight = g_sink_weight[gid] ;
			min_flow_pushed = flow_push ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == 1 ) {
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				g_sink_weight[gid] = temp_weight ;
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
			}
			__threadfence();
			flow_push = g_push_reser[gid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_left_weight[gid] ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == height_fn[LOCAL_LEFT(lid)] + 1 ) {
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_left_weight[gid] , min_flow_pushed);
				atomicAdd(&g_right_weight[GLOBAL_LEFT(gid)],min_flow_pushed);
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
				atomicAdd(&g_push_reser[GLOBAL_LEFT(gid)], min_flow_pushed);

			}
			__threadfence();
			flow_push = g_push_reser[gid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_up_weight[gid] ;

			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == height_fn[LOCAL_TOP(lid)] + 1) {
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;

				atomicSub(&g_up_weight[gid] , min_flow_pushed);
				atomicAdd(&g_down_weight[GLOBAL_TOP(gid)],min_flow_pushed);
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
				atomicAdd(&g_push_reser[GLOBAL_TOP(gid)], min_flow_pushed);

			}
 			__threadfence();
			flow_push = g_push_reser[gid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_right_weight[gid] ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == height_fn[LOCAL_RIGHT(lid)] + 1) {
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_right_weight[gid] , min_flow_pushed);
				atomicAdd(&g_left_weight[GLOBAL_RIGHT(gid)],min_flow_pushed);
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
				atomicAdd(&g_push_reser[GLOBAL_RIGHT(gid)], min_flow_pushed);
			}
			__threadfence();
			flow_push = g_push_reser[gid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_down_weight[gid] ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == height_fn[LOCAL_BOTTOM(lid)] + 1 ) {
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_down_weight[gid] , min_flow_pushed);
				atomicAdd(&g_up_weight[GLOBAL_BOTTOM(gid)], min_flow_pushed);
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
				atomicAdd(&g_push_reser[GLOBAL_BOTTOM(gid)], min_flow_pushed);
			}
			__threadfence();
		}	
		__syncthreads() ; 
		min_flow_pushed = g_left_weight[gid] ;
		flow_push = g_push_reser[gid] ;

		if(flow_push <= 0 || (g_left_weight[gid] == 0 && g_right_weight[gid] == 0 && g_down_weight[gid] == 0 && g_up_weight[gid] == 0 && g_sink_weight[gid] == 0))
			g_relabel_mask[gid] = 2 ;
		else {
			( flow_push > 0 && ( ( (height_fn[lid] == height_fn[LOCAL_LEFT(lid)] + 1 ) && g_left_weight[gid] > 0  ) ||( (height_fn[lid] == height_fn[LOCAL_RIGHT(lid)]+1 ) && g_right_weight[gid] > 0) || ( ( height_fn[lid] == height_fn[LOCAL_BOTTOM(lid)]+1 ) && g_down_weight[gid] > 0) || ( (height_fn[lid] == height_fn[LOCAL_TOP(lid)]+1 ) && g_up_weight[gid] > 0 ) || ( height_fn[lid] == 1 && g_sink_weight[gid] > 0 )  ) ) ? g_relabel_mask[gid] = 1 : g_relabel_mask[gid] = 0 ;
		}
		__syncthreads() ;
		if( gid < gSizeTotal && g_relabel_mask[gid] == 1 && gX < gRealSizeX-1 && gX > 0 && gY < gRealSizeY-1 && gY > 0 )
		{
			int temp_weight = 0;
			temp_weight = g_sink_weight[gid] ;
			min_flow_pushed = flow_push ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == 1 ) {
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				g_sink_weight[gid] = temp_weight ;
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
			}
			__threadfence();
			flow_push = g_push_reser[gid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_left_weight[gid] ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == height_fn[LOCAL_LEFT(lid)] + 1 ) {
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_left_weight[gid] , min_flow_pushed);
				atomicAdd(&g_right_weight[GLOBAL_LEFT(gid)],min_flow_pushed);
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
				atomicAdd(&g_push_reser[GLOBAL_LEFT(gid)], min_flow_pushed);
			}
			__threadfence();
			flow_push = g_push_reser[gid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_up_weight[gid] ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == height_fn[LOCAL_TOP(lid)] + 1) {
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_up_weight[gid] , min_flow_pushed);
				atomicAdd(&g_down_weight[GLOBAL_TOP(gid)],min_flow_pushed);
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
				atomicAdd(&g_push_reser[GLOBAL_TOP(gid)], min_flow_pushed);
			}
			__threadfence();
			flow_push = g_push_reser[gid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_right_weight[gid] ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == height_fn[LOCAL_RIGHT(lid)] + 1) {
				(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_right_weight[gid] , min_flow_pushed);
				atomicAdd(&g_left_weight[GLOBAL_RIGHT(gid)],min_flow_pushed);
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
				atomicAdd(&g_push_reser[GLOBAL_RIGHT(gid)], min_flow_pushed);
			}
			__threadfence();
			flow_push = g_push_reser[gid] ;
			min_flow_pushed = flow_push ;
			temp_weight = g_down_weight[gid] ;
			if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == height_fn[LOCAL_BOTTOM(lid)] + 1 ) {
				(temp_weight<flow_push) ? min_flow_pushed = temp_weight : 0 ;
				temp_weight = temp_weight - min_flow_pushed ;
				atomicSub(&g_down_weight[gid] , min_flow_pushed);
				atomicAdd(&g_up_weight[GLOBAL_BOTTOM(gid)], min_flow_pushed);
				atomicSub(&g_push_reser[gid] , min_flow_pushed);
				atomicAdd(&g_push_reser[GLOBAL_BOTTOM(gid)], min_flow_pushed);
			}
			__threadfence();
		}
	}
}

__global__ void
kernel_bfs_t(int *g_push_reser, int  *g_sink_weight, int *g_graph_height, bool *g_pixel_mask,
		int vertex_num, int gRealSizeX, int gRealSizeY, int vertex_num1, int gSizeX, int gSizeY) {
	int gid = __umul24(blockIdx.x, blockDim.x) + threadIdx.x ;
	if(gid < vertex_num && g_pixel_mask[gid] == true ) {
		int col = gid % gSizeX , row = gid / gSizeX ;
		if(col > 0 && row > 0 && col < gRealSizeX - 1 && row < gRealSizeY - 1 && g_push_reser[gid] > 0 ) {
			g_graph_height[gid] = 1 ;
			g_pixel_mask[gid] = false ;
		}
		else
			if(g_sink_weight[gid] > 0) {
				g_graph_height[gid] = -1 ;
				g_pixel_mask[gid] = false ;
			}
	}
}	

__global__ void
kernel_push_stochastic1( int *g_push_reser, int *s_push_reser, int *g_count_blocks, bool *g_finish, int *g_block_num, int gSizeX) {
	int gX  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
	int gY  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
	int gid = __umul24( gY , gSizeX ) + gX ;
	s_push_reser[gid] = g_push_reser[gid] ;
	if( gid == 0 ) {
		if((*g_count_blocks) < 50 )
			(*g_finish) = false ; 
	}
}

__global__ void
kernel_push_stochastic2( int *g_push_reser, int *s_push_reser, int *d_stochastic, int *g_block_num, int gSizeX) {
	int gX  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
	int gY  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
	int gid = __umul24( gY , gSizeX ) + gX ;
	int stochastic = 0 ;
	stochastic = ( s_push_reser[gid] - g_push_reser[gid]) ;
	if(stochastic != 0) {
		d_stochastic[blockIdx.y * (*g_block_num) + blockIdx.x] = 1 ;
	}
}

__global__ void
kernel_push1_start_stochastic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
		int *g_sink_weight, int *g_push_reser,
		int *g_relabel_mask, int *g_graph_height, int *g_height_write,
		int gRealSizeTotal, int gRealSizeX, int gRealSizeY, int gSizeTotal, int gSizeX, int gSizeY, int *d_relabel, int *d_stochastic, int *d_counter, bool *d_finish) {
    const int gX  = __umul24( blockIdx.x, blockDim.x ) + threadIdx.x ;
    const int gY  = __umul24( blockIdx.y , blockDim.y ) + threadIdx.y ;
    const int gid = __umul24( gY , gSizeX ) + gX ;
    const int lX = threadIdx.x;
    const int lY = threadIdx.y;
    const int lid = LOCAL_INDEX(lX, lY);

    __shared__ int height_fn[356];
    load_shared_mem(height_fn, g_graph_height, gSizeX, gSizeY);
	__syncthreads();
	int flow_push = 0, min_flow_pushed = 0 ;
	flow_push = g_push_reser[gid] ;
	if( gid < gSizeTotal && g_relabel_mask[gid] == 1 && gX < gRealSizeX-1 && gX > 0 && gY < gRealSizeY-1 && gY > 0) {
		int temp_weight = 0;
		temp_weight = g_sink_weight[gid] ;
		min_flow_pushed = flow_push ;
		if(temp_weight > 0 && flow_push > 0 && height_fn[lid] == 1 ) {
			(temp_weight < flow_push) ? min_flow_pushed = temp_weight : 0 ;
			temp_weight = temp_weight - min_flow_pushed ;
			g_sink_weight[gid] = temp_weight ;
			atomicSub(&g_push_reser[gid] , min_flow_pushed);
			flow_push = flow_push - min_flow_pushed ;
		}
	}
	__syncthreads() ;
	min_flow_pushed = g_left_weight[gid] ;

	( flow_push > 0 && ( ((height_fn[lid] == height_fn[LOCAL_LEFT(lid)] + 1 ) && min_flow_pushed > 0  ) ||( (height_fn[lid] == height_fn[LOCAL_RIGHT(lid)]+1 ) && g_right_weight[gid] > 0) || ( ( height_fn[lid] == height_fn[LOCAL_BOTTOM(lid)]+1 ) && g_down_weight[gid] > 0) || ( (height_fn[lid] == height_fn[LOCAL_TOP(lid)]+1 ) && g_up_weight[gid] > 0 ) || ( height_fn[lid] == 1 && g_sink_weight[gid] > 0 )  ) ) ? g_relabel_mask[gid] = 1 : g_relabel_mask[gid] = 0 ;
	if(gid < gSizeTotal && gX < gRealSizeX - 1  && gX > 0 && gY < gRealSizeY - 1  && gY > 0  ) {
		if(g_sink_weight[gid] > 0) {
			g_height_write[gid] = 1 ;
		}
		else {
			int min_height = gRealSizeTotal ;
			(min_flow_pushed > 0 && min_height > height_fn[LOCAL_LEFT(lid)] ) ? min_height = height_fn[LOCAL_LEFT(lid)] : 0 ;
			(g_right_weight[gid] > 0 && min_height > height_fn[LOCAL_RIGHT(lid)]) ? min_height = height_fn[LOCAL_RIGHT(lid)] : 0 ;
			(g_down_weight[gid] > 0 && min_height > height_fn[LOCAL_BOTTOM(lid)] ) ? min_height = height_fn[LOCAL_BOTTOM(lid)] : 0 ;
			(g_up_weight[gid] > 0 && min_height > height_fn[LOCAL_TOP(lid)] ) ? min_height = height_fn[LOCAL_TOP(lid)] : 0 ;
			g_height_write[gid] = min_height + 1 ;
		}
	}

}

__global__ void
kernel_bfs(int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight,
		int *g_graph_height, bool *g_pixel_mask, int vertex_num,int gRealSizeX,int gRealSizeY,
		int vertex_num1, int gSizeX, int gSizeY, bool *g_over, int *g_counter) {
	/*******************************
	 *threadId is calculated ******
	 *****************************/

	int gid = __umul24(blockIdx.x, blockDim.x) + threadIdx.x ;
	if(gid < vertex_num && g_pixel_mask[gid] == true) {
		int col = gid % gSizeX , row = gid / gSizeX ;
		if(col < gRealSizeX - 1 && col > 0 && row < gRealSizeY - 1 && row > 0 ) {
			int height_l = 0, height_d = 0, height_u = 0 , height_r = 0 ;
			height_r = g_graph_height[GLOBAL_RIGHT(gid)] ;
			height_l = g_graph_height[GLOBAL_LEFT(gid)] ;
			height_d = g_graph_height[GLOBAL_BOTTOM(gid)] ;
			height_u = g_graph_height[GLOBAL_TOP(gid)] ;

			if(((height_l == (*g_counter) && g_right_weight[GLOBAL_LEFT(gid)] > 0)) ||((height_d == (*g_counter) && g_up_weight[GLOBAL_BOTTOM(gid)] > 0) || ( height_r == (*g_counter) && g_left_weight[GLOBAL_RIGHT(gid)] > 0 ) || ( height_u == (*g_counter) && g_down_weight[GLOBAL_TOP(gid)] > 0 ) ))
			{
				g_graph_height[gid] = (*g_counter) + 1 ;
				g_pixel_mask[gid] = false ;
				*g_over = true ;
			}
		}
	}
}

/************************************************************
 * functions to construct the graph on the device          **
 * *********************************************************/
__device__ void add_edge(int from, int to, int cap, int rev_cap, int type, int *d_left_weight,
		int *d_right_weight, int *d_down_weight, int *d_up_weight) {
	if(type==1) {
		d_left_weight[from] = d_left_weight[from]+cap;
		d_right_weight[to] = d_right_weight[to]+rev_cap;
	}
	if(type==2) {
		d_right_weight[from] = d_right_weight[from]+cap;
		d_left_weight[to] = d_left_weight[to]+rev_cap;
	}
	if(type==3) {
		d_down_weight[from] = d_down_weight[from]+cap;
		d_up_weight[to] = d_up_weight[to]+rev_cap;
	}
	if(type==4) {
		d_up_weight[from] = d_up_weight[from]+cap;
		d_down_weight[to] = d_down_weight[to]+cap;
	}
}

__device__ void add_tweights(int i, int cap_source, int  cap_sink, int *d_push_reser, int *d_sink_weight) {
	int diff = cap_source - cap_sink ;
	if(diff>0) {
		d_push_reser[i] = d_push_reser[i] + diff ;
	}
	else {
		d_sink_weight[i] = d_sink_weight[i] - diff ;
	}
}

__device__
void add_term1(int i, int A, int B, int *d_push_reser, int *d_sink_weight) {
	add_tweights(i,B,A, d_push_reser, d_sink_weight);
}

__device__
void add_t_links_Cue(int alpha_label, int gid, int *d_left_weight, int *d_right_weight,
		int *d_down_weight, int *d_up_weight, int *d_push_reser, int *d_sink_weight,
		int *dPixelLabel, int *dDataTerm, int gRealSizeX , int gRealSizeY, int num_labels) {
	{
		if(dPixelLabel[gid]!=alpha_label) {
			add_term1(gid , dDataTerm[gid*num_labels+alpha_label] , dDataTerm[gid * num_labels + dPixelLabel[gid]], d_push_reser, d_sink_weight  );
		}
	}
}

__device__
void add_t_links(int alpha_label, int gid, int *d_left_weight, int *d_right_weight,
		int *d_down_weight, int *d_up_weight, int *d_push_reser, int *d_sink_weight,
		int *dPixelLabel, int *dDataTerm, int gRealSizeX , int gRealSizeY, int num_labels) {
	{
		if(dPixelLabel[gid]!=alpha_label) {
			add_term1(gid , dDataTerm[gid*num_labels+alpha_label] , dDataTerm[gid * num_labels + dPixelLabel[gid]], d_push_reser, d_sink_weight  );
		}
	}
}

__device__
void add_term2(int x, int y, int A, int B, int C, int D, int type, int *d_left_weight,
		int *d_right_weight, int *d_down_weight, int *d_up_weight, int *d_push_reser, int *d_sink_weight) {
	if ( A+D > C+B) {
		int delta = A+D-C-B;
		int subtrA = delta/3;
		A = A-subtrA;
		C = C+subtrA;
		B = B+(delta-subtrA*2);
#ifdef COUNT_TRUNCATIONS
		truncCnt++;
#endif
	}
#ifdef COUNT_TRUNCATIONS
	totalCnt++;
#endif
	add_tweights(x, D, A, d_push_reser, d_sink_weight);
	B -= A; C -= D;
	if (B < 0) {
		add_tweights(x, 0, B, d_push_reser, d_sink_weight);
		add_tweights(y, 0, -B, d_push_reser, d_sink_weight ) ;
		add_edge(x, y, 0, B+C,type , d_left_weight, d_right_weight, d_down_weight, d_up_weight );
	}
	else if (C < 0) {
		add_tweights(x, 0, -C, d_push_reser, d_sink_weight);
		add_tweights(y, 0, C , d_push_reser, d_sink_weight);
		add_edge(x, y, B+C, 0,type , d_left_weight, d_right_weight, d_down_weight, d_up_weight);
	}
	else {
		add_edge(x, y, B, C,type, d_left_weight, d_right_weight , d_down_weight, d_up_weight);
	}
}

__device__
void set_up_expansion_energy_G_ARRAY(int alpha_label,int gid, int *d_left_weight,int *d_right_weight,
		int *d_down_weight, int *d_up_weight, int *d_push_reser,
		int *d_sink_weight, int *dPixelLabel, int *dDataTerm, int *dSmoothTerm,
		int gRealSizeX , int gRealSizeY, int num_labels ) {
	int x,y,nPix;
	int weight;
	int i = gid ;
	{
		if(dPixelLabel[i]!=alpha_label) {
			y = i/gRealSizeX;
			x = i - y*gRealSizeX;
			if ( x < gRealSizeX - 1 ) {
				nPix = i + 1;
				weight = 1 ;
				if ( dPixelLabel[nPix] != alpha_label ) {
					add_term2(i,nPix,
					( dSmoothTerm[alpha_label + alpha_label * num_labels]) * weight,
					( dSmoothTerm[alpha_label + dPixelLabel[nPix]*num_labels]) * weight,
					( dSmoothTerm[ dPixelLabel[i] +  alpha_label * num_labels] ) * weight,
					( dSmoothTerm[ dPixelLabel[i] +  dPixelLabel[nPix] * num_labels] )  * weight,
							2, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight); // 1-left, 2-right, 3-down, 4-up
				}
				else   add_term1(i,
						( dSmoothTerm[alpha_label + dPixelLabel[nPix] * num_labels]) * weight,
						( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
						d_push_reser, d_sink_weight);
			}

			if ( y < gRealSizeY - 1 ) {
				nPix = i + gRealSizeX;
				weight = 1 ;
				if ( dPixelLabel[nPix] != alpha_label ) {
					add_term2(i,nPix,
					( dSmoothTerm[alpha_label + alpha_label * num_labels]) * weight,
					( dSmoothTerm[alpha_label + dPixelLabel[nPix]*num_labels]) * weight,
					( dSmoothTerm[ dPixelLabel[i] +  alpha_label * num_labels] ) * weight,
					( dSmoothTerm[ dPixelLabel[i] +  dPixelLabel[nPix] * num_labels] )  * weight,
					3, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight );
				}
				else   add_term1(i,
						( dSmoothTerm[alpha_label + dPixelLabel[nPix] * num_labels]) * weight,
						( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
						d_push_reser, d_sink_weight);
			}
			if ( x > 0 ) {
				nPix = i - 1;
				weight = 1 ;
				if ( dPixelLabel[nPix] == alpha_label )
					add_term1(i,
						( dSmoothTerm[alpha_label + dPixelLabel[nPix] * num_labels]) * weight,
						( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
						d_push_reser, d_sink_weight );
			}

			if ( y > 0 ) {
				nPix = i - gRealSizeX;
				weight = 1 ;
				if ( dPixelLabel[nPix] == alpha_label ) {
					add_term1(i,
						( dSmoothTerm[alpha_label + alpha_label * num_labels]) * weight,
						( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
						d_push_reser, d_sink_weight);
				}
			}
		}
	}
}

__device__
void set_up_expansion_energy_G_ARRAY_Cue(int alpha_label,int gid, int *d_left_weight,int *d_right_weight,
		int *d_down_weight, int *d_up_weight, int *d_push_reser,
		int *d_sink_weight, int *dPixelLabel, int *dDataTerm, int *dSmoothTerm,
		int *dHcue, int *dVcue, int gRealSizeX , int gRealSizeY, int num_labels ) {
	int x,y,nPix;
	int weight;
	int i = gid ;
	{
		if(dPixelLabel[i]!=alpha_label) {
			y = i/gRealSizeX;
			x = i - y*gRealSizeX;

			if ( x < gRealSizeX - 1 ) {
				nPix = i + 1;
				weight=dHcue[i];
				if ( dPixelLabel[nPix] != alpha_label ) {
					add_term2(i,nPix,
					( dSmoothTerm[alpha_label + alpha_label * num_labels]) * weight,
					( dSmoothTerm[alpha_label + dPixelLabel[nPix]*num_labels]) * weight,
					( dSmoothTerm[ dPixelLabel[i] +  alpha_label * num_labels] ) * weight,
					( dSmoothTerm[ dPixelLabel[i] +  dPixelLabel[nPix] * num_labels] )  * weight,

					2, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight); // 1-left, 2-right, 3-down, 4-up
				}
				else   add_term1(i,
						( dSmoothTerm[alpha_label + dPixelLabel[nPix] * num_labels]) * weight,
						( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
						d_push_reser, d_sink_weight);
			}
			if ( y < gRealSizeY - 1 ) {
				nPix = i + gRealSizeX;
				weight=dVcue[i];
				if ( dPixelLabel[nPix] != alpha_label ) {
					add_term2(i,nPix,
					( dSmoothTerm[alpha_label + alpha_label * num_labels]) * weight,
					( dSmoothTerm[alpha_label + dPixelLabel[nPix]*num_labels]) * weight,
					( dSmoothTerm[ dPixelLabel[i] +  alpha_label * num_labels] ) * weight,
					( dSmoothTerm[ dPixelLabel[i] +  dPixelLabel[nPix] * num_labels] )  * weight,
			3, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight );
				}
				else   add_term1(i,
						( dSmoothTerm[alpha_label + dPixelLabel[nPix] * num_labels]) * weight,
						( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
						d_push_reser, d_sink_weight);
			}
			if ( x > 0 ) {
				nPix = i - 1;
				weight=dHcue[nPix];
				if ( dPixelLabel[nPix] == alpha_label )
					add_term1(i,
						( dSmoothTerm[alpha_label + dPixelLabel[nPix] * num_labels]) * weight,
						( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
						d_push_reser, d_sink_weight );
			}
			if ( y > 0 ) {
				nPix = i - gRealSizeX;
				weight = dVcue[nPix] ;
				if ( dPixelLabel[nPix] == alpha_label ) {
					add_term1(i,
						( dSmoothTerm[alpha_label + alpha_label * num_labels]) * weight,
						( dSmoothTerm[dPixelLabel[i] + alpha_label*num_labels]) * weight,
						d_push_reser, d_sink_weight);
				}
			}
		}
	}
}

__global__
void CudaWeightCue(int alpha_label, int *d_left_weight, int *d_right_weight, int *d_down_weight,
		int *d_up_weight, int *d_push_reser, int *d_sink_weight, int *dPixelLabel,
		int *dDataTerm, int *dSmoothTerm, int *dHcue, int *dVcue, int gRealSizeX, int gRealSizeY, int num_labels ) {
	int gid = blockIdx.x * 256 + threadIdx.x ;
	add_t_links_Cue(alpha_label, gid, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm, gRealSizeX, gRealSizeY, num_labels);
	set_up_expansion_energy_G_ARRAY_Cue(alpha_label, gid, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm, dSmoothTerm, dHcue, dVcue, gRealSizeX, gRealSizeY, num_labels);
}

__global__
void CudaWeight(int alpha_label, int *d_left_weight, int *d_right_weight, int *d_down_weight,
		int *d_up_weight, int *d_push_reser, int *d_sink_weight, int *dPixelLabel,
		int *dDataTerm, int *dSmoothTerm, int gRealSizeX, int gRealSizeY, int num_labels) {
	int gid = blockIdx.x * 256 + threadIdx.x ;
	add_t_links(alpha_label, gid, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm, gRealSizeX, gRealSizeY, num_labels);
	set_up_expansion_energy_G_ARRAY(alpha_label, gid, d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm, dSmoothTerm, gRealSizeX, gRealSizeY, num_labels);
}



/*********************************************************
 * function which adjusts the array size for efficiency **
 * consideration                                        **
 * ******************************************************/

__global__
void adjustedgeweight(int *d_left_weight, int *d_right_weight, int *d_down_weight, int *d_up_weight,
		int *d_push_reser, int *d_sink_weight, int *temp_left_weight, int *temp_right_weight,
		int *temp_down_weight, int *temp_up_weight, int *temp_push_reser, int *temp_sink_weight,
		int gRealSizeX, int gRealSizeY, int gRealSizeTotal, int gSizeX, int gSizeY, int gSizeTotal) {
	int gid = blockIdx.x * 256 + threadIdx.x ;
	if( gid < gSizeTotal ) {
		int row = gid / gSizeX , col = gid % gSizeX ;
		if(row < gRealSizeY && col < gRealSizeX) {
			temp_left_weight[row* gSizeX + col] = d_left_weight[row * gRealSizeX + col] ;
			temp_right_weight[row * gSizeX + col] = d_right_weight[row * gRealSizeX + col] ;
			temp_down_weight[row * gSizeX + col] = d_down_weight[row * gRealSizeX + col] ;
			temp_up_weight[row * gSizeX + col] = d_up_weight[row * gRealSizeX + col] ;
			temp_push_reser[row * gSizeX + col] = d_push_reser[row * gRealSizeX + col] ;
			temp_sink_weight[row * gSizeX + col] = d_sink_weight[row * gRealSizeX + col] ;

		}
		else {
			temp_left_weight[row * gSizeX + col] = 0 ;
			temp_right_weight[row * gSizeX + col] = 0 ;
			temp_down_weight[row * gSizeX + col] = 0 ;
			temp_up_weight[row * gSizeX + col] = 0 ;
			temp_push_reser[row * gSizeX + col] = 0 ;
			temp_sink_weight[row * gSizeX + col] = 0 ;
		}
	}
}

// Intializes memory on the gpu
__global__
void copyedgeweight( int *d_left_weight, int *d_right_weight, int *d_down_weight, int *d_up_weight,
		int *d_push_reser, int *d_sink_weight, int *temp_left_weight, int *temp_right_weight,
		int *temp_down_weight, int *temp_up_weight, int *temp_push_reser, int *temp_sink_weight,
		int *d_relabel_mask,
		int *d_graph_heightr, int *d_graph_heightw, int gRealSizeX, int height, int gRealSizeTotal, int gSizeX, int gSizeY, int gSizeTotal) {
	int gid = blockIdx.x * 256 + threadIdx.x ;
	if( gid < gSizeTotal ) {
		d_left_weight[gid] = temp_left_weight[gid] ;
		d_right_weight[gid] = temp_right_weight[gid] ;
		d_down_weight[gid] = temp_down_weight[gid] ;
		d_up_weight[gid] = temp_up_weight[gid] ;
		d_push_reser[gid] = temp_push_reser[gid] ;
		d_sink_weight[gid] = temp_sink_weight[gid] ;
		d_relabel_mask[gid] = 0 ;
		d_graph_heightr[gid] = 1 ;
		d_graph_heightw[gid] = 1 ;
	}
}

#endif
