#include "hip/hip_runtime.h"

/********************************************************************************************
* Implementing Graph Cuts on CUDA using algorithm given in CVGPU '08                       ** 
* paper "CUDA Cuts: Fast Graph Cuts on GPUs"                                               **
* Copyright (c) 2008 International Institute of Information Technology.                    **  
* All rights reserved.                                                                     **
* Created By Vibhav Vineet.                                                                ** 
********************************************************************************************/

#include "CudaCuts.cu"
#include "Example.h"
#include <assert.h>

using namespace std; 

int main(int argc, char** argv) {
	assert(argc >= 2);
	load_files(argv[1]);
	int initCheck = cudaCutsInit(gRealSizeX, gRealSizeY ,num_Labels) ;
//	printf("Compute Capability %d\n",initCheck);
	if( initCheck > 0 ) {
//		printf("The grid is initialized successfully\n");
	}
	else 
		if( initCheck == -1 ) {
			printf("Error: Please check the device present on the system\n");
		}
	int dataCheck   =  cudaCutsSetupDataTerm( dataTerm );
	if( dataCheck == 0 ) {
//		printf("The dataterm is set properly\n");	
	}
	else 
		if( dataCheck == -1 ) {
			printf("Error: Please check the device present on the system\n");
		}
	int smoothCheck =  cudaCutsSetupSmoothTerm( smoothTerm );
	if( smoothCheck == 0 ) {
//		printf("The smoothnessterm is set properly\n");
	}
	else
		if( smoothCheck == -1 ) {
			printf("Error: Please check the device present on the system\n");
		}
	int hcueCheck   =  cudaCutsSetupHCue( hCue );
	if( hcueCheck == 0 ) {
//		printf("The HCue is set properly\n");
	}
	else
		if( hcueCheck == -1 ) {
			printf("Error: Please check the device present on the system\n");
		}
	int vcueCheck   =  cudaCutsSetupVCue( vCue );
	if( vcueCheck == 0 ) {
//		printf("The VCue is set properly\n");
	}
	else 
		if( vcueCheck == -1 ) {
			printf("Error: Please check the device present on the system\n");
		}
	int graphCheck = cudaCutsSetupGraph();
	if( graphCheck == 0 ) {
//		printf("The graph is constructed successfully\n");
	}
	else 
		if( graphCheck == -1 ) {
			printf("Error: Please check the device present on the system\n");
		}
	int optimizeCheck = -1; 
	if( initCheck == 1 ) {
		// Get energy before starting
//		printf("\nStarting energies...\n");
		cudaCutsGetEnergy( );
//		printf("\n\n");
		//CudaCuts involving atomic operations are called
		optimizeCheck = cudaCutsAtomicOptimize();
		//CudaCuts involving stochastic operations are called
		//optimizeCheck = cudaCutsStochasticOptimize();
	}
	if( optimizeCheck == 0 ) {
		printf("The algorithm successfully converged\n");
	}
	else 
		if( optimizeCheck == -1 ) {
			printf("Error: Please check the device present on the system\n");
		}
	int resultCheck = cudaCutsGetResult( );
	if( resultCheck == 0 ) {
		printf("The pixel labels are successfully stored\n");
	}
	else 
		if( resultCheck == -1 ) {
			printf("Error: Please check the device present on the system\n");
		}
//	printf("\nFinal energies...\n");
	int energy = cudaCutsGetEnergy();
//	printf("\n\n");
	initFinalImage();
	cudaCutsFreeMem();
	exit(1);
//	CUT_EXIT(argc,argv);
}

void load_files(char *filename) {
	LoadDataFile(filename, gRealSizeX, gRealSizeY, num_Labels, dataTerm, smoothTerm, hCue, vCue);
}

void initFinalImage() {
	out_pixel_values=(int**)malloc(sizeof(int*)*gRealSizeY);
	for(int i = 0 ; i < gRealSizeY ; i++ ) {
		out_pixel_values[i] = (int*)malloc(sizeof(int) * gRealSizeX );
		for(int j = 0 ; j < gRealSizeX ; j++ ) {
			out_pixel_values[i][j]=0;
		}
	}
	writeImage() ;
}

void writeImage() {
	for(int i = 0 ; i <  gSizeTotal ; i++) {
		int row = i / gSizeX, col = i % gSizeX ;
		if(row >= 0 && col >= 0 && row <= gRealSizeY -1 && col <= gRealSizeX - 1 )
			out_pixel_values[row][col]=pixelLabel[i]*255;
	}
	write_image();
}

void write_image() {
	FILE* fp=fopen("result_cuda_test.pgm","w");
	fprintf(fp,"%c",'P');
	fprintf(fp,"%c",'2');
	fprintf(fp,"%c",'\n');
	fprintf(fp,"%d %c %d %c ",gRealSizeX,' ',gRealSizeY,'\n');
	fprintf(fp,"%d %c",255,'\n');
	for(int i=0;i<gRealSizeY;i++) {
		for(int j=0;j<gRealSizeX;j++) {
			fprintf(fp,"%d\n",out_pixel_values[i][j]);
		}
	}
	fclose(fp);
}

void write_data_to_file(char *filename, int my_width, int my_height, int skip, int max, int *data) {
	FILE* fp=fopen(filename,"w");
	fprintf(fp,"%c",'P');
	fprintf(fp,"%c",'2');
	fprintf(fp,"%c",'\n');
	fprintf(fp,"%d %c %d %c ",my_width,' ',my_height,'\n');
	fprintf(fp,"%d %c",max,'\n');
	for(int i=0;i<gRealSizeY * gRealSizeX * skip; i += skip) {
		fprintf(fp,"%d\n",data[i]);
	}
	fclose(fp);
}

void SubsampleData(int ratio, int width, int height, int *srcData, int *&dstData) {
   assert( width % ratio == 0 ); 
   assert( height % ratio == 0 );
   int dstWidth = width / ratio; 
   int dstHeight = height / ratio; 

   // allocation and initialization
   dstData = new int[dstWidth * dstHeight]; 
   for (int n = 0; n < dstHeight * dstWidth; n++) 
      dstData[n] = 0; 

   // summation of multiple pixel to a single destionation
   for (int y = 0; y < height; y++) {
      int dy = y / ratio; 
      for (int x = 0; x < width; x++) {
         int dx = x / ratio; 
         int n = x + y * width; 
         dstData[dx + dy * dstWidth] += srcData[n]; 
      }
   }

   // normalize 
   int ratiosq = ratio * ratio; 
   for (int n = 0; n < dstHeight * dstWidth; n++) 
      dstData[n] /= ratiosq; 
}


void WriteSubsampleDataSet(char *filename, int width, int height, int nLabels, int ratio, int *dataCostArray, int *smoothCostArray, int *hCue, int *vCue) {
	int dstWidth = width / ratio; 
	int dstHeight = height / ratio; 
	int i, n, x, y;
	FILE *fp = fopen(filename, "w"); 

   // demux the data cost array 
   int *dataCostArray0 = new int[width * height]; 
   int *dataCostArray1 = new int[width * height]; 
   for (n = 0; n < width * height; n++) {
      dataCostArray0[n] = dataCostArray[n * nLabels + 0]; 
      dataCostArray1[n] = dataCostArray[n * nLabels + 1]; 
   }
   int *d_Cost0;
   int *d_Cost1; 
   int *d_hCue; 
   int *d_vCue;
   SubsampleData(ratio, width, height, dataCostArray0, d_Cost0); 
   SubsampleData(ratio, width, height, dataCostArray1, d_Cost1); 
   SubsampleData(ratio, width, height, hCue, d_hCue); 
   SubsampleData(ratio, width, height, vCue, d_vCue); 

	fprintf(fp,"%d %d %d \n",dstWidth,dstHeight,nLabels);
	printf("[WriteSubsampleDataSet] width=%d height=%d nLabels=%d\n", dstWidth, dstHeight, nLabels);

	int gt = 1;
	for(i = 0; i < dstWidth * dstHeight; i++)
		fprintf(fp,"%d ",gt);
	fprintf(fp, "\n"); 

	assert(nLabels == 2); 
	for(n = 0; n < dstWidth * dstHeight; n++) {
		fprintf(fp,"%d ",d_Cost0[n]);
	}
	fprintf(fp, "\n"); 
	for(n = 0; n < dstWidth * dstHeight; n++) {
		fprintf(fp,"%d ",d_Cost1[n]);
	}
	fprintf(fp, "\n"); 

	n = 0;
	for(y = 0; y < dstHeight; y++) {
		for(x = 0; x < dstWidth-1; x++) {
			fprintf(fp,"%d ",d_hCue[n++]);
		}
		n++; // skip one blank column.... 
	}
	fprintf(fp, "\n"); 

	n = 0;
	for(y = 0; y < dstHeight-1; y++) {
		for(x = 0; x < dstWidth; x++) {
			fprintf(fp,"%d ",d_vCue[n++]);
		}
	}
	// skip the last row
	fprintf(fp, "\n");
	fclose(fp);
	delete[] d_Cost0;
	delete[] d_Cost1; 
	delete[] d_hCue; 
	delete[] d_vCue; 
	delete[] dataCostArray0; 
	delete[] dataCostArray1; 
}

void LoadDataFile(char *filename, int &width, int &height, int &nLabels, int *&dataCostArray, int *&smoothCostArray, int *&hCue, int *&vCue) {
//	printf("enterd\n");
	FILE *fp = fopen(filename,"r");
	fscanf(fp,"%d %d %d",&width,&height,&nLabels);
	printf("width=%d height=%d nLabels=%d\n", width, height, nLabels);
	int i, n, x, y;
	int gt;
	for(i = 0; i < width * height; i++)
		fscanf(fp,"%d",&gt);
	dataCostArray = new int[width * height * nLabels];
	for(int c=0; c < nLabels; c++) {
		n = c;
		for(i = 0; i < width * height; i++) {
			fscanf(fp,"%d",&dataCostArray[n]);
			n += nLabels;
		}
	}
	write_data_to_file("datacost0.pgm", width, height, 2, 16384, dataCostArray); 
	write_data_to_file("datacost1.pgm", width, height, 2, 16384, dataCostArray + 1); 
	hCue = new int[width * height];
	vCue = new int[width * height];
	n = 0;
	for(y = 0; y < height; y++) {
		for(x = 0; x < width-1; x++) {
			fscanf(fp,"%d",&hCue[n++]);
		}
		hCue[n++] = 0;
	}
	write_data_to_file("hCue.pgm", width, height, 1, 1024, hCue);
	n = 0;
	for(y = 0; y < height-1; y++) {
		for(x = 0; x < width; x++) {
			fscanf(fp,"%d",&vCue[n++]);
		}
	}
	for(x = 0; x < width; x++) {
		vCue[n++] = 0;
	}
	write_data_to_file("vCue.pgm", width, height, 1, 1024, hCue); 
	fclose(fp);
	smoothCostArray = new int[nLabels * nLabels];
	smoothCostArray[0] = 0 ;
	smoothCostArray[1] = 1 ;
	smoothCostArray[2] = 1 ;
	smoothCostArray[3] = 0 ;
   // create subsampled working set 
   // WriteSubsampleDataSet("flower2.txt", width, height, nLabels, 2, dataCostArray, smoothCostArray, hCue, vCue); 
   // WriteSubsampleDataSet("flower3.txt", width, height, nLabels, 3, dataCostArray, smoothCostArray, hCue, vCue); 
   // WriteSubsampleDataSet("flower5.txt", width, height, nLabels, 5, dataCostArray, smoothCostArray, hCue, vCue); 
}

