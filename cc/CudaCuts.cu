#include "hip/hip_runtime.h"
/********************************************************************************************
* Implementing Graph Cuts on CUDA using algorithm given in CVGPU '08                       ** 
* paper "CUDA Cuts: Fast Graph Cuts on GPUs"                                               **  
* Copyright (c) 2008 International Institute of Information Technology.                    **  
* All rights reserved.                                                                     **
* Created By Vibhav Vineet.                                                                ** 
********************************************************************************************/


#ifndef _CUDACUTS_CU_
#define _CUDACUTS_CU_

#include "CudaCuts.h"

/********************************************************************
 * cudaCutsInit(width, height, numOfLabels) function sets the      **
 * width, height and numOfLabels of grid. It also initializes the  **
 * block size  on the device and finds the total number of blocks  **
 * running in parallel on the device. It calls checkDevice         **
 * function which checks whether CUDA compatible device is present **
 * on the system or not. It allocates the memory on the host and   **
 * the device for the arrays which are required through the        **
 * function call h_mem_init and segment_init respectively. This    **
 * function returns 0 on success or -1 on failure if there is no   **
 * * * CUDA compatible device is present on the system             **
 * *****************************************************************/

int cudaCutsInit(int widthGrid, int heightGrid, int labels)
{
	deviceCount = checkDevice();
//	printf("No. of devices %d\n",deviceCount);
	if( deviceCount < 1 )
		return -1;
	int cuda_device = 0;
	hipSetDevice( cuda_device );
	hipDeviceProp_t device_properties;
	CUDA_SAFE_CALL( hipGetDeviceProperties(&device_properties, cuda_device) );
	int deviceVersion = device_properties.major * 10 + device_properties.minor; 
	if (deviceVersion == 10) 
		deviceCheck = 2; 
	else if (deviceVersion > 10) 
		deviceCheck = 1;
	else 
		deviceCheck = 0;
	gRealSizeX =  widthGrid; 
	gRealSizeY = heightGrid; 
	num_Labels = labels;
	blocks_x = 1;
	blocks_y = 1;
	num_of_blocks = 1; 
	num_of_threads_per_block = 256; 
	threads_x = 32;
	threads_y = 8;
	gSizeX = threads_x * ((int)ceil((float)gRealSizeX/ (float)threads_x));
	gSizeY = threads_y * ( (int)ceil((float) gRealSizeY / (float) threads_y ));
	gRealSizeTotal = gRealSizeX * gRealSizeY;
	gSizeTotal = gSizeX * gSizeY;
	size_int = sizeof(int) * gSizeTotal;
	blocks_x = (int)((ceil)((float)gSizeX/(float)threads_x));
	blocks_y = (int)((ceil)((float)gSizeY/(float)threads_y));
	num_of_blocks = (int)((ceil)((float)gSizeTotal/(float)num_of_threads_per_block));
	h_mem_init();
	d_mem_init();
	cueValues = 0;
	return deviceCheck;
}

int checkDevice() {
	int deviceCount ;
	hipGetDeviceCount( &deviceCount );
	if(deviceCount == 0) {
		return -1;
	}
	return deviceCount ;
}

void h_mem_init() {
	h_reset_mem = (int* )malloc(sizeof(int) * gSizeTotal);
	h_graph_height = (int* )malloc(size_int);
	pixelLabel = (int*)malloc( size_int);
	h_pixel_mask = (bool*)malloc( sizeof(bool) * gSizeTotal);
	for( int i = 0; i < gSizeTotal; i++ ) {
		pixelLabel[i] = 0 ;
		h_graph_height[i] = 0 ;
	}
	for( int i = 0; i < gSizeTotal; i++ ) {
		h_reset_mem[i] = 0 ;
	}
}

void d_mem_init() {
	CUDA_SAFE_CALL( hipMalloc((void**)&d_left_weight, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&d_right_weight, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&d_down_weight, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&d_up_weight, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&d_push_reser, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&d_sink_weight, sizeof(int) * gSizeTotal ) );

	CUDA_SAFE_CALL( hipMalloc((void**)&d_pull_left, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&d_pull_right, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&d_pull_down, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&d_pull_up, sizeof(int) * gSizeTotal ) );

	CUDA_SAFE_CALL( hipMalloc((void**)&s_left_weight, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&s_right_weight, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&s_down_weight, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&s_up_weight, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&s_push_reser, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&s_sink_weight, sizeof(int) * gSizeTotal ) );

	CUDA_SAFE_CALL( hipMalloc((void**)&d_stochastic, sizeof(int) * num_of_blocks ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&d_stochastic_pixel, sizeof(int) * gSizeTotal ) );

	//CUDA_SAFE_CALL( hipMalloc((void**)&d_sink_weight, sizeof(int) * graph_size1 ) );
	//CUDA_SAFE_CALL( hipMalloc((void**)&d_sink_weight, sizeof(int) * graph_size1 ) );
	//CUDA_SAFE_CALL( hipMalloc((void**)&d_sink_weight, sizeof(int) * graph_size1 ) );
	//CUDA_SAFE_CALL( hipMalloc((void**)&d_sink_weight, sizeof(int) * graph_size1 ) );
	
	CUDA_SAFE_CALL( hipMalloc((void**)&d_graph_heightr, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&d_graph_heightw, sizeof(int) * gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc((void**)&d_relabel_mask, sizeof(int) * gSizeTotal ) );

	CUDA_SAFE_CALL( hipMalloc( ( void**)&d_pixel_mask, sizeof(bool)*gSizeTotal ) );
	CUDA_SAFE_CALL( hipMalloc( ( void**)&d_over, sizeof(bool)*1 ) );
	CUDA_SAFE_CALL(hipMalloc((void**)&d_counter,sizeof(int)));

	CUDA_SAFE_CALL( hipMalloc( ( void **)&dPixelLabel, sizeof(int) * gSizeX * gSizeY ));
	CUDA_SAFE_CALL( hipMemcpy( d_left_weight, h_reset_mem, sizeof( int ) * gSizeTotal , hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( d_right_weight, h_reset_mem, sizeof( int ) * gSizeTotal , hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( d_down_weight, h_reset_mem, sizeof( int ) * gSizeTotal , hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( d_up_weight, h_reset_mem, sizeof( int ) * gSizeTotal , hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( d_push_reser, h_reset_mem, sizeof( int ) * gSizeTotal , hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( d_sink_weight, h_reset_mem, sizeof( int ) * gSizeTotal , hipMemcpyHostToDevice));

	CUDA_SAFE_CALL( hipMemcpy( d_pull_left, h_reset_mem, sizeof( int ) * gSizeTotal , hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( d_pull_right, h_reset_mem, sizeof( int ) * gSizeTotal , hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( d_pull_down, h_reset_mem, sizeof( int ) * gSizeTotal , hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( d_pull_up, h_reset_mem, sizeof( int ) * gSizeTotal , hipMemcpyHostToDevice));
	
	h_relabel_mask = (int*)malloc(sizeof(int)*gSizeX*gSizeY);
	h_stochastic = (int *)malloc(sizeof(int) * num_of_blocks);
	h_stochastic_pixel = (int *)malloc(sizeof(int) * gSizeTotal);
	for(int i = 0; i < gSizeTotal; i++ )
		h_relabel_mask[i] = 1;
	CUDA_SAFE_CALL( hipMemcpy( d_relabel_mask, h_relabel_mask, sizeof(int) * gSizeTotal, hipMemcpyHostToDevice));
	int *dpixlab = (int*)malloc(sizeof(int)*gSizeX*gSizeY);
	for( int i = 0 ; i < gSizeX * gSizeY ; i++ ) {
		dpixlab[i] = 0 ;
		h_stochastic_pixel[i] = 1 ; 
	}
	for(int i = 0 ; i < num_of_blocks ; i++ ) {
		h_stochastic[i] = 1 ; 
	}
	CUDA_SAFE_CALL(hipMemcpy(d_stochastic, h_stochastic, sizeof(int) * num_of_blocks , hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_stochastic_pixel, h_stochastic_pixel, sizeof(int)* gSizeTotal, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( dPixelLabel, dpixlab, sizeof(int) * gSizeX * gSizeY , hipMemcpyHostToDevice));
}

int cudaCutsSetupDataTerm( int *dataTerm ) {
	if( deviceCheck < 1 )
		return -1 ; 
	datacost  =  (int*)malloc(sizeof(int) * gRealSizeX *gRealSizeY * num_Labels );
	CUDA_SAFE_CALL( hipMalloc( ( void **)&dDataTerm, sizeof(int) * gRealSizeX * gRealSizeY * num_Labels ));
	CUDA_SAFE_CALL( hipMemcpy( dDataTerm, dataTerm, sizeof(int) * gRealSizeX * gRealSizeY * num_Labels , hipMemcpyHostToDevice   ) ) ;
	for( int i = 0 ; i < gRealSizeX * gRealSizeY * num_Labels ; i++) {
		datacost[i] = dataTerm[i] ; 
	}
	return 0 ; 
}

int cudaCutsSetupSmoothTerm( int *smoothTerm ) {
	if( deviceCheck < 1 )
		return -1 ; 
	smoothnesscost  =  (int*)malloc(sizeof(int) * num_Labels * num_Labels );
	CUDA_SAFE_CALL( hipMalloc( ( void **)&dSmoothTerm, sizeof(int) * num_Labels * num_Labels ));
	CUDA_SAFE_CALL( hipMemcpy( dSmoothTerm, smoothTerm, sizeof(int) * num_Labels * num_Labels, hipMemcpyHostToDevice));
	for( int i = 0 ; i < num_Labels * num_Labels ; i++) {
		smoothnesscost[i] = smoothTerm[i] ; 
	}
	return 0 ; 
}

int cudaCutsSetupHCue( int *hCue )
{

	if( deviceCheck < 1 )
		return -1 ; 

	hcue  =  (int*)malloc(sizeof(int) * gRealSizeX * gRealSizeY );

	CUDA_SAFE_CALL( hipMalloc( ( void **)&dHcue, sizeof(int) * gRealSizeX * gRealSizeY ));

	CUDA_SAFE_CALL( hipMemcpy( dHcue, hCue, sizeof(int) * gRealSizeX * gRealSizeY , hipMemcpyHostToDevice   ) ) ;

	for( int i = 0 ; i < gRealSizeX * gRealSizeY ; i++)
	{
		hcue[i] = hCue[i] ; 
	}

	cueValues = 1 ; 

	return 0 ; 
}

int cudaCutsSetupVCue( int *vCue )
{
	if( deviceCheck < 1 )
		return -1 ; 

	vcue  =  (int*)malloc(sizeof(int) * gSizeX * gSizeY );

	CUDA_SAFE_CALL( hipMalloc( ( void **)&dVcue, sizeof(int) * gRealSizeX * gRealSizeY ));

	CUDA_SAFE_CALL( hipMemcpy( dVcue, vCue, sizeof(int) * gRealSizeX * gRealSizeY , hipMemcpyHostToDevice   ) ) ;

	for( int i = 0 ; i < gRealSizeX * gRealSizeY ; i++)
	{
		vcue[i] = vCue[i] ; 
	}

	return 0 ; 
}


int cudaCutsSetupGraph() {
	if( deviceCheck < 1 )
		return -1 ; 
	int alpha_label = 1 ;
	for( int i = 0 ; i < gSizeTotal ; i++ ) {
		h_reset_mem[i] = 0 ;
		h_graph_height[i] = 0 ;
	}
	int blockEdge = (int)((ceil)((float)( gRealSizeX * gRealSizeY )/ ( float ) 256 ));
	dim3 block_weight(256, 1, 1);
	dim3 grid_weight(blockEdge,1,1);
	if( cueValues == 1 ) {
		CudaWeightCue<<< grid_weight, block_weight >>>( alpha_label, d_left_weight, d_right_weight, d_down_weight,
						d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm, 
						dSmoothTerm, dHcue, dVcue, gRealSizeX, gRealSizeY, 2) ;
	} else {
		CudaWeight<<< grid_weight , block_weight >>>( alpha_label, d_left_weight, d_right_weight, d_down_weight, 
						d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm, 
						dSmoothTerm, gRealSizeX, gRealSizeY, 2) ;
	}
	int *temp_left_weight, *temp_right_weight, *temp_down_weight, *temp_up_weight, *temp_source_weight, *temp_terminal_weight ;

	CUDA_SAFE_CALL( hipMalloc( ( void **)&temp_left_weight, sizeof( int ) * gSizeTotal ) ) ;
	CUDA_SAFE_CALL( hipMalloc( ( void **)&temp_right_weight, sizeof( int ) * gSizeTotal ) ) ;
	CUDA_SAFE_CALL( hipMalloc( ( void **)&temp_down_weight, sizeof( int ) * gSizeTotal ) ) ;
	CUDA_SAFE_CALL( hipMalloc( ( void **)&temp_up_weight, sizeof( int ) * gSizeTotal ) ) ;
	CUDA_SAFE_CALL( hipMalloc( ( void **)&temp_source_weight, sizeof( int ) * gSizeTotal ) ) ;
	CUDA_SAFE_CALL( hipMalloc( ( void **)&temp_terminal_weight, sizeof( int ) * gSizeTotal ) ) ;

	int blockEdge1 = (int)((ceil)((float)( gSizeX * gSizeY )/ ( float ) 256 ));
	dim3 block_weight1(256, 1, 1);
	dim3 grid_weight1(blockEdge1,1,1);
	adjustedgeweight<<<grid_weight1, block_weight1>>>(d_left_weight, d_right_weight, d_down_weight, d_up_weight, 
		d_push_reser,d_sink_weight, temp_left_weight, temp_right_weight, temp_down_weight, temp_up_weight,
		temp_source_weight, temp_terminal_weight, gRealSizeX,  gRealSizeY,  gRealSizeTotal,  gSizeX, 
		gSizeY, gSizeTotal) ;
	copyedgeweight<<<grid_weight1, block_weight1>>>(d_left_weight, d_right_weight, d_down_weight, d_up_weight, 
		d_push_reser, d_sink_weight, temp_left_weight, temp_right_weight, temp_down_weight,
		temp_up_weight,temp_source_weight, temp_terminal_weight, d_relabel_mask,
		d_graph_heightr, d_graph_heightw, gRealSizeX,  gRealSizeY,  gRealSizeTotal,  gSizeX, gSizeY, gSizeTotal);
	return 0 ; 
}

int cudaCutsAtomicOptimize(  ) {
	if( deviceCheck < 1 ) {
		return -1 ; 
	}
	cudaCutsAtomic();
	bfsLabeling( );
	return 0 ; 
}

int cudaCutsStochasticOptimize() {
	if( deviceCheck < 1 ) {
		return -1 ; 
	}
	cudaCutsStochastic();
	bfsLabeling();
	return 0 ; 
}

void cudaCutsAtomic() {
    printf("Grid dimensions (non-persistent):\n");
    printf("\tthreads_x=%d, threads_y=%d, threads_per_block=%d\n", threads_x, threads_y, threads_x*threads_y);
    printf("\tblocks_x=%d, blocks_y=%d, blocks_per_grid=%d\n", blocks_x, blocks_y, blocks_x*blocks_y);

    // Calculate grid dimensions for persistent threads
    const int max_blocks = 6*16;
    int blocks_xp = blocks_x;
    int blocks_yp = blocks_y;
    if(max_blocks < blocks_x*blocks_y) {
        // Scale worksize tile proportionally
        float blocks_xp_f = sqrt((float) max_blocks * (float) blocks_x / (float) blocks_y);
        blocks_xp = (int) blocks_xp_f;
        blocks_yp = (int)(blocks_xp_f * (float)blocks_y / (float)blocks_x);
        if((blocks_xp+1)*blocks_yp <= max_blocks || blocks_xp*(blocks_yp+1) <= max_blocks) {
            // We can safely increase dimension and still fit
            if( (blocks_xp+1)*blocks_yp > blocks_xp*(blocks_yp+1) )
                blocks_xp++;
            else
                blocks_yp++;
        }
    }
    printf("Grid dimensions (persistent):\n");
    printf("\tthreads_x=%d, threads_y=%d, threads_per_block=%d\n", threads_x, threads_y, threads_x*threads_y);
    printf("\tblocks_x=%d, blocks_y=%d, blocks_per_grid=%d\n", blocks_xp, blocks_yp, blocks_xp*blocks_yp);
    printf("\n");

	dim3 block_push(threads_x, threads_y, 1);
	dim3 grid_push(blocks_x, blocks_y, 1);
	dim3 block_push_p(threads_x, threads_y, 1);
	dim3 grid_push_p(blocks_xp, blocks_yp, 1);
	dim3 d_block(num_of_threads_per_block,1,1);
	dim3 d_grid(num_of_blocks,1,1);
	bool finish = true ;
	counter = num_of_blocks ;
	int numThreadsEnd = 256, numBlocksEnd = 1 ;
	if( numThreadsEnd > counter) {
		numBlocksEnd = 1 ;
		numThreadsEnd = counter ;
	}
	else {
		numBlocksEnd = (int)ceil(counter/(double)numThreadsEnd);
	}
	dim3 End_block(numThreadsEnd,1,1);
	dim3 End_grid(numBlocksEnd,1,1);
	int *d_counter;
	bool *d_finish; 
	for(int i = 0 ; i < num_of_blocks ; i++ ) {
		h_stochastic[i] = 0 ; 
	}

	CUDA_SAFE_CALL( hipMalloc((void**)&d_counter, sizeof(int)));
	CUDA_SAFE_CALL( hipMalloc((void**)&d_finish, sizeof(bool)));
	CUDA_SAFE_CALL( hipMemcpy( d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));
	counter = 0;
	int *d_relabel;
	CUDA_SAFE_CALL( hipMalloc((void**)&d_relabel,sizeof(int) ));
	int h_relabel = 0;
	int block_num = gSizeX / 32;
	int *d_block_num;
	CUDA_SAFE_CALL( hipMalloc((void**)&d_block_num, sizeof(int)));
	CUDA_SAFE_CALL( hipMemcpy( d_block_num, &block_num, sizeof(int), hipMemcpyHostToDevice));
	int h_count_blocks = num_of_blocks ; 
	int *d_count_blocks;
	CUDA_SAFE_CALL( hipMalloc((void**)&d_count_blocks, sizeof(int)));
	CUDA_SAFE_CALL( hipMemcpy( d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));
	h_count_blocks = 0;
	CUDA_SAFE_CALL( hipMemcpy(d_relabel, &h_relabel, sizeof(int), hipMemcpyHostToDevice));
	int sum_at_start = cudaCutsTotalWeightSum(d_push_reser, d_sink_weight);
	printf("SUM AT START = %d\n", sum_at_start);
	counter = 1 ; 
	kernel_start_sink<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight,
			d_sink_weight, d_push_reser,
			d_relabel_mask,d_graph_heightr, gRealSizeTotal,gRealSizeX,gRealSizeY,
			gSizeTotal, gSizeX , gSizeY,d_relabel, d_stochastic, d_counter, d_finish );
	kernel_relabel<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight,
	                        d_sink_weight, d_push_reser,
	                        d_relabel_mask,d_graph_heightr,gRealSizeTotal,gRealSizeX,gRealSizeY,
	                        gSizeTotal, gSizeX , gSizeY);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
//	unsigned int timer = 0;
//	CUT_SAFE_CALL(cutCreateTimer(&timer));
//	CUT_SAFE_CALL(cutStartTimer(timer));
	do {
        kernel_push<<<grid_push_p,block_push_p>>>(9, blocks_x, blocks_y,
                d_left_weight,d_right_weight, d_down_weight, d_up_weight,
                d_pull_left, d_pull_right, d_pull_down, d_pull_up,
                d_sink_weight, d_push_reser,
                d_relabel_mask,d_graph_heightr,gRealSizeTotal,gRealSizeX,gRealSizeY,
                gSizeTotal, gSizeX , gSizeY );
        kernel_pull_end<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight,
                d_pull_left, d_pull_right, d_pull_down, d_pull_up,
                d_push_reser,
                gRealSizeTotal,gRealSizeX,gRealSizeY,gSizeTotal,gSizeX,gSizeY );
        // Check finish criteria
        finish = true ;
        CUDA_SAFE_CALL( hipMemcpy( d_finish, &finish, sizeof(bool), hipMemcpyHostToDevice));
        kernel_push_stochastic1<<<grid_push,block_push>>>(d_push_reser, s_push_reser,  d_count_blocks, d_finish, d_block_num, gSizeX);
        CUDA_SAFE_CALL( hipMemcpy( &finish, d_finish, sizeof(bool), hipMemcpyDeviceToHost));
        // Run 1 iteration (k=1)
        kernel_push<<<grid_push_p,block_push_p>>>(1, blocks_x, blocks_y,
                        d_left_weight,d_right_weight, d_down_weight, d_up_weight,
                        d_pull_left, d_pull_right, d_pull_down, d_pull_up,
                        d_sink_weight, d_push_reser,
                        d_relabel_mask,d_graph_heightr,gRealSizeTotal,gRealSizeX,gRealSizeY,
                        gSizeTotal, gSizeX , gSizeY );
        kernel_pull_end<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight,
                d_pull_left, d_pull_right, d_pull_down, d_pull_up,
                d_push_reser,
                gRealSizeTotal,gRealSizeX,gRealSizeY,gSizeTotal,gSizeX,gSizeY );
        CUDA_SAFE_CALL(hipMemset(d_stochastic, 0, sizeof(int)*num_of_blocks));
        h_count_blocks = 0 ;
        CUDA_SAFE_CALL( hipMemcpy( d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));
        kernel_push_stochastic2<<<grid_push,block_push>>>(d_push_reser, s_push_reser, d_stochastic, d_block_num, gSizeX);
        kernel_End<<<End_grid, End_block>>>(d_stochastic, d_count_blocks, d_counter);
	counter++;
//	printf("counter=%d\n", counter);
	}
	while(finish);
    int sum_at_end = cudaCutsTotalWeightSum(d_push_reser, d_sink_weight);
    printf("SUM AT END = %d\n", sum_at_end);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
//	CUT_SAFE_CALL(cutStopTimer(timer));
//	printf("TT Cuts :: %f\n",cutGetTimerValue(timer));
//	CUT_SAFE_CALL(cutDeleteTimer(timer));

}

int cudaCutsTotalWeightSum(int* d_push_reser, int* d_sink_weight) {
    int* h_temp_push_reser = (int*)malloc(sizeof(int) * gSizeTotal);
    int* h_temp_sink_weight = (int*)malloc(sizeof(int) * gSizeTotal);
    CUDA_SAFE_CALL( hipMemcpy( h_temp_push_reser, d_push_reser, sizeof(int) * gSizeTotal, hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL( hipMemcpy( h_temp_sink_weight, d_sink_weight, sizeof(int) * gSizeTotal, hipMemcpyDeviceToHost));
    int sum = 0;
    for(int i=0; i<gSizeTotal; i++)
        sum += h_temp_push_reser[i];
    for(int i=0; i<gSizeTotal; i++)
        sum -= h_temp_sink_weight[i];
    free(h_temp_push_reser);
    free(h_temp_sink_weight);
    return sum;
}

void cudaCutsStochastic() {
	dim3 block_push(threads_x, threads_y, 1);
	dim3 grid_push(blocks_x, blocks_y, 1);
	dim3 d_block(num_of_threads_per_block,1,1);
	dim3 d_grid(num_of_blocks,1,1);
	bool finish = true ;
	counter = num_of_blocks ;
	int numThreadsEnd = 256, numBlocksEnd = 1 ; 
	if( numThreadsEnd > counter) {
		numBlocksEnd = 1 ; 
		numThreadsEnd = counter ; 
	}
	else {
		numBlocksEnd = (int)ceil(counter/(double)numThreadsEnd);
	}
	dim3 End_block(numThreadsEnd,1,1);
	dim3 End_grid(numBlocksEnd,1,1);
	bool *d_finish ; 
	for(int i = 0 ; i < num_of_blocks ; i++ ) {
		h_stochastic[i] = 0 ; 
	}
	CUDA_SAFE_CALL( hipMalloc((void**)&d_counter, sizeof(int)));
	CUDA_SAFE_CALL( hipMalloc((void**)&d_finish, sizeof(bool)));
	CUDA_SAFE_CALL( hipMemcpy( d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));
	counter = 0;
	int *d_relabel;
	CUDA_SAFE_CALL( hipMalloc((void**)&d_relabel,sizeof(int) ));
	int h_relabel = 0;
	int block_num = gSizeX / 32;
	int *d_block_num;
	CUDA_SAFE_CALL( hipMalloc((void**)&d_block_num, sizeof(int)));
	CUDA_SAFE_CALL( hipMemcpy( d_block_num, &block_num, sizeof(int), hipMemcpyHostToDevice));
	int h_count_blocks = num_of_blocks ; 
	int *d_count_blocks;
	CUDA_SAFE_CALL( hipMalloc((void**)&d_count_blocks, sizeof(int)));
	CUDA_SAFE_CALL( hipMemcpy( d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));
	h_count_blocks = 0 ;
	CUDA_SAFE_CALL( hipMemcpy(d_relabel, &h_relabel, sizeof(int), hipMemcpyHostToDevice));
	counter = 1 ; 
	kernel_push1_start_stochastic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight, 
			d_sink_weight, d_push_reser,
			d_relabel_mask,d_graph_heightr,d_graph_heightw, gRealSizeTotal,gRealSizeX,gRealSizeY, 
			gSizeTotal, gSizeX , gSizeY,d_relabel, d_stochastic, d_counter, d_finish );
	CUDA_SAFE_CALL(hipDeviceSynchronize());
//	unsigned int timer = 0;
//	CUT_SAFE_CALL(cutCreateTimer(&timer));
//	CUT_SAFE_CALL(cutStartTimer(timer));
	do {
		if(counter%10 == 0) {
			finish = true ; 
			CUDA_SAFE_CALL( hipMemcpy( d_finish, &finish, sizeof(bool), hipMemcpyHostToDevice));
			kernel_push_stochastic1<<<grid_push,block_push>>>(d_push_reser, s_push_reser,  d_count_blocks, d_finish, d_block_num, gSizeX);
			CUDA_SAFE_CALL( hipMemcpy( &finish, d_finish, sizeof(bool), hipMemcpyDeviceToHost));
		}
		if(counter%11 == 0 ) {
			CUDA_SAFE_CALL(hipMemset(d_stochastic, 0, sizeof(int)*num_of_blocks));
			h_count_blocks = 0 ; 
			CUDA_SAFE_CALL( hipMemcpy( d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));
			kernel_push_stochastic2<<<grid_push,block_push>>>(d_push_reser, s_push_reser, d_stochastic, d_block_num, gSizeX);
			
			kernel_End<<<End_grid, End_block>>>(d_stochastic, d_count_blocks, d_counter);
		}
		if( counter % 2 == 0 ) {
			kernel_push1_stochastic<<<grid_push,block_push>>>(d_left_weight, d_right_weight, d_down_weight, 
					d_up_weight, d_sink_weight, d_push_reser,
					d_relabel_mask, d_graph_heightr, d_graph_heightw, gRealSizeTotal, gRealSizeX,
					gRealSizeY, gSizeTotal, gSizeX , gSizeY, d_stochastic, d_block_num);
			kernel_relabel_stochastic<<<grid_push,block_push>>>(d_left_weight, d_right_weight, d_down_weight, 
					d_up_weight, d_sink_weight, d_push_reser,
					d_relabel_mask,d_graph_heightr,d_graph_heightw, gRealSizeTotal, gRealSizeX, 
					gRealSizeY, gSizeTotal, gSizeX , gSizeY, d_stochastic,d_block_num );
		}
		else {
			kernel_push1_stochastic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, 
					d_up_weight, d_sink_weight, d_push_reser,
					d_relabel_mask,d_graph_heightw,d_graph_heightr, gRealSizeTotal, gRealSizeX, 
					gRealSizeY, gSizeTotal, gSizeX , gSizeY, d_stochastic, d_block_num);
			
			kernel_relabel_stochastic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, 
					d_up_weight, d_sink_weight, d_push_reser,
					d_relabel_mask,d_graph_heightw,d_graph_heightr, gRealSizeTotal,gRealSizeX,
					gRealSizeY, gSizeTotal, gSizeX , gSizeY, d_stochastic, d_block_num );
		}
		counter++ ;
	}
	while(finish);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
//	CUT_SAFE_CALL(cutStopTimer(timer));
//	printf("TT Cuts :: %f\n",cutGetTimerValue(timer));
//	CUT_SAFE_CALL(cutDeleteTimer(timer));
}

void bfsLabeling() {
	dim3 block_push(threads_x, threads_y, 1);
	dim3 grid_push(blocks_x, blocks_y, 1);
	dim3 d_block(num_of_threads_per_block,1,1);
	dim3 d_grid(num_of_blocks,1,1);
	CUDA_SAFE_CALL( hipMemcpy( d_graph_heightr, h_graph_height, size_int, hipMemcpyHostToDevice));
	for(int i = 0 ; i < gRealSizeTotal ; i++ )
		h_pixel_mask[i]=true;
	CUDA_SAFE_CALL( hipMemcpy( d_pixel_mask, h_pixel_mask, sizeof(bool) * gSizeTotal, hipMemcpyHostToDevice));
	kernel_bfs_t<<<d_grid,d_block,0>>>(d_push_reser,d_sink_weight,d_graph_heightr,d_pixel_mask,gRealSizeTotal,gRealSizeX,gRealSizeY, gSizeTotal, gSizeX, gSizeY);
	counter=1;

	CUDA_SAFE_CALL( hipMemcpy( d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));
	do {
		h_over=false;
		CUDA_SAFE_CALL( hipMemcpy( d_over, &h_over, sizeof(bool), hipMemcpyHostToDevice) );
		kernel_bfs<<< d_grid,d_block, 0 >>>(d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_graph_heightr, d_pixel_mask,
				gRealSizeTotal, gRealSizeX, gRealSizeY, gSizeTotal, gSizeX, gSizeY, d_over,d_counter);
		CUT_CHECK_ERROR("Kernel execution failed");
		CUDA_SAFE_CALL( hipMemcpy( &h_over, d_over, sizeof(bool), hipMemcpyDeviceToHost) );
		counter++;
		CUDA_SAFE_CALL(hipMemcpy(d_counter,&counter,sizeof(int),hipMemcpyHostToDevice));
	}
	while(h_over);
	CUDA_SAFE_CALL(hipMemcpy(h_graph_height,d_graph_heightr,size_int,hipMemcpyDeviceToHost));
}


int cudaCutsGetResult( ) {
	if( deviceCheck < 1 )
		return -1 ; 
	int alpha = 1 ;
	for(int i = 0 ; i < gSizeTotal ; i++ ) {
		int row_here = i / gSizeX, col_here = i % gSizeX ;
		if(h_graph_height[i]>0 && row_here < gRealSizeY && row_here > 0 && col_here < gRealSizeX && col_here > 0 ) {
			pixelLabel[i]=alpha;
		}
	}
	return 0;
}

int cudaCutsGetEnergy() {
	return data_energy() + smooth_energy() ;
}

int data_energy() {
	int eng=0;
	for(int i = 0 ; i < gRealSizeY ; i ++) {
		for(int j = 0 ; j < gRealSizeX ; j++) {
			eng += datacost(i*gRealSizeX+j, pixelLabel[i*gSizeX+j]);
		}
	}
	printf("DATA ENERGY: %d\n",eng);
	return(eng);
}

int smooth_energy() {
	int eng = 0;
	int x,y;
	for ( y = 0; y < gRealSizeY; y++ )
		for ( x = 1; x < gRealSizeX; x++ ) {
			if( cueValues == 1 )
				eng = eng + smoothnesscost(pixelLabel[y*gSizeX+x],pixelLabel[y*gSizeX+x-1])*hcue[y*gRealSizeX+x-1];
			else
				eng = eng + smoothnesscost(pixelLabel[y*gSizeX+x],pixelLabel[y*gSizeX+x-1]);
		}
	for ( y = 1; y < gSizeY; y++ )
		for ( x = 0; x < gSizeX; x++ ) {
			if(cueValues == 1)
				eng = eng + smoothnesscost(pixelLabel[y*gSizeX+x],pixelLabel[y*gSizeX+x-gSizeX])*vcue[y*gRealSizeX+x-gRealSizeX];
			else
				eng = eng + smoothnesscost(pixelLabel[y*gSizeX+x],pixelLabel[y*gSizeX+x-gSizeX]);
		}
	printf("SMOOTHNESS ENERGY: %d\n",eng);
	return(eng);
}


void cudaCutsFreeMem() {
	free(h_reset_mem);
	free(h_graph_height);
	CUDA_SAFE_CALL(hipFree(d_left_weight));
	CUDA_SAFE_CALL(hipFree(d_right_weight));
	CUDA_SAFE_CALL(hipFree(d_down_weight));
	CUDA_SAFE_CALL(hipFree(d_up_weight));
	CUDA_SAFE_CALL(hipFree(d_sink_weight));
	CUDA_SAFE_CALL(hipFree(d_push_reser));
	CUDA_SAFE_CALL(hipFree(d_pull_left));
	CUDA_SAFE_CALL(hipFree(d_pull_right));
	CUDA_SAFE_CALL(hipFree(d_pull_down));
	CUDA_SAFE_CALL(hipFree(d_pull_up));
	CUDA_SAFE_CALL(hipFree(d_graph_heightr));
	CUDA_SAFE_CALL(hipFree(d_graph_heightw));
}

#endif

