#include "hip/hip_runtime.h"
#include "common.h"

/*
#ifdef LOCKFREE
__device__ inline void lockfree_barrier(volatile uint *counter, int *partial_sums, int *predicate, int current_value, uint bx, uint tx) {
	if(tx==0) {
//		if(bx>0) while(counter[bx-1] != 1) {}
		if(bx>0) while(counter[bx] != 1) {}
	}
	__syncthreads();

	if(tx==0) {
		predicate[0] = partial_sums[bx];
		partial_sums[bx+1] = predicate[0] + current_value;
	}
	__threadfence();

	if(tx==0) {
//		counter[bx] = 1;
		counter[bx+1] = 1;
	}
//	__syncthreads();
}
#endif
//*/
///*
#ifdef LOCKFREE
__device__ inline void lockfree_barrier(volatile unsigned *counter, int* partial_sums, int *predicate, int current_value, uint bx, uint tx) {
	if(tx==0) {

//		while(1) {
//			unsigned flag = counter[bx];
//			if(bx==0 || flag!=0) break;
//		} // end while


		if(bx>0) while(counter[bx] != 1) {}
		predicate[0] = partial_sums[bx];
		partial_sums[bx+1] = predicate[0] + current_value;
	}
	__threadfence();
//	__syncthreads();
	if(tx==0) {
		counter[bx+1] = 1;
	}
}
#endif
//*/

#ifdef ATOMIC
__device__ inline void atomic_barrier(unsigned *counter, int* partial_sums, int *predicate, int current_value, uint bx, uint tx) {
	if(tx==0) {
		unsigned flag = 0;
		while(1) {
			flag = atomicAdd(counter+bx, flag);
			if(bx==0 || flag!=0) {
				break;
			}
		} // end while
		predicate[0] = partial_sums[bx];
		partial_sums[bx+1] = predicate[0] + current_value;
	}
	__threadfence();
	if(tx==0) {
		atomicAdd(counter+bx+1, 1);
	}
}
#endif


#ifdef FREE
__device__ inline void free_barrier(int* partial_sums, int *predicate, int current_value, uint bx, uint tx) {

	__syncthreads();
	if(tx==0) {
//		predicate[0] += predicate[0];

		predicate[0] = partial_sums[bx];
		partial_sums[bx+1] = predicate[0] + current_value;
	}

	__syncthreads();
}
#endif

#ifdef UNSAFE
__device__ inline void unsafe_barrier(volatile int* partial_sums, int *predicate, int current_value, uint bx, uint tx) {
	__syncthreads();
	if(tx==0) {
		int cc=0;
		while(1) {
			cc = partial_sums[bx];
			if(bx==0 || cc!=0) {
				break;
			}
		}
//		printf("bx=%d, cc=%d, current_value=%d\n", bx, cc, current_value);
		predicate[0] = cc;
		partial_sums[bx+1] = predicate[0] + current_value;
//		printf("partial_sums[%d]=%d\n", bx+1, partial_sums[bx+1]);
	}
	__syncthreads();
}
#endif

#ifdef FORWARD
//double the size of partial_sums
__device__ inline void forward_barrier(volatile int* partial_sums, int *predicate, int current_value, uint bx, uint tx) {
	if(tx<2) {
		int cc;
		while(1) {
			cc = partial_sums[2*bx+tx];
			// cc 0: flag   1: value
			if(tx==1)
				predicate[0]=cc;
			
			//check value
			if( bx==0|| predicate[0]!=0) { //assume initialized as 0
				break;
			}
			
			//if value == initialized
			//check flag
			if(tx==0)
				predicate[0]=cc;
			if(predicate[0]==1)
				break;
			//repeat polling
		} // end while	

		//got value
		if(tx==1)
			predicate[0]=cc;
		if(tx==0) {
			//if value is equal to initialized
			if((predicate[0]+current_value)==0) {
				//write flag
				partial_sums[2*bx+2]=1;
			}
			else {
				//write value
				partial_sums[2*bx+3]=predicate[0]+current_value;
			}
		}
	}
}
#endif

#ifdef FORWARD_ATOMIC
//double the size of partial_sums
__device__ inline void forward_atomic_barrier(int* partial_sums, int *predicate, int current_value, uint bx, uint tx) {
	if(tx<2) {
		int cc=0;
		while(1) {
			cc = atomicExch(partial_sums+2*bx+tx,cc);
			// cc 0: flag   1: value
			if(tx==1)
				predicate[0]=cc;
			
			//check value
			if( bx==0|| predicate[0]!=0) { //assume initialized as 0
				break;
			}
			
			//if value == initialized
			//check flag
			if(tx==0)
				predicate[0]=cc;
			if(predicate[0]==1)
				break;
			//repeat polling
		}
		//got value
		if(tx==1)
			predicate[0]=cc;
		if(tx==0) {
			//if value is equal to initialized
			if((predicate[0]+current_value)==0) {
				//write flag
//				partial_sums[2*bx+2]=1;
				atomicAdd(partial_sums+2*bx+2,1);
			}
			else {
				//write value
//				partial_sums[2*bx+3]=predicate[0]+warp_sums[15];
				atomicAdd(partial_sums+2*bx+3,predicate[0]+current_value);
			}
		}
	}	
}
#endif

