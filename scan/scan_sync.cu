#include "hip/hip_runtime.h"
// Scan, IMPACT UIUC
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "barrier.cu"

#define BLOCK_SIZE 256
#define NUM_ELEMENTS 1024*128*32;

#ifdef NAIVE
#ifndef HW_BARRIER
__device__ inline void __wait_for_blocks(volatile uint *flags, uint bid, uint tx) {
	if(tx==0) {
		while(flags[bid] != 1) {}// waiting for dependency
	}
//	__threadfence();
	__syncthreads();
}

__device__ inline void __block_check_in(volatile uint *flags, uint bx, uint tx) {
	if(tx==0) {
		flags[bx] = 1;// this block checks in
	}
//	__syncthreads();
}
#endif
#endif

__global__ void scan_kernel(int *data, unsigned *counter, int *partial_sums) {
	uint tx = threadIdx.x;
	uint bx = blockIdx.x;
	uint bs = blockDim.x;
	// Step 1: Load
	extern __shared__ int sums[];
	const unsigned id = bx*bs + tx;
	sums[tx] = data[id];
	__syncthreads();

	// Step 2: Computation
	for (int i=1; i<=bs/2; i*=2) {
		if(tx<bs/2/i) {
			sums[2*i*tx+2*i-1]+=sums[2*i*tx+i-1];
		}
		__syncthreads();
	}

	// Step 3: Synchronization
	// block[i+1] depends on block[i]
#ifdef LOCKFREE
//	__ib_sync_lockfree(counter, partial_sums, sums);
	lockfree_barrier(counter, partial_sums, sums+bs, sums[bs-1], bx, tx);
#endif
#ifdef ATOMIC
	atomic_barrier(counter, partial_sums, sums+bs, sums[bs-1], bx, tx);
#endif
#ifdef FREE
	free_barrier(partial_sums, sums+bs, sums[bs-1], bx, tx);
#endif
#ifdef UNSAFE
	unsafe_barrier(partial_sums, sums+bs, sums[bs-1], bx, tx);
#endif
#ifdef FORWARD
	forward_barrier(partial_sums, sums+bs, sums[bs-1], bx, tx);
#endif
#ifdef FORWARD_ATOMIC
	forward_atomic_barrier(partial_sums, sums+bs, sums[bs-1], bx, tx);
#endif

#ifdef NAIVE
	if(tx==0)partial_sums[bx+1] = sums[bs-1];
	if(bx>0) {
#ifdef HW_BARRIER
//		__syncthreads();
		__threadfence();
#else
		__wait_for_blocks(counter, bx-1, tx);
#endif
	}
	if(tx==0) {
		sums[bs] = partial_sums[bx];
//		partial_sums[bx+1] = sums[bs] + sums[bs-1];
		partial_sums[bx+1] += sums[bs];
	}
	__threadfence();
#ifndef HW_BARRIER
	__block_check_in(counter, bx, tx);
#endif
#endif //end ifdef NAIVE

	// Step 4: Computation
	for (int i=bs/4; i>=1; i/=2) {
		if(tx<bs/2/i-1) {
			sums[3*i+2*i*tx-1]+=sums[2*i+2*i*tx-1];
		}
		__syncthreads();
	}

	// Step 5: Store
	data[id] = sums[tx] + sums[bs];
}

void check_scan_all_one(int *x, int n) {
	for(int i=0;i<n;i++) {
		if(x[i]!=(i+1)) {
			printf("[ERROR] out[%d]=%d, ref=%d\n", i, x[i], i+1);
			printf("[ERROR] out[%d]=%d, ref=%d\n", i+1, x[i+1], i+2);
			return;
		}
	}
	printf("[BENCH] Result correct\n");
}

int main() {
	int n_elements = NUM_ELEMENTS;
	int blockSize = BLOCK_SIZE;
	int *h_data, *h_partial_sums, *h_result;
	int *d_data, *d_partial_sums;
//	unsigned *d_zero;
	unsigned *d_counter;
	int sz = sizeof(int) *n_elements;
	hipHostMalloc((void **)&h_data, sizeof(int)*n_elements);
	hipHostMalloc((void **)&h_result, sizeof(int)*n_elements);
	for (int i=0; i<n_elements; i++) {
		h_data[i] = 1;
	}
	int gridSize = n_elements/blockSize;
	int shmem_sz = blockSize * sizeof(int)+2;
	int n_partialSums = n_elements/blockSize;
	int partial_sz = n_partialSums*sizeof(int);

	printf("[BENCH] Scan with %d elements, %d partial sums\n", n_elements, n_partialSums);
	printf("[BENCH] gridSize=%d, blockSize=%d, shmem_sz=%d\n", gridSize, blockSize, shmem_sz);
#ifdef LOCKFREE
	printf("[BENCH] Lock Free Barrier\n");
#endif
#ifdef ATOMIC
	printf("[BENCH] Atomic Barrier\n");
#endif
#ifdef FREE
	printf("[BENCH] Free Barrier, result is supposed to be incorrect\n");
#endif
#ifdef UNSAFE
	printf("[BENCH] Unsafe Barrier\n");
#endif
#ifdef FORWARD
	printf("[BENCH] Forward Barrier\n");
#endif
#ifdef FORWARD_ATOMIC
	printf("[BENCH] Forward Atomic Barrier\n");
#endif
#ifdef HW_BARRIER
	printf("[BENCH] Hardware Barrier\n");
#endif
	hipMalloc((void **)&d_data, sz);
	hipMalloc((void **)&d_partial_sums, partial_sz);
	hipMemset(d_partial_sums, 0, partial_sz);
	hipHostMalloc((void **)&h_partial_sums, partial_sz);
	hipMemcpy(d_data, h_data, sz, hipMemcpyHostToDevice);

//	hipMalloc((void **)&d_zero, sizeof(unsigned));
	hipMalloc((void **)&d_counter, n_partialSums*sizeof(unsigned));
//	hipMemset(d_zero, 0, sizeof(unsigned));
	hipMemset(d_counter, 0, n_partialSums*sizeof(unsigned));

	hipDeviceSynchronize();
	scan_kernel<<< gridSize, blockSize, shmem_sz >>>(d_data, d_counter, d_partial_sums);
	hipDeviceSynchronize();

	hipMemcpy(h_result, d_data, sz, hipMemcpyDeviceToHost);
	hipMemcpy(h_partial_sums, d_partial_sums, partial_sz, hipMemcpyDeviceToHost);
//	printf("Test Sum: %d\n", h_partial_sums[n_partialSums-1]);
	check_scan_all_one(h_result,n_elements);
	hipHostFree(h_data);
	hipHostFree(h_result);
	hipHostFree(h_partial_sums);
	hipFree(d_data);
	hipFree(d_partial_sums);
	hipDeviceReset();
	return 0;
}

