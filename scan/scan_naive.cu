#include "hip/hip_runtime.h"
// Scan, IMPACT UIUC
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void scan_local(int *data, int *partial_sums) {
	uint tx = threadIdx.x;
	uint bx = blockIdx.x;
	uint bs = blockDim.x;
	extern __shared__ int sums[];
	const unsigned id = bx*bs + tx;
	sums[tx] = data[id];
	__syncthreads();
	for (int i=1; i<=bs/2; i*=2) {
		if(tx<bs/2/i) {
			sums[2*i*tx+2*i-1]+=sums[2*i*tx+i-1];
		}
		__syncthreads();
	}
	if(tx==0)partial_sums[bx+1] = sums[bs-1];
	data[id] = sums[tx];
}

__global__ void scan_partial(int *partial_sums) {
	uint tx = threadIdx.x;
	uint bx = blockIdx.x;
	uint bs = blockDim.x;
	extern __shared__ int sums[];
	const unsigned id = bx*bs + tx;
	sums[tx] = partial_sums[id];
	__syncthreads();
	for (int i=1; i<=bs/2; i*=2) {
		if(tx<bs/2/i) {
			sums[2*i*tx+2*i-1]+=sums[2*i*tx+i-1];
		}
		__syncthreads();
	}
	partial_sums[id] = sums[tx];
}

__global__ void scan_global(int *data, int *partial_sums) {
	uint tx = threadIdx.x;
	uint bx = blockIdx.x;
	uint bs = blockDim.x;
	extern __shared__ int sums[];
	const unsigned id = bx*bs + tx;
	sums[tx] = data[id];
	if(tx==0) sums[bs] = partial_sums[bx];
	__syncthreads();

	for (int i=bs/4; i>=1; i/=2) {
		if(tx<bs/2/i-1) {
			sums[3*i+2*i*tx-1]+=sums[2*i+2*i*tx-1];
		}
		__syncthreads();
	}
	data[id] = sums[tx] + sums[bs];
}

void check_scan_all_one(int *x, int n) {
	for(int i=0;i<n;i++) {
		if(x[i]!=(i+1)) {
			printf("[ERROR] out[%d]=%d, ref=%d\n", i, x[i], i+1);
			printf("[ERROR] out[%d]=%d, ref=%d\n", i+1, x[i+1], i+2);
			return;
		}
	}
	printf("[BENCH] Result correct\n");
}

int main() {
	int n_elements = NUM_ELEMENTS;
	int blockSize = BLOCK_SIZE;
	int *h_data, *h_partial_sums, *h_result;
	int *d_data, *d_partial_sums;
	unsigned *d_counter;
	int sz = sizeof(int) *n_elements;
	hipHostMalloc((void **)&h_data, sizeof(int)*n_elements);
	hipHostMalloc((void **)&h_result, sizeof(int)*n_elements);
	for (int i=0; i<n_elements; i++) {
		h_data[i] = 1;
	}
	int gridSize = n_elements/blockSize;
	int shmem_sz = blockSize * sizeof(int)+2;
	int n_partialSums = n_elements/blockSize;
	int partial_sz = n_partialSums*sizeof(int);
	printf("[BENCH] Scan with %d elements, %d partial sums\n", n_elements, n_partialSums);
	printf("[BENCH] gridSize=%d, blockSize=%d, shmem_sz=%d\n", gridSize, blockSize, shmem_sz);
	hipMalloc((void **)&d_data, sz);
	hipMalloc((void **)&d_partial_sums, partial_sz);
	hipMemset(d_partial_sums, 0, partial_sz);
	hipHostMalloc((void **)&h_partial_sums, partial_sz);
	hipMemcpy(d_data, h_data, sz, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_counter, n_partialSums*sizeof(unsigned));
	hipMemset(d_counter, 0, n_partialSums*sizeof(unsigned));

	hipDeviceSynchronize();
	scan_local<<< gridSize, blockSize, shmem_sz >>>(d_data, d_partial_sums);
	scan_partial<<< gridSize, blockSize, shmem_sz >>>(d_data, d_partial_sums);
	scan_global<<< gridSize, blockSize, shmem_sz >>>(d_data, d_partial_sums);
	hipDeviceSynchronize();

	hipMemcpy(h_result, d_data, sz, hipMemcpyDeviceToHost);
	hipMemcpy(h_partial_sums, d_partial_sums, partial_sz, hipMemcpyDeviceToHost);
	check_scan_all_one(h_result,n_elements);
	hipHostFree(h_data);
	hipHostFree(h_result);
	hipHostFree(h_partial_sums);
	hipFree(d_data);
	hipFree(d_partial_sums);
	hipDeviceReset();
	return 0;
}

