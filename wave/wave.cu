#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "common.h"
#include "barrier.cu"
//#define DEBUG

#define PI 3.14159
#define IDX(x,y,z) ((z)*LATT_DIM_X*LATT_DIM_Y + (y)*LATT_DIM_X + (x))

#define LATT_DIM_X 48
#define LATT_DIM_Y 48
#define LATT_DIM_Z 48
#define LATT_SIZE (LATT_DIM_X*LATT_DIM_Y*LATT_DIM_Z)

#define CUDA_SAFE_CALL_NO_SYNC(call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }

#define CUDA_SAFE_CALL(call) CUDA_SAFE_CALL_NO_SYNC(call);
#define cutilSafeCall(call) CUDA_SAFE_CALL(call); 

#define CUT_CHECK_ERROR(errorMessage) {                                    \
    hipError_t err = hipGetLastError();                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
  }

void wave_gold(float* input_t0, float* input_t1, float vsq, float coeff[5], int iterations, float* result) {
    float* input = (float*) malloc( sizeof(float)*(LATT_SIZE) );
    float* output = (float*) malloc( sizeof(float)*(LATT_SIZE) );
    memcpy(input, input_t1, sizeof(float)*(LATT_SIZE));
    memcpy(output, input_t0, sizeof(float)*(LATT_SIZE));    // output initially contains t-2

    for(int i=0 ;i<iterations; i++) {
        for(int z=4; z<LATT_DIM_Z-4; z++) {
            for(int y=4; y<LATT_DIM_Y-4; y++) {
                for(int x=4; x<LATT_DIM_X-4; x++) {
                    float current = input[IDX(x,y,z)];
                    float prev = output[IDX(x,y,z)];
                    float temp = 2.0f*current - prev;
                    float div = coeff[0] * current;
                    div += coeff[1]*( input[IDX(x+1,y,z)] + input[IDX(x-1,y,z)] +
                                      input[IDX(x,y+1,z)] + input[IDX(x,y-1,z)] +
                                      input[IDX(x,y,z+1)] + input[IDX(x,y,z-1)] );
                    div += coeff[2]*( input[IDX(x+2,y,z)] + input[IDX(x-2,y,z)] +
                                      input[IDX(x,y+2,z)] + input[IDX(x,y-2,z)] +
                                      input[IDX(x,y,z+2)] + input[IDX(x,y,z-2)] );
                    div += coeff[3]*( input[IDX(x+3,y,z)] + input[IDX(x-3,y,z)] +
                                      input[IDX(x,y+3,z)] + input[IDX(x,y-3,z)] +
                                      input[IDX(x,y,z+3)] + input[IDX(x,y,z-3)] );
                    div += coeff[4]*( input[IDX(x+4,y,z)] + input[IDX(x-4,y,z)] +
                                      input[IDX(x,y+4,z)] + input[IDX(x,y-4,z)] +
                                      input[IDX(x,y,z+4)] + input[IDX(x,y,z-4)] );
                    output[IDX(x,y,z)] = temp + div*vsq;
                }
            }
        }

        // Switch buffers
        float* temp_buff = input;
        input = output;
        output = temp_buff;
    }

    // Copy result
    memcpy(result, input, sizeof(float)*(LATT_SIZE));
    free( input );
    free( output );
}

void print_error(float* real_result, float* gold_result) {
    float error_sum = 0.0f;
    for(int z=4; z<LATT_DIM_Z-4; z++) {
        for(int y=4; y<LATT_DIM_Y-4; y++) {
            for(int x=4; x<LATT_DIM_X-4; x++) {
                error_sum += abs(real_result[IDX(x,y,z)] - gold_result[IDX(x,y,z)]);
            }
        }
    }
    printf("[BENCH] Total error = %f\n", error_sum);
    printf("[BENCH] Average error = %f\n", error_sum/((LATT_DIM_Z-4)*(LATT_DIM_Y-4)*(LATT_DIM_X-4)));
}

void generateInputLattice(float* input_t0, float* input_t1, float vsq) {
    for(int z=0; z<LATT_DIM_Z; z++) {
        for(int y=0; y<LATT_DIM_Y; y++) {
            for(int x=0; x<LATT_DIM_X; x++) {
                int index = IDX(x,y,z);

                // Fit N periods of a sinusoid in each dimension
                const float maxAmp = 5.0f;
                const float N = 2;
                float fx, fy, fz;
                fx = (float) x / (float)LATT_DIM_X * 2.0f * PI * N;
                fy = (float) y / (float)LATT_DIM_Y * 2.0f * PI * N;
                fz = (float) z / (float)LATT_DIM_Z * 2.0f * PI * N;
                float val_t0, val_t1;
                val_t0 = maxAmp*sin( sqrt(fx*fx + fy*fy + fz*fz));
                if(x>=4 and x<(LATT_DIM_X-4) and y>=4 and y<(LATT_DIM_Y-4) and z>=4 and z<(LATT_DIM_Z-4))
                    val_t1 = maxAmp*sin( sqrt(fx*fx + fy*fy + fz*fz) + sqrt(vsq));
                else
                    val_t1 = val_t0;

                input_t0[index] = val_t0;
                input_t1[index] = val_t1;
            }
        }
    }
}

void debugPrintDimension(float* input, int dimToPrint) {
    int xStart, yStart, zStart;
    int xEnd, yEnd, zEnd;
    xStart = (dimToPrint==0) ? 0 : 4;
    yStart = (dimToPrint==1) ? 0 : 4;
    zStart = (dimToPrint==2) ? 0 : 4;
    xEnd = (dimToPrint==0) ? LATT_DIM_X : xStart+1;
    yEnd = (dimToPrint==1) ? LATT_DIM_Y : yStart+1;
    zEnd = (dimToPrint==2) ? LATT_DIM_Z : zStart+1;

    for(int z=zStart; z<zEnd; z++) {
        for(int y=yStart; y<yEnd; y++) {
            for(int x=xStart; x<xEnd; x++) {
                int index = IDX(x,y,z);
                float val = input[index];
                printf("%f\t", val);
            }
        }
    }
    printf("\n");
}

// Kernel to solve finite difference wave equation - output stores t-2 step, input stores t-1 step
#ifdef LOCKFREE
__global__ void wave_kernel(float* output_buffer, float* input_buffer, float vsq, float* coeff, int iterations,
                   volatile int* arrayIn, volatile int* arrayOut) {
#else
__global__ void wave_kernel(float* output_buffer, float* input_buffer, float vsq, float* coeff, int iterations) {
#endif
    const int bid_start = blockIdx.x;
    const int num_conc_blocks = gridDim.x;
    const int num_blocks_x = ((LATT_DIM_X-8)/blockDim.x) + (((LATT_DIM_X-8)%blockDim.x)?1:0);
    const int num_blocks_y = ((LATT_DIM_Y-8)/blockDim.y) + (((LATT_DIM_Y-8)%blockDim.y)?1:0);
    const int num_blocks_z = ((LATT_DIM_Z-8)/blockDim.z) + (((LATT_DIM_Z-8)%blockDim.z)?1:0);
    const int num_blocks = num_blocks_x*num_blocks_y*num_blocks_z;
    float* input = input_buffer;
    float* output = output_buffer;
    for(int i=0; i<iterations; i++) {
        for(int bid=bid_start; bid<num_blocks; bid+=num_conc_blocks) {
            // Get bid x,y,z from bid
            const int bidz = bid / (num_blocks_x*num_blocks_y);
            const int bidy = (bid % (num_blocks_x*num_blocks_y)) / num_blocks_x;
            const int bidx = bid % num_blocks_x;

            // Get tid x,y,z with offset of 4 included
            const int x = bidx*blockDim.x + threadIdx.x + 4;
            const int y = bidy*blockDim.y + threadIdx.y + 4;
            const int z = bidz*blockDim.z + threadIdx.z + 4;

            if(x<(LATT_DIM_X-4) and y<(LATT_DIM_Y-4) and z<(LATT_DIM_Z-4)) {
                // Do the computation
                float current = input[IDX(x,y,z)];
                float prev = output[IDX(x,y,z)];
                float temp = 2.0f*current - prev;
                float div = coeff[0] * current;
                div += coeff[1]*( input[IDX(x+1,y,z)] + input[IDX(x-1,y,z)] +
                                  input[IDX(x,y+1,z)] + input[IDX(x,y-1,z)] +
                                  input[IDX(x,y,z+1)] + input[IDX(x,y,z-1)] );
                div += coeff[2]*( input[IDX(x+2,y,z)] + input[IDX(x-2,y,z)] +
                                  input[IDX(x,y+2,z)] + input[IDX(x,y-2,z)] +
                                  input[IDX(x,y,z+2)] + input[IDX(x,y,z-2)] );
                div += coeff[3]*( input[IDX(x+3,y,z)] + input[IDX(x-3,y,z)] +
                                  input[IDX(x,y+3,z)] + input[IDX(x,y-3,z)] +
                                  input[IDX(x,y,z+3)] + input[IDX(x,y,z-3)] );
                div += coeff[4]*( input[IDX(x+4,y,z)] + input[IDX(x-4,y,z)] +
                                  input[IDX(x,y+4,z)] + input[IDX(x,y-4,z)] +
                                  input[IDX(x,y,z+4)] + input[IDX(x,y,z-4)] );
                output[IDX(x,y,z)] = temp + div*vsq;
            }
        }   // next block

        __threadfence();
//        __gpu_sync(i+1, arrayIn, arrayOut);
#ifdef LOCKFREE
        __syncblocks_lockfree(i+1, arrayIn, arrayOut);
#endif
#ifdef ATOMIC
        __syncblocks_atomic((i+1)*num_conc_blocks);
#endif
#ifdef HW_BARRIER
        __syncthreads();
#endif

        // Swap the buffers
        float* temp_buffer = input;
        input = output;
        output = temp_buffer;
    } // next iteration
}
/*
__device__ void __gpu_sync(int goalVal, volatile int* arrayIn, volatile int* arrayOut) {
    const int tid_in_block = threadIdx.z*(blockDim.x*blockDim.y) + threadIdx.y*(blockDim.x) + threadIdx.x;
    const int bid = blockIdx.x;
    const int num_conc_blocks = gridDim.x;

    // Notify that current block has reached sync
    if(tid_in_block == 0) {
        arrayIn[bid] = goalVal;
    }

    if(bid == 0) {
        if(tid_in_block < num_conc_blocks) {
            while(arrayIn[tid_in_block] != goalVal){
                // Wait for block to reach sync
            }
        }
        __syncthreads();

        // Notify all blocks of sync completion
        if(tid_in_block < num_conc_blocks) {
            arrayOut[tid_in_block] = goalVal;
        }
    }

    if(tid_in_block == 0) {
        while(arrayOut[bid] != goalVal) {
            // Wait for global sync notification
        }
    }
    __syncthreads();
}
*/

int main( int argc, const char** argv) {
	printf("[BENCH] Stencil-Wave <cxh@illinois.edu>\n");
#ifdef LOCKFREE
	printf("[BENCH] Lock Free Barrier\n");
#endif
#ifdef ATOMIC
	printf("[BENCH] Atomic Barrier\n");
#endif
#ifdef HW_BARRIER
	printf("[BENCH] Hardware Barrier\n");
#endif
   int thd_per_block_x = 8;
   int thd_per_block_y = 8;
   int thd_per_block_z = 8;

   // Get command line arguments if any
//   cutGetCmdLineArgumenti(argc, argv, "thdx", &thd_per_block_x);
//   cutGetCmdLineArgumenti(argc, argv, "thdy", &thd_per_block_y);
//   cutGetCmdLineArgumenti(argc, argv, "thdz", &thd_per_block_z);

   const int threads_per_block = thd_per_block_x*thd_per_block_y*thd_per_block_z;
   hipDeviceProp_t prop;
   hipGetDeviceProperties(&prop, 0);
   int num_blocks = prop.multiProcessorCount*2;
   printf("[BENCH] Block Size: %d (%d,%d,%d) \n", threads_per_block,
                   thd_per_block_x, thd_per_block_y, thd_per_block_z);
   printf("[BENCH] Number of Blocks: %d\n", num_blocks);

   // allocate host memory
   float* h_input_lattice_t0 = (float*) malloc( sizeof(float)*LATT_SIZE );
   float* h_input_lattice_t1 = (float*) malloc( sizeof(float)*LATT_SIZE );
   float vsq = 0.5f;
   float h_coeff[5] = {0.05f, -0.03f, 0.02f, -0.1f, 0.005f};
   const int iterations = 10;

   // Initialize input lattice on host side
   srand(2012);      // set seed for rand()
   generateInputLattice(h_input_lattice_t0, h_input_lattice_t1, vsq);

   // allocate and initialize device memory
   float* d_input_lattice_t0;
   float* d_input_lattice_t1;
   cutilSafeCall( hipMalloc( (void**) &d_input_lattice_t0, sizeof(float)*LATT_SIZE));
   cutilSafeCall( hipMalloc( (void**) &d_input_lattice_t1, sizeof(float)*LATT_SIZE));
   cutilSafeCall( hipMemcpy( d_input_lattice_t0, h_input_lattice_t0, sizeof(float)*LATT_SIZE, hipMemcpyHostToDevice) );
   cutilSafeCall( hipMemcpy( d_input_lattice_t1, h_input_lattice_t1, sizeof(float)*LATT_SIZE, hipMemcpyHostToDevice) );

   float* d_coeff;
   cutilSafeCall( hipMalloc( (void**) &d_coeff, sizeof(float)*5));
   cutilSafeCall( hipMemcpy( d_coeff, h_coeff, sizeof(float)*5, hipMemcpyHostToDevice) );

   // Device memory for synchronization buffers
#ifdef LOCKFREE
   int* d_arrayIn;
   int* d_arrayOut;
   cutilSafeCall( hipMalloc( (void**) &d_arrayIn, sizeof(int)*num_blocks));
   cutilSafeCall( hipMalloc( (void**) &d_arrayOut, sizeof(int)*num_blocks));
   cutilSafeCall( hipMemset( (void*) d_arrayIn, 0, sizeof(int)*num_blocks));
   cutilSafeCall( hipMemset( (void*) d_arrayOut, 0, sizeof(int)*num_blocks));
#endif

   // setup execution parameters
   dim3 grid( num_blocks, 1, 1 );
   dim3 threads( thd_per_block_x, thd_per_block_y, thd_per_block_z );

   // execute the kernel
#ifdef LOCKFREE
   wave_kernel<<<grid,threads>>>(d_input_lattice_t0, d_input_lattice_t1, vsq, d_coeff, iterations, d_arrayIn, d_arrayOut);
#else
   wave_kernel<<<grid,threads>>>(d_input_lattice_t0, d_input_lattice_t1, vsq, d_coeff, iterations);
#endif
   CUT_CHECK_ERROR("Kernel execution failed")

   // allocate memory for the result on host side
   float* h_output_lattice = (float*) malloc( sizeof(float)*LATT_SIZE );
   float* d_output_lattice = (iterations%2)? d_input_lattice_t0 : d_input_lattice_t1;   // select correct output buffer
   // copy result from device to host
   cutilSafeCall( hipMemcpy( h_output_lattice, d_output_lattice, sizeof(float)*LATT_SIZE, hipMemcpyDeviceToHost) );
   // allocate memory for gold result
   float* h_result_lattice = (float*) malloc( sizeof(float)*LATT_SIZE );
   // Run gold version on CPU
   wave_gold(h_input_lattice_t0, h_input_lattice_t1, vsq, h_coeff, iterations, h_result_lattice);
   // Print error
   print_error(h_output_lattice, h_result_lattice);
   // cleanup memory
   free( h_input_lattice_t0 );
   free( h_input_lattice_t1 );
   free( h_output_lattice );
   cutilSafeCall(hipFree(d_input_lattice_t0));
   cutilSafeCall(hipFree(d_input_lattice_t1));
   cutilSafeCall(hipFree(d_coeff));
   hipDeviceReset();
}

