
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void vecAdd_kernel(float *a, float *b, float *c, int n) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < n)
        c[id] = a[id] + b[id];
}

void vecAdd(float *h_a, float *h_b, float *h_c, int n) {
    float *d_a;
    float *d_b;
    float *d_c;
    size_t bytes = n*sizeof(float);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    int blockSize, gridSize;
    blockSize = 256;
    gridSize = (int)ceil((float)n/blockSize);
    vecAdd_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost );
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

int main(int argc, char* argv[]) {
    int n = 1024;
/*
    if(argc<2) {
      printf("Usage: ./vecAdd num_elements\n");
      return 0;
    }
*/
    if(argc>1)
      n = atoi(argv[1]);
    printf("[BENCH] CUDA Vector Addition, n = %d\n", n);
    printf("[BENCH] Xuhao Chen <cxh@illinois.edu>\n");
    float *h_a;
    float *h_b;
    float *h_c;
    size_t bytes = n*sizeof(float);
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);
    int i;
    for( i = 0; i < n; i++ ) {
        h_a[i] = ((float) rand() / (RAND_MAX));
        h_b[i] = ((float) rand() / (RAND_MAX));
    }

    vecAdd(h_a, h_b, h_c, n);

    float sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("[BENCH] Final result: %f\n", sum/n);
    free(h_a);
    free(h_b);
    free(h_c);
    return 0;
}
