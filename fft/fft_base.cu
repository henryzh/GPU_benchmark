#include "hip/hip_runtime.h"
// (C) Copyright 2013 University of Illinois, All Rights Reserved

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "parboil.h"
//#include "ib_sync.cu"

#define CUERR { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
  printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
  return -1; }}

#define bx  blockIdx.x
#define by  blockIdx.y
#define tx  threadIdx.x

// Possible values are 2, 4, 8 and 16
#define R 2
//#define SYNC

inline __device__ float2 operator*(float2 a, float2 b) { return make_float2( a.x*b.x-a.y*b.y, a.x*b.y+a.y*b.x ); }
inline __device__ float2 operator+(float2 a, float2 b) { return make_float2( a.x + b.x, a.y + b.y ); }
inline __device__ float2 operator-(float2 a, float2 b) { return make_float2( a.x - b.x, a.y - b.y ); }
inline __device__ float2 operator*(float2 a, float b ) { return make_float2( b*a.x , b*a.y); }

#define COS_PI_8  0.923879533f
#define SIN_PI_8  0.382683432f
#define exp_1_16  make_float2(  COS_PI_8, -SIN_PI_8 )
#define exp_3_16  make_float2(  SIN_PI_8, -COS_PI_8 )
#define exp_5_16  make_float2( -SIN_PI_8, -COS_PI_8 )
#define exp_7_16  make_float2( -COS_PI_8, -SIN_PI_8 )
#define exp_9_16  make_float2( -COS_PI_8,  SIN_PI_8 )
#define exp_1_8   make_float2(  1, -1 )
#define exp_1_4   make_float2(  0, -1 )
#define exp_3_8   make_float2( -1, -1 )
  
__device__ void GPU_FFT2( float2 &v1,float2 &v2 ) { 
  float2 v0 = v1;  
  v1 = v0 + v2; 
  v2 = v0 - v2; 
}

__device__ void GPU_FFT4( float2 &v0,float2 &v1,float2 &v2,float2 &v3) { 
   GPU_FFT2(v0, v2);
   GPU_FFT2(v1, v3);
   v3 = v3 * exp_1_4;
   GPU_FFT2(v0, v1);
   GPU_FFT2(v2, v3);    
}

inline __device__ void GPU_FFT2(float2* v) {
  GPU_FFT2(v[0],v[1]);
}

inline __device__ void GPU_FFT4(float2* v) {
  GPU_FFT4(v[0],v[1],v[2],v[3] );
}

inline __device__ void GPU_FFT8(float2* v) {
  GPU_FFT2(v[0],v[4]);
  GPU_FFT2(v[1],v[5]);
  GPU_FFT2(v[2],v[6]);
  GPU_FFT2(v[3],v[7]);
  v[5]=(v[5]*exp_1_8)*M_SQRT1_2;
  v[6]=v[6]*exp_1_4;
  v[7]=(v[7]*exp_3_8)*M_SQRT1_2;
  GPU_FFT4(v[0],v[1],v[2],v[3]);
  GPU_FFT4(v[4],v[5],v[6],v[7]);
}

inline __device__ void GPU_FFT16( float2 *v ) {
    GPU_FFT4( v[0], v[4], v[8], v[12] );
    GPU_FFT4( v[1], v[5], v[9], v[13] );
    GPU_FFT4( v[2], v[6], v[10], v[14] );
    GPU_FFT4( v[3], v[7], v[11], v[15] );

    v[5]  = (v[5]  * exp_1_8 ) * M_SQRT1_2;
    v[6]  =  v[6]  * exp_1_4;
    v[7]  = (v[7]  * exp_3_8 ) * M_SQRT1_2;
    v[9]  =  v[9]  * exp_1_16;
    v[10] = (v[10] * exp_1_8 ) * M_SQRT1_2;
    v[11] =  v[11] * exp_3_16;
    v[13] =  v[13] * exp_3_16;
    v[14] = (v[14] * exp_3_8 ) * M_SQRT1_2;
    v[15] =  v[15] * exp_9_16;

    GPU_FFT4( v[0],  v[1],  v[2],  v[3] );
    GPU_FFT4( v[4],  v[5],  v[6],  v[7] );
    GPU_FFT4( v[8],  v[9],  v[10], v[11] );
    GPU_FFT4( v[12], v[13], v[14], v[15] );
}
     
__device__ int GPU_expand(int idxL, int N1, int N2 ) { 
  return (idxL/N1)*N1*N2 + (idxL%N1);
}

__device__ void GPU_FftIteration(int j, int Ns, float2* data0, float2* data1, int N) { 
  float2 v[R];
  int idxS = j;
  float angle = -2*M_PI*(j%Ns)/(Ns*R);
  for(int r=0; r<R; r++) {
    v[r] = data0[idxS+r*N/R];
    v[r] = v[r]*make_float2(cos(r*angle), sin(r*angle));
  }

#if R == 2 
  GPU_FFT2( v ); 
#endif

#if R == 4
  GPU_FFT4( v );
#endif

#if R == 8
  GPU_FFT8( v );
#endif

#if R == 16
  GPU_FFT16( v );
#endif

  int idxD = GPU_expand(j,Ns,R);
  for( int r=0; r<R; r++ ) {
    data1[idxD+r*Ns] = v[r];
  }
}      

__global__ void GPU_FFT_Global(int Ns, float2* data0, float2* data1, int N) { 
  data0+=bx*N;
  data1+=bx*N;
  GPU_FftIteration(tx, Ns, data0, data1, N);
}

#ifdef SYNC
__global__ void GPU_FFT_Sync(float2* data0, float2* data1, int N) {
//  int numBlocks = gridDim.x * gridDim.y * gridDim.z;
//  int goalVal = 0;
  for(int Ns=1; Ns<N; Ns*=R) {
    float2* d0 = data0 + bx*N;
    float2* d1 = data1 + bx*N;
    GPU_FftIteration(tx, Ns, d0, d1, N);
    float2 *tmp = data0;
    data0 = data1;
    data1 = tmp;
    __syncthreads();
//    goalVal += numBlocks;
//    __ib_sync(goalVal);
  }
}
#endif

void inputData(char* fName, float* dat, int numwords) {
  FILE* fid = fopen(fName, "r");
  if (fid == NULL) {
      fprintf(stderr, "Cannot open input file\n");
      exit(-1);
  }
  size_t result = fread(dat, sizeof(float), numwords, fid);
  if (result != numwords) {fputs("Reading error", stderr); exit(3);}
  fclose(fid); 
}

void outputData(char* fName, float* outdat, int numwords) {
  FILE* fid = fopen(fName, "w");
  unsigned size;
  if (fid == NULL) {
      fprintf(stderr, "Cannot open output file\n");
      exit(-1);
  }
  size = numwords;
  fwrite(&size, sizeof(unsigned), 1, fid);
  fwrite (outdat, sizeof (float), numwords, fid);
  fclose (fid);
}

#define TIMING
#define READ_FILE
int main( int argc, char **argv ) {
  printf("[BENCH] CUDA Naive FFT, Xuhao Chen, IMPACT UIUC\n");
#ifndef SYNC
  printf("[BENCH] Running original version\n");
#else
  printf("[BENCH] Running block sync version\n");
#endif

  int N=256, B=1024;
#ifdef READ_FILE
  struct pb_Parameters *params;
  params = pb_ReadParameters(&argc, argv);
  if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL)) {
      fprintf(stderr, "Expecting one input filename\n");
      exit(-1);
  }
  int err = 0;
  if(argc != 3)
    err |= 1;
  else {
    char* numend;
    N = strtol(argv[1], &numend, 10);
    if(numend == argv[1])
      err |= 2;
    B = strtol(argv[2], &numend, 10);
    if(numend == argv[2])
      err |= 4;
  }
  if(err) {
    fprintf(stderr, "Expecting two integers for N and B\n");
    exit(-1);
  }
#endif
  printf("[BENCH] R=%d, N=%d, B=%d\n", R, N, B);

#ifdef TIMING
  struct pb_TimerSet timers;
  pb_InitializeTimerSet(&timers);
#endif

  int n_bytes = N*B*sizeof(float2);
  float2 *source = (float2 *)malloc(n_bytes);
  float2 *result = (float2 *)malloc(n_bytes);

#ifdef READ_FILE
#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
#endif
  inputData(params->inpFiles[0], (float*)source, N*B*2);
#else
  for( int i = 0; i < N*B; i++ ) {
    source[i].x = (rand()/(float)RAND_MAX)*2-1;
    source[i].y = (rand()/(float)RAND_MAX)*2-1;
  }
#endif

#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_COPY);
#endif
  float2 *d_source, *d_work;
  hipMalloc((void**) &d_source, n_bytes);
  CUERR;
  hipMemcpy(d_source, source, n_bytes,hipMemcpyHostToDevice);
  CUERR;
  hipMalloc((void**) &d_work, n_bytes);
  CUERR;
  hipMemset(d_work, 0, n_bytes);
  CUERR;

#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);
#endif

#ifdef SYNC
  GPU_FFT_Sync<<<dim3(B), dim3(N/R)>>>(d_source, d_work, N);
#else
  for(int Ns=1; Ns<N; Ns*=R) {
    GPU_FFT_Global<<<dim3(B), dim3(N/R)>>>(Ns, d_source, d_work, N);
    float2 *tmp = d_source;
    d_source = d_work;
    d_work = tmp;
  }
#endif

#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_COPY);
#endif

  hipMemcpy(result, d_source, n_bytes, hipMemcpyDeviceToHost);
  CUERR;
  hipFree(d_source);
  CUERR;
  hipFree(d_work);
  CUERR;

#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
#endif
  printf("[BENCH] Writing output to file <result.dat>\n");
  outputData("result.dat", (float*)result, N*B*2);

  free(source);
  free(result);
#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);
#endif
}

