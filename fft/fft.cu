#include "hip/hip_runtime.h"
// Author: Sara Baghsorkhi. This implementation is partly based on the SC08 paper by Naga K. Govindaraju et al.

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../parboil.h"

#define CUERR { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
  printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
  return -1; }}

#define bx blockIdx.x
#define by blockIdx.y
#define tx threadIdx.x

#define DEBUG 0
#define EMUL 0

#define R2 1
#define R4 0
#define R8 0
#define R16 0

#if R2
#define R 2
#endif

#if R4
#define R 4
#endif

#if R8
#define R 8
#endif

#if R16
#define R 16
#endif

#define N 256
#define B 1024
#define T N/R

inline __device__ float2 operator*(float2 a, float2 b) { return make_float2(a.x*b.x-a.y*b.y, a.x*b.y+a.y*b.x); }
inline __device__ float2 operator+(float2 a, float2 b) { return make_float2(a.x + b.x, a.y + b.y); }
inline __device__ float2 operator-(float2 a, float2 b) { return make_float2(a.x - b.x, a.y - b.y); }
inline __device__ float2 operator*(float2 a, float b)  { return make_float2(b*a.x , b*a.y); }

#define COS_PI_8  0.923879533f
#define SIN_PI_8  0.382683432f
#define exp_1_16  make_float2(  COS_PI_8, -SIN_PI_8 )
#define exp_3_16  make_float2(  SIN_PI_8, -COS_PI_8 )
#define exp_5_16  make_float2( -SIN_PI_8, -COS_PI_8 )
#define exp_7_16  make_float2( -COS_PI_8, -SIN_PI_8 )
#define exp_9_16  make_float2( -COS_PI_8,  SIN_PI_8 )
#define exp_1_8   make_float2(  1, -1 )//requires post-multiply by 1/sqrt(2)
#define exp_1_4   make_float2(  0, -1 )
#define exp_3_8   make_float2( -1, -1 )//requires post-multiply by 1/sqrt(2)
  
void FFT2( float2* v ) { 
  float2 v0 = v[0];  
  v[0] = v0 + v[1]; 
  v[1] = v0 - v[1]; 
}        

__device__ void GPU_FFT2( float2 &v1,float2 &v2 ) { 
  float2 v0 = v1;  
  v1 = v0 + v2; 
  v2 = v0 - v2; 
}

__device__ void GPU_FFT4( float2 &v0,float2 &v1,float2 &v2,float2 &v3) { 
   GPU_FFT2(v0, v2);
   GPU_FFT2(v1, v3);
   v3 = v3 * exp_1_4;
   GPU_FFT2(v0, v1);
   GPU_FFT2(v2, v3);    
}

inline __device__ void GPU_FFT2(float2* v) {
  GPU_FFT2(v[0],v[1]);
}

inline __device__ void GPU_FFT4(float2* v) {
  GPU_FFT4(v[0],v[1],v[2],v[3] );
}

inline __device__ void GPU_FFT8(float2* v) {
  GPU_FFT2(v[0],v[4]);
  GPU_FFT2(v[1],v[5]);
  GPU_FFT2(v[2],v[6]);
  GPU_FFT2(v[3],v[7]);

  v[5]=(v[5]*exp_1_8)*M_SQRT1_2;
  v[6]=v[6]*exp_1_4;
  v[7]=(v[7]*exp_3_8)*M_SQRT1_2;

  GPU_FFT4(v[0],v[1],v[2],v[3]);
  GPU_FFT4(v[4],v[5],v[6],v[7]);
  
}

inline __device__ void GPU_FFT16( float2 *v ) {
    GPU_FFT4( v[0], v[4], v[8], v[12] );
    GPU_FFT4( v[1], v[5], v[9], v[13] );
    GPU_FFT4( v[2], v[6], v[10], v[14] );
    GPU_FFT4( v[3], v[7], v[11], v[15] );

    v[5]  = (v[5]  * exp_1_8 ) * M_SQRT1_2;
    v[6]  =  v[6]  * exp_1_4;
    v[7]  = (v[7]  * exp_3_8 ) * M_SQRT1_2;
    v[9]  =  v[9]  * exp_1_16;
    v[10] = (v[10] * exp_1_8 ) * M_SQRT1_2;
    v[11] =  v[11] * exp_3_16;
    v[13] =  v[13] * exp_3_16;
    v[14] = (v[14] * exp_3_8 ) * M_SQRT1_2;
    v[15] =  v[15] * exp_9_16;

    GPU_FFT4( v[0],  v[1],  v[2],  v[3] );
    GPU_FFT4( v[4],  v[5],  v[6],  v[7] );
    GPU_FFT4( v[8],  v[9],  v[10], v[11] );
    GPU_FFT4( v[12], v[13], v[14], v[15] );
}
     
__device__ int GPU_expand(int idxL, int N1, int N2) { 
  return (idxL/N1)*N1*N2 + (idxL%N1); 
}      

__device__ void GPU_exchange(float2* v, int stride, int idxD, int incD, int idxS, int incS) { 
  __shared__ float work[T*R*2];//T*R*2
  float* sr = work;
  float* si = work+T*R;  
  __syncthreads(); 
  for( int r=0; r<R; r++ ) { 
    int i = (idxD + r*incD)*stride; 
    sr[i] = v[r].x;
    si[i] = v[r].y;  
  }   
  __syncthreads(); 

  for( int r=0; r<R; r++ ) { 
    int i = (idxS + r*incS)*stride;     
    v[r] = make_float2(sr[i], si[i]);  
  }        
}      

__device__ void GPU_DoFft(float2* v, int j, int stride=1) { 
  for( int Ns=1; Ns<N; Ns*=R ) { 
    float angle = -2*M_PI*(j%Ns)/(Ns*R); 
    for( int r=0; r<R; r++ ) {
      v[r] = v[r]*make_float2(cos(r*angle), sin(r*angle));
    }
#if R2
    GPU_FFT2(v);
#endif

#if R4
    GPU_FFT4(v);
#endif

#if R8
    GPU_FFT8(v);	
#endif

#if R16
    GPU_FFT16(v);
#endif

    int idxD = GPU_expand(j, Ns, R); 
    int idxS = GPU_expand(j, N/R, R); 
    GPU_exchange(v, stride, idxD, Ns, idxS, N/R);
  }      
}    

__global__ void GPU_FftShMem(float2* data) { 
  float2 v[R];
  data+=bx*N; 
  	
  int idxG = tx; 
  for( int r=0; r<R; r++ ) {  
    v[r] = data[idxG + r*T];
  } 
  GPU_DoFft( v, tx );  
  for( int r=0; r<R; r++ )  
    data[idxG + r*T] = v[r]; 
}

void inputData(char* fName, float* dat, int numwords) {
  FILE* fid = fopen(fName, "r");
  if (fid == NULL) {
      fprintf(stderr, "Cannot open input file\n");
      exit(-1);
  }
  size_t result = fread(dat, sizeof(float), numwords, fid);
  if (result != numwords) {fputs("Reading error", stderr); exit(3);}
  fclose(fid); 
}

void outputData(char* fName, float* outdat, int numwords) {
  FILE* fid = fopen(fName, "w");
  unsigned size;
  if (fid == NULL) {
      fprintf(stderr, "Cannot open output file\n");
      exit(-1);
  }
  size = numwords;
  fwrite(&size, sizeof(unsigned), 1, fid);
  fwrite (outdat, sizeof (float), numwords, fid);
  fclose (fid);
}

#define TIMING
#define READ_FILE
int main( int argc, char **argv ) {
  printf("[BENCH] CUDA FFT, Xuhao Chen, IMPACT UIUC\n");
#ifdef READ_FILE
  struct pb_Parameters *params;
  params = pb_ReadParameters(&argc, argv);
  if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL)) {
      fprintf(stderr, "Expecting one input filename\n");
      exit(-1);
  }
#endif
  printf("[BENCH] R=%d, N=%d, B=%d\n", R, N, B);
  
#ifdef TIMING
  struct pb_TimerSet timers;
  pb_InitializeTimerSet(&timers);
#endif

  int n_bytes = N*B*sizeof(float2);
  int nthreads = N/R;
//  srand(54321);
  float *shared_source =(float *)malloc(n_bytes);  
  float2 *source    = (float2 *)malloc( n_bytes );
  float2 *result    = (float2 *)malloc( n_bytes );

#ifdef READ_FILE
#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
#endif
  inputData(params->inpFiles[0], (float*)source, N*B*2);
#else
  for(int b=0; b<B; b++){	
    for(int i=0; i<N; i++){
      source[b*N+i].x = (rand()/(float)RAND_MAX)*2-1;
      source[b*N+i].y = (rand()/(float)RAND_MAX)*2-1;
    }
  }
#endif

#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_COPY);
#endif
  float2 *d_source, *d_work;
  float *d_shared_source;
  hipMalloc((void**) &d_shared_source, n_bytes);
  CUERR;
  hipMemcpy(d_shared_source, shared_source, n_bytes,hipMemcpyHostToDevice);
  CUERR;
  hipMalloc((void**) &d_source, n_bytes);
  CUERR;
  hipMemcpy(d_source, source, n_bytes,hipMemcpyHostToDevice);
  CUERR;
  hipMalloc((void**) &d_work, n_bytes);
  CUERR;
  hipMemset(d_work, 0,n_bytes);
  CUERR;

#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);
#endif

  GPU_FftShMem<<<dim3(B), dim3(nthreads)>>>(d_source);

#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_COPY);
#endif

  hipMemcpy(result, d_source, n_bytes, hipMemcpyDeviceToHost);
  CUERR;
  hipFree(d_source);
  CUERR;
  hipFree(d_work);
  CUERR;

#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
#endif
  printf("[BENCH] Writing output to file <result.dat>\n");
  outputData("result.dat", (float*)result, N*B*2);
  free(shared_source);  
  free(source);
  free(result);
#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);
#endif
}

