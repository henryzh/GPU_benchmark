
#include <hip/hip_runtime.h>
__device__ void __ib_sync_lockfree(int goalVal, volatile int *Arrayin, volatile int *Arrayout) {
	int tx = threadIdx.x;// * blockDim.y + threadIdx.y;
	int numBlocks = gridDim.x;// * gridDim.y;
	int bid = blockIdx.x;// * gridDim.y + blockIdx.y;
	if(tx == 0) {
		Arrayin[bid] = goalVal;
	}
	if(bid == 1) {
		if(tx < numBlocks) {
			while (Arrayin[tx] != goalVal) {}
		}
		__syncthreads();
		if(tx < numBlocks) {
			Arrayout[tx] = goalVal;
		}
	}
	if(tx == 0) {
		while(Arrayout[bid] != goalVal) {}
	}
	__syncthreads();
}

__device__ volatile int g_mutex;
__device__ void __ib_sync(int goal) {
//	__syncthreads();
	int tx = threadIdx.x;// * blockDim.y + threadIdx.y;
	if (tx == 0) {
		atomicAdd((int *)&g_mutex, 1);
		while(g_mutex != goal) {}
	}
	__syncthreads();
}
