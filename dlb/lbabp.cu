/**
 * Octree Partitioning
 * Benchmark for dynamic load balancing using
 * work-stealing on graphics processors.
 * --------------------------------------------------------
 * Copyright 2011 Daniel Cederman and Philippas Tsigas
 *
 * This work is licensed under the Creative Commons
 * Attribution 3.0 Unported (CC BY 3.0) License.
 * To view a copy of this license, visit
 * http://creativecommons.org/licenses/by/3.0 .
 *
**/

#include "lbabp.h"
#include "helper.h"


LBABP::~LBABP()
{
	if(init)
	{
		hipFree(dwq);
		hipFree(wq->deq);
		hipFree(wq->dh);
	}

}


bool LBABP::setQueueSize(unsigned int dequelength, unsigned int blocks)
{
	init = true;
	wq = (DLBABP*)malloc(sizeof(DLBABP));

	CUDA_SAFE_CALL(hipMalloc((void**)&dwq,sizeof(DLBABP)));

	CUDA_SAFE_CALL(hipMalloc((void**)&(wq->deq),sizeof(Task)*dequelength*blocks));
	CUDA_SAFE_CALL(hipMalloc((void**)&(wq->dh),sizeof(DequeHeader)*blocks));

	CUDA_SAFE_CALL(hipMemset(wq->deq,0,sizeof(Task)*dequelength*blocks));
	CUDA_SAFE_CALL(hipMemset(wq->dh,0,sizeof(DequeHeader)*blocks));

	wq->maxlength = dequelength;
	CUDA_SAFE_CALL(hipMemcpy(dwq,wq,sizeof(DLBABP),hipMemcpyHostToDevice));
	return true;
}

int LBABP::getMaxMem()
{
	int maxle;
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&maxle,HIP_SYMBOL(maxl),sizeof(int),0,hipMemcpyDeviceToHost));
	return maxle;
}
