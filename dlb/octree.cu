#include "hip/hip_runtime.h"
/**
 * Octree Partitioning
 * Benchmark for dynamic load balancing using
 * work-stealing on graphics processors.
 * --------------------------------------------------------
 * Copyright 2011 Daniel Cederman and Philippas Tsigas
 *
 * This work is licensed under the Creative Commons
 * Attribution 3.0 Unported (CC BY 3.0) License.
 * To view a copy of this license, visit
 * http://creativecommons.org/licenses/by/3.0 .
 *
**/

#include "octree.h"
#include "helper.h"
#include "octree_kernel.h"

void Octree::printStats()
{
	unsigned int* htree = new unsigned int[MAXTREESIZE];
	unsigned int htreeSize;
	unsigned int hparticlesDone;

	CUDA_SAFE_CALL(hipMemcpy(&hparticlesDone,particlesDone,sizeof(unsigned int),hipMemcpyDeviceToHost));

	CUDA_SAFE_CALL(hipMemcpy(&htreeSize,treeSize,sizeof(unsigned int),hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(htree,tree,sizeof(unsigned int)*MAXTREESIZE,hipMemcpyDeviceToHost));

	if(htreeSize>=MAXTREESIZE)
	{
		printf("Tree to large!\n");
		return;
	}

	unsigned int sum = 0;
	for(unsigned int i=0;i<htreeSize;i++)
	{
		if(htree[i]&0x80000000)
		{
			sum+=htree[i]&0x7fffffff;
		}
	}

	printf("Tree size: %d\n",htreeSize);
	printf("Particles in tree: %d (%d) [%d]\n",sum,numParticles,hparticlesDone);

	delete htree;
}

bool Octree::run(unsigned int threads, unsigned int blocks, LBMethod method, int maxChildren, int numParticles)
{
	this->method = method;
	this->numParticles = numParticles;

	CUDA_SAFE_CALL(hipMalloc((void**)&tree,sizeof(unsigned int)*MAXTREESIZE));
	CUDA_SAFE_CALL(hipMemset((void*)tree,0,sizeof(unsigned int)*MAXTREESIZE));

	CUDA_SAFE_CALL(hipMalloc((void**)&particles,sizeof(float4)*numParticles));
	CUDA_SAFE_CALL(hipMalloc((void**)&newParticles,sizeof(float4)*numParticles));
	CUDA_SAFE_CALL(hipMalloc((void**)&particlesDone,sizeof(unsigned int)));

	CUDA_SAFE_CALL(hipMalloc((void**)&treeSize,sizeof(unsigned int)));


	generateParticles();

	if(method==Dynamic)
		lbws.setQueueSize(64,blocks);
	else
		if(method==Static)
			lbstat.setQueueSize(900000,blocks);

	if(method == Dynamic)
		initOctree<DLBABP><<<1,1>>>(lbws.deviceptr(),treeSize,particlesDone,numParticles);

	if(method == Static)
		initOctree<DLBStatic><<<1,1>>>(lbstat.deviceptr(),treeSize,particlesDone,numParticles);

	CUT_CHECK_ERROR("initOctree failed!\n");

	Time timer(1);
	timer.start();

	if(method == Dynamic)
		makeOctree<DLBABP><<<blocks,threads>>>(lbws.deviceptr(),particles,newParticles,tree,numParticles,treeSize,particlesDone,maxChildren,false);
	else
		if(method == Static)
		{
			while((lbstat.blocksleft())!=0)
			{

				makeOctree<DLBStatic><<<blocks,threads>>>(lbstat.deviceptr(),particles,newParticles,tree,numParticles,treeSize,particlesDone,maxChildren,true);
			}
		}

		CUT_CHECK_ERROR("makeOctree failed!\n");

		float time = timer.stop();

		totalTime = time;

		CUDA_SAFE_CALL(hipFree(newParticles));
		return true;
}

double genrand_real1(void);
void Octree::generateParticles()
{
	float4* lparticles = new float4[numParticles];

	char fname[256];
	sprintf(fname,"particles-%d.dat",numParticles);
	FILE* f = fopen(fname,"rb");
	if(!f)
	{
		printf("Generating and caching data\n");

		int clustersize = 100;
		for(unsigned int i=0;i<numParticles/clustersize;i++)
		{
			float x = ((float)genrand_real1()*800.0f-400.0f);
			float y = ((float)genrand_real1()*800.0f-400.0f);
			float z = ((float)genrand_real1()*800.0f-400.0f);

			for(int x=0;x<clustersize;x++)
			{	
				lparticles[i*clustersize+x].x = x + ((float)genrand_real1()*100.0f-50.0f);
				lparticles[i*clustersize+x].y = y + ((float)genrand_real1()*100.0f-50.0f);
				lparticles[i*clustersize+x].z = z + ((float)genrand_real1()*100.0f-50.0f);

			}

		}

		FILE* f = fopen(fname,"wb");
		if(f == NULL) {
		    printf("Error: Cannot create output file '%s' for particle data.\n", fname);
		    abort();
		}
		fwrite(lparticles,sizeof(float4),numParticles,f);
		fclose(f);
	}
	else
	{
		fread(lparticles,sizeof(float4),numParticles,f);
		fclose(f);
	}

	CUDA_SAFE_CALL(hipMemcpy(particles,lparticles,sizeof(float4)*numParticles,hipMemcpyHostToDevice));
	delete lparticles;
}

float Octree::getTime()
{
	return totalTime;
}

int Octree::getMaxMem()
{
	if(method==Dynamic)
		return lbws.getMaxMem();
	else
		if(method == Static)
			return lbstat.getMaxMem();

	return -1;
}
