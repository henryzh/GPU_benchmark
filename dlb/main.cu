/**
 * Octree Partitioning
 * Benchmark for dynamic load balancing using
 * work-stealing on graphics processors.
 * Copyright 2011 Daniel Cederman and Philippas Tsigas
**/

#include <string.h>
#include <stdio.h>
#include "octree.h"
#include "helper.h"

int main(int argc, char* argv[]) {
	if(argc!=6&&argc!=5) {
		printf("\nUsage:\t./octreepart threads blocks [abp|static] particleCount maxChildren\n\n");
		return 1;
	}
	int threads = atoi(argv[1]);
	if(threads<=0||threads>512) {
		printf("Threads must be between 1 and 128\n");
		return 1;
	}
	int blocks = atoi(argv[2]);
	if(blocks<=0||blocks>512) {
		printf("Blocks must be between 1 and 512\n");
		return 1;
	}
	int particleCount = atoi(argv[4]);
	if(particleCount<=0||particleCount>50000000) {
		printf("particleCount must be between 1 and 50000000\n");
		return 1;
	}
	int maxChildren = atoi(argv[5]);
	if(maxChildren<=0||maxChildren>100) {
		printf("maxChildren must be between 1 and 100\n");
		return 1;
	}
	LBMethod method;
	if(!strcmp(argv[3],"abp"))
		method=Dynamic;
	else
	if(!strcmp(argv[3],"static"))
		method=Static;
	else {
		printf("Load balancing method needs to be either 'abp' or 'static'\n");
		return 1;
	}
	Octree o;
	o.run(threads,blocks,method,maxChildren,particleCount);
	printf("Threads: %d Blocks: %d Method: %s ParticleCount: %d maxChildren: %d MaxMem: %d Time: %f\n",threads,blocks,argv[3],particleCount,maxChildren,o.getMaxMem(),o.getTime());
	o.printStats();
	return 0;
}
