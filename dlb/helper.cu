/**
 * Octree Partitioning
 * Benchmark for dynamic load balancing using
 * work-stealing on graphics processors.
 * --------------------------------------------------------
 * Copyright 2011 Daniel Cederman and Philippas Tsigas
 *
 * This work is licensed under the Creative Commons
 * Attribution 3.0 Unported (CC BY 3.0) License.
 * To view a copy of this license, visit
 * http://creativecommons.org/licenses/by/3.0 .
 *
**/


#include "helper.h"

Time::Time(unsigned int iterations):iterations(iterations)
{
	hipDeviceSynchronize();
	hipEventCreate(&startevent);
	hipEventCreate(&endevent);
}

void Time::start()
{
	hipEventRecord(startevent, 0);
}

float Time::stop()
{
   hipEventRecord(endevent, 0);
   hipEventSynchronize(endevent);

   float runTime;
   hipEventElapsedTime(&runTime, startevent, endevent);
   runTime /= float(iterations);

   return runTime;
}
