/**
 * Octree Partitioning
 * Benchmark for dynamic load balancing using
 * work-stealing on graphics processors.
 * --------------------------------------------------------
 * Copyright 2011 Daniel Cederman and Philippas Tsigas
 *
 * This work is licensed under the Creative Commons
 * Attribution 3.0 Unported (CC BY 3.0) License.
 * To view a copy of this license, visit
 * http://creativecommons.org/licenses/by/3.0 .
 *
**/

#include "task.h"
#include "lbstatic.h"
#include "helper.h"

LBStatic::~LBStatic()
{
	if(init)
	{
		hipFree(dwq);
		hipFree(wq->indeq);
		hipFree(wq->outdeq);
		free(wq);
	}
}

bool LBStatic::setQueueSize(unsigned int dequelength, unsigned int blocks)
{
	init = true;
	this->blocks = blocks;
	wq = (DLBStatic*)malloc(sizeof(DLBStatic));
	CUDA_SAFE_CALL(hipMalloc((void**)&dwq,sizeof(DLBStatic)));

	CUDA_SAFE_CALL(hipMalloc((void**)&(wq->indeq),sizeof(Task)*dequelength));
	CUDA_SAFE_CALL(hipMalloc((void**)&(wq->outdeq),sizeof(Task)*dequelength));

	CUDA_SAFE_CALL(hipMalloc((void**)&(wq->ctrs),sizeof(unsigned int)*blocks));

	CUDA_SAFE_CALL(hipMemset(wq->ctrs,0,sizeof(unsigned int)*blocks));

	wq->ctr=0;
	wq->ctr2=0;

	CUDA_SAFE_CALL(hipMemcpy(dwq,wq,sizeof(DLBStatic),hipMemcpyHostToDevice));

	return true;
}

int LBStatic::getMaxMem()
{
	return smaxl;
}

unsigned int LBStatic::blocksleft()
{
	CUDA_SAFE_CALL(hipMemcpy(wq,dwq,sizeof(DLBStatic),hipMemcpyDeviceToHost));
	if(wq->ctr==0)
		return 0;

	Task* t = wq->indeq;
	wq->indeq = wq->outdeq;
	wq->outdeq = t;

	int rval = wq->ctr;
	wq->ctr = 0;
	wq->ctr2 = rval;

	CUDA_SAFE_CALL(hipMemcpy(dwq,wq,sizeof(DLBStatic),hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemset(wq->ctrs,0,sizeof(unsigned int)*blocks));

	if(smaxl<(int)wq->ctr2)
		smaxl=(int)wq->ctr2;

	return rval;
}
