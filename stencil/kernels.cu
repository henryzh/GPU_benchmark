#include "hip/hip_runtime.h"
// CUDA Stencil, Xuhao Chen
#include "common.h"
#ifdef NAIVE
__global__ void naive(float c0, float c1, float *A0, float *Anext, int nx, int ny, int nz) {
	int i = threadIdx.x;
	int j = blockIdx.x+1;
	int k = blockIdx.y+1;

	if(i>0) {
	Anext[Index3D (nx, ny, i, j, k)] = 
	(A0[Index3D (nx, ny, i, j, k + 1)] +
	A0[Index3D (nx, ny, i, j, k - 1)] +
	A0[Index3D (nx, ny, i, j + 1, k)] +
	A0[Index3D (nx, ny, i, j - 1, k)] +
	A0[Index3D (nx, ny, i + 1, j, k)] +
	A0[Index3D (nx, ny, i - 1, j, k)])*c1
	- A0[Index3D (nx, ny, i, j, k)]*c0;
	}
}
#endif

#ifdef TILE_3D_OLD
__global__ void tile_3D_old(float c0, float c1, float *A0, float *Anext, int nx, int ny, int nz) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int tz = threadIdx.z;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int bz = blockIdx.z;
	int i = bx*blockDim.x+tx;
	int j = by*blockDim.y+ty;
	int k = bz*blockDim.z+tz;
	const int sh_id=tx+ty*blockDim.x+tz*blockDim.x*blockDim.y;
	extern __shared__ float sh_A0[];
	sh_A0[sh_id]=0.0f;
	__syncthreads();

	bool w_region = (i>0 && j>0 && k>0 && (i<nx-1) && (j<ny-1) && (k<nz-1));
	bool x_l_bound = (tx==0);
	bool x_h_bound = (tx==(blockDim.x-1));
	bool y_l_bound = (ty==0);
	bool y_h_bound = (ty==(blockDim.y-1));
	bool z_l_bound = (tz==0);
	bool z_h_bound = (tz==(blockDim.z-1));
	sh_A0[sh_id] = A0[Index3D(nx, ny, i, j, k)];
	__syncthreads();

	if(w_region) {
		float front, back, left, right, top, down;
		if(x_l_bound) left=A0[Index3D(nx, ny, i-1, j, k)];
		else left=sh_A0[sh_id-1];
		if(x_h_bound) right=A0[Index3D(nx, ny, i+1, j, k)];
		else right=sh_A0[sh_id+1];
		if(y_l_bound) back=A0[Index3D(nx, ny, i, j-1, k)];
		else back=sh_A0[sh_id-blockDim.x];
		if(y_h_bound) front=A0[Index3D(nx, ny, i, j+1, k)];
		else front=sh_A0[sh_id+blockDim.x];
		if(z_l_bound) down=A0[Index3D(nx, ny, i, j, k-1)];
		else down=sh_A0[sh_id-blockDim.x*blockDim.y];
		if(z_h_bound) top=A0[Index3D(nx, ny, i, j, k+1)];
		else top=sh_A0[sh_id+blockDim.x*blockDim.y];
		Anext[Index3D(nx, ny, i, j, k)] = (front+back+left+right+top+down)*c1 - sh_A0[sh_id]*c0;
	}
}
#endif
#ifdef TILE_3D_NEW
__global__ void tile_3D_new(int iter, float c0, float c1, float *A0, float *Anext, int nx, int ny, int nz) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int tz = threadIdx.z;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int bz = blockIdx.z;
	int i = bx*blockDim.x+tx;
	int j = by*blockDim.y+ty;
	int k = bz*blockDim.z+tz;
	const int num_blocks = gridDim.x;//*gridDim.y*gridDim.z;
	const int tid=tx+ty*blockDim.x+tz*blockDim.x*blockDim.y;
//	if(tid==0) printf("num_blocks = %d\n", num_blocks);
	__shared__ float sh_mem[2*BSX*BSY*BSZ];
	float *sh_A0=sh_mem;
	float *sh_Anext=sh_mem+BSX*BSY*BSZ;
	sh_A0[tid]=0.0f;
	sh_Anext[tid]=0.0f;
	__syncthreads();

	bool w_region = (i>0 && j>0 && k>0 && (i<nx-1) && (j<ny-1) && (k<nz-1));
	bool x_l_bound = (tx==0);
	bool x_h_bound = (tx==(blockDim.x-1));
	bool y_l_bound = (ty==0);
	bool y_h_bound = (ty==(blockDim.y-1));
	bool z_l_bound = (tz==0);
	bool z_h_bound = (tz==(blockDim.z-1));

	sh_A0[tid] = A0[Index3D(nx, ny, i, j, k)];
	__syncthreads();
///*
	for(int t=0;t<iter;t++) {
		if(w_region) {
			float front, back, left, right, top, down;
			if(x_l_bound) left=A0[Index3D(nx, ny, i-1, j, k)];
			else left=sh_A0[tid-1];
			if(x_h_bound) right=A0[Index3D(nx, ny, i+1, j, k)];
			else right=sh_A0[tid+1];
			if(y_l_bound) back=A0[Index3D(nx, ny, i, j-1, k)];
			else back=sh_A0[tid-blockDim.x];
			if(y_h_bound) front=A0[Index3D(nx, ny, i, j+1, k)];
			else front=sh_A0[tid+blockDim.x];
			if(z_l_bound) down=A0[Index3D(nx, ny, i, j, k-1)];
			else down=sh_A0[tid-blockDim.x*blockDim.y];
			if(z_h_bound) top=A0[Index3D(nx, ny, i, j, k+1)];
			else top=sh_A0[tid+blockDim.x*blockDim.y];
			sh_Anext[tid] = (front+back+left+right+top+down)*c1 - sh_A0[tid]*c0;
			Anext[Index3D(nx, ny, i, j, k)] = sh_Anext[tid];
		}
		__threadfence();
#ifdef ATOMIC
		__syncblocks_atomic((t+1)*num_blocks);
#endif
		float *sh_temp = sh_A0;
		sh_A0 = sh_Anext;
		sh_Anext = sh_temp;
	}
//*/
	Anext[Index3D(nx, ny, i, j, k)] = sh_A0[tid];
}
#endif
#ifdef TILE_2D_OLD
__global__ void tile_2D_old(float c0, float c1, float *A0,float *Anext, int nx, int ny, int nz) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	int j = blockIdx.y*blockDim.y+threadIdx.y;
	const int sh_id=threadIdx.x+threadIdx.y*blockDim.x;
	extern __shared__ float sh_A0[];
	sh_A0[sh_id]=0.0f;
	__syncthreads();

	bool w_region =  i>0 && j>0 &&(i<nx-1) &&(j<ny-1);
	bool x_l_bound = (threadIdx.x==0);
	bool x_h_bound = (threadIdx.x==(blockDim.x-1));
	bool y_l_bound = (threadIdx.y==0);
	bool y_h_bound = (threadIdx.y==(blockDim.y-1));
	
	for(int k=1;k<nz-1;k++) {
		sh_A0[sh_id] = A0[Index3D(nx, ny, i, j, k)];
		__syncthreads();
		if(w_region) {
			float partial=A0[Index3D (nx, ny, i, j, k+1)] + A0[Index3D(nx, ny, i, j, k-1)];
			float a_left,a_right,a_top,a_down;
			if(x_l_bound) a_left=A0[Index3D(nx, ny, i-1, j, k)];
			else a_left=sh_A0[sh_id-1];
			if(x_h_bound) a_right=A0[Index3D(nx, ny, i+1, j, k)];
			else a_right=sh_A0[sh_id+1];
			if(y_l_bound) a_down=A0[Index3D(nx, ny, i, j-1, k)];
			else a_down=sh_A0[sh_id-blockDim.x];
			if(y_h_bound) a_top=A0[Index3D(nx, ny, i, j+1, k)];
			else a_top=sh_A0[sh_id+blockDim.x];
			Anext[Index3D(nx, ny, i, j, k)] = (a_left+a_right+a_top+a_down+partial)*c1-sh_A0[sh_id]*c0;
		}
		__syncthreads();
	}
}
#endif
#ifdef TILE_2D_NEW
__global__ void tile_2D_new(int iter, float c0, float c1, float *A0,float *Anext, int nx, int ny, int nz) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	int j = blockIdx.y*blockDim.y+threadIdx.y;
	const int sh_id=threadIdx.x+threadIdx.y*blockDim.x;
	extern __shared__ float sh_A0[];
	float* sh_Anext=sh_A0+blockDim.x*blockDim.y;
	sh_A0[sh_id]=0.0f;
	sh_Anext[sh_id]=0.0f;
	__syncthreads();

	bool w_region =  i>0 && j>0 &&(i<nx-1) &&(j<ny-1);
	bool x_l_bound = (threadIdx.x==0);
	bool x_h_bound = (threadIdx.x==(blockDim.x-1));
	bool y_l_bound = (threadIdx.y==0);
	bool y_h_bound = (threadIdx.y==(blockDim.y-1));
	
	for(int t=0;t<iter;t++) {
		for(int k=1;k<nz-1;k++) {
			sh_A0[sh_id] = A0[Index3D(nx, ny, i, j, k)];
			__syncthreads();
			if(w_region) {
				float a_left, a_right, a_front, a_back, a_top_down;
				a_top_down=A0[Index3D(nx, ny, i, j, k+1)]+A0[Index3D(nx, ny, i, j, k-1)];
				if(x_l_bound) a_left=A0[Index3D(nx, ny, i-1, j, k)];
				else a_left=sh_A0[sh_id-1];
				if(x_h_bound) a_right=A0[Index3D(nx, ny, i+1, j, k)];
				else a_right=sh_A0[sh_id+1];
				if(y_l_bound) a_back=A0[Index3D(nx, ny, i, j-1, k)];
				else a_back=sh_A0[sh_id-blockDim.x];
				if(y_h_bound) a_front=A0[Index3D(nx, ny, i, j+1, k)];
				else a_front=sh_A0[sh_id+blockDim.x];
				sh_Anext[Index3D(nx, ny, i, j, k)] = (a_left+a_right+a_back+a_front+a_top_down)*c1-sh_A0[sh_id]*c0;
			}
			__syncblocks_atomic();
		}
	}
}
#endif
