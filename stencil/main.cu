#include "hip/hip_runtime.h"
//Stencil, IMPACT UIUC
#include <stdio.h>
#include <stdlib.h>
#include "parboil.h"
#include "common.h"
#include "barrier.cu"
#include "kernels.cu"
void stencil_gold(float c0, float c1, float* A0, int iterations, float* result, int nx, int ny, int nz) {
	int size=nx*ny*nz;
	float* input = (float*) malloc( sizeof(float)*size);
	float* output = (float*) malloc( sizeof(float)*size);
	memcpy(input, A0, sizeof(float)*size);
	for(int i=0 ;i<iterations; i++) {
		for(int z=1; z<nz-1; z++) {
			for(int y=1; y<ny-1; y++) {
				for(int x=1; x<nx-1; x++) {
					output[Index3D(nx,ny,x,y,z)] = (input[Index3D(nx,ny,x+1,y,z)] +
						input[Index3D(nx,ny,x-1,y,z)] + input[Index3D(nx,ny,x,y+1,z)] + 
						input[Index3D(nx,ny,x,y-1,z)] + input[Index3D(nx,ny,x,y,z+1)] + 
						input[Index3D(nx,ny,x,y,z-1)])*c1-c0*input[Index3D(nx,ny,x,y,z)];
				}
			}
		}
		float* temp_buff = input;
		input = output;
		output = temp_buff;
	}
	memcpy(result, input, sizeof(float)*size);
	free(input);
	free(output);
}

void print_error(float* real_result, float* gold_result, int nx, int ny, int nz) {
	float error_sum = 0.0f;
	for(int z=1; z<nz-1; z++) {
		for(int y=1; y<ny-1; y++) {
			for(int x=1; x<nx-1; x++) {
				error_sum += abs(real_result[Index3D(nx,ny,x,y,z)] - gold_result[Index3D(nx,ny,x,y,z)]);
			}
		}
	}
	printf("[BENCH] Total error = %f\n", error_sum);
	printf("[BENCH] Average error = %f\n", error_sum/((nz-1)*(ny-1)*(nx-1)));
}

void generateInput(float* input, int nx, int ny, int nz) {
    for(int z=0; z<nz; z++) {
        for(int y=0; y<ny; y++) {
            for(int x=0; x<nx; x++) {
                int index = Index3D(nx, ny, x, y, z);
                // Fit N periods of a sinusoid in each dimension
                const float maxAmp = 5.0f;
                const float N = 2;
                float fx, fy, fz;
                fx = (float) x / (float)nx * 2.0f * PI * N;
                fy = (float) y / (float)ny * 2.0f * PI * N;
                fz = (float) z / (float)nz * 2.0f * PI * N;
                input[index] = maxAmp*sin( sqrt(fx*fx + fy*fy + fz*fz));
            }
        }
    }
}
/*
static void inputData(float *A0, int nx, int ny, int nz, FILE *fp) {
	unsigned size;
	size = nx*ny*nz;
	if(fread(A0,sizeof(float),size,fp) != size)
		printf("Reading error\n");
}
*/
void outputData(char* fName, float *h_A0, int nx, int ny, int nz) {
	FILE* fid = fopen(fName, "w");
	unsigned size;
	if (fid == NULL) {
		fprintf(stderr, "Cannot open output file\n");
		exit(-1);
	}
	size = nx*ny*nz;
	fwrite(&size, sizeof(unsigned), 1, fid);
	fwrite(h_A0, sizeof(float), size, fid);
	fclose (fid);
}

int main(int argc, char** argv) {
#ifdef TIMING
	struct pb_TimerSet timers;
	pb_InitializeTimerSet(&timers);
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
#endif
	struct pb_Parameters *parameters;
	parameters = pb_ReadParameters(&argc, argv);
	int nx=1024, ny=4, nz=4;
	int size;
	int iteration = 10;
	float c0=1.0f/6.0f;
	float c1=1.0f/6.0f/6.0f;
/*
	if (argc<5) {
          printf("Usage: probe nx ny nz tx ty t\n"
	     "nx: the grid size x\n"
	     "ny: the grid size y\n"
	     "nz: the grid size z\n"
	     "t: the iteration time\n");
	  return -1;
	}
	nx = atoi(argv[1]);
	if (nx<1)
		return -1;
	ny = atoi(argv[2]);
	if (ny<1)
		return -1;
	nz = atoi(argv[3]);
	if (nz<1)
		return -1;
	iteration = atoi(argv[4]);
	if(iteration<1)
		return -1;
*/
#ifdef TILE_2D
	printf("[BENCH] CUDA 2D-Tiled Stencil\n");
#endif
#ifdef TILE_3D_NEW
	printf("[BENCH] CUDA 3D-Tiled-new Stencil\n");
#endif
#ifdef TILE_3D_OLD
	printf("[BENCH] CUDA 3D-Tiled-old Stencil\n");
#endif
#ifdef NAIVE
	printf("[BENCH] CUDA Naive Stencil\n");
#endif
	printf("[BENCH] iteration=%d\n", iteration);

	float *h_A0;
	float *h_Anext;
	float *d_A0;
	float *d_Anext;
	size=nx*ny*nz;
	h_A0=(float*)malloc(sizeof(float)*size);
	h_Anext=(float*)malloc(sizeof(float)*size);

#ifdef TIMING
	pb_SwitchToTimer(&timers, pb_TimerID_IO);
#endif
	generateInput(h_A0, nx, ny, nz);
/*
	FILE *fp = fopen(parameters->inpFiles[0], "rb");
	inputData(h_A0, nx, ny, nz, fp);
	fclose(fp);
*/
#ifdef TIMING
	pb_SwitchToTimer(&timers, pb_TimerID_COPY);
#endif

	hipMalloc((void **)&d_A0, size*sizeof(float));
	hipMalloc((void **)&d_Anext, size*sizeof(float));
	hipMemset(d_Anext,0,size*sizeof(float));
	hipMemcpy(d_A0, h_A0, size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Anext, d_A0, size*sizeof(float), hipMemcpyDeviceToDevice);

#ifdef TIMING
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
#endif
	printf("[BENCH] nx=%d, ny=%d, nz=%d\n", nx, ny, nz);

#ifdef TILE_3D
	int tx=BSX;
	int ty=BSY;
	int tz=BSZ;
	int bx=(nx+tx-1)/tx;
	int by=(ny+ty-1)/ty;
	int bz=(nz+tz-1)/tz;
#endif
#ifdef TILE_2D
	int tx=32;
	int ty=4;
	int tz=1;
	int bx=(nx+tx-1)/tx;
	int by=(ny+ty-1)/ty;
	int bz=1;
#endif
#ifdef NAIVE
	int tx=nx-1;
	int ty=1;
	int tz=1;
	int bx=ny-2;
	int by=nz-2;
	int bz=1;
#endif
	printf("[BENCH] bx=%d, by=%d, bz=%d\n", bx, by, bz);
	printf("[BENCH] tx=%d, ty=%d, tz=%d\n", tx, ty, tz);
	dim3 block(tx, ty, tz);
	dim3 grid(bx, by, bz);

#if defined(TILE_3D_OLD) || defined(TILE_2D_OLD) 
	int sh_size = tx*ty*tz*sizeof(float);
	printf("[BENCH] sh_size=%d\n", sh_size);
#endif

	hipDeviceSynchronize();
#ifdef TIMING
	pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);
#endif

#ifdef TILE_3D_NEW
	tile_3D_new<<<grid, block>>>(iteration, c0, c1, d_A0, d_Anext, nx, ny, nz);
#endif
#ifdef TILE_2D_NEW
	tile_2D_new<<<grid, block>>>(iteration, c0, c1, d_A0, d_Anext, nx, ny, nz);
#endif
///*
	for(int t=0;t<iteration;t++) {
#ifdef TILE_3D_OLD
		tile_3D_old<<<grid, block, sh_size>>>(c0, c1, d_A0, d_Anext, nx, ny, nz);
#endif
#ifdef TILE_2D_OLD
		tile_2D_old<<<grid, block, sh_size>>>(c0, c1, d_A0, d_Anext, nx, ny, nz);
#endif
#ifdef NAIVE
		naive<<<grid, block>>>(c0, c1, d_A0, d_Anext, nx, ny, nz);
#endif
		float *d_temp = d_A0;
		d_A0 = d_Anext;
		d_Anext = d_temp;
	}
	float *d_temp = d_A0;
	d_A0 = d_Anext;
	d_Anext = d_temp;
//*/
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernl Launch failed")
#ifdef TIMING
	pb_SwitchToTimer(&timers, pb_TimerID_COPY);
#endif
	hipMemcpy(h_Anext, d_Anext, size*sizeof(float), hipMemcpyDeviceToHost);

	float *h_result_ref = (float*)malloc(sizeof(float)*size);
	stencil_gold(c0, c1, h_A0, iteration, h_result_ref, nx, ny, nz);
	print_error(h_Anext, h_result_ref, nx, ny, nz);
	outputData("result.out", h_Anext, nx, ny, nz);
/*
	if(parameters->outFile) {
#ifdef TIMING
		pb_SwitchToTimer(&timers, pb_TimerID_IO);
#endif
		outputData(parameters->outFile,h_Anext,nx,ny,nz);
	}
*/
#ifdef TIMING
	pb_SwitchToTimer(&timers, pb_TimerID_NONE);
#endif
	hipFree(d_A0);
	hipFree(d_Anext);
	free(h_A0);
	free(h_Anext);
#ifdef TIMING
	pb_PrintTimerSet(&timers);
#endif
	pb_FreeParameters(parameters);
	return 0;
}
