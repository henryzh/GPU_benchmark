#include "hip/hip_runtime.h"
#include "common.h"
#ifdef LOCK_FREE
__device__ inline void __syncblocks_lockfree(int goalVal, volatile int *Arrayin, volatile int *Arrayout) {
	int tx = threadIdx.x;// * blockDim.y + threadIdx.y;
	int numBlocks = gridDim.x;// * gridDim.y;
	int bid = blockIdx.x;// * gridDim.y + blockIdx.y;
	if(tx == 0) {
		Arrayin[bid] = goalVal;
	}
	if(bid == 1) {
		if(tx < numBlocks) {
			while (Arrayin[tx] != goalVal) {}
		}
		__syncthreads();
		if(tx < numBlocks) {
			Arrayout[tx] = goalVal;
		}
	}
	if(tx == 0) {
		while(Arrayout[bid] != goalVal) {}
	}
	__syncthreads();
}
#endif

#ifdef ATOMIC
__device__ volatile int g_mutex = 0;
__device__ inline void __syncblocks_atomic(int goal) {
	__syncthreads();
//	__threadfence();
//	int bx = blockIdx.x;
	int tx = threadIdx.x;// * blockDim.y + threadIdx.y;
	if (tx == 0) {
		atomicAdd((int *)&g_mutex, 1);
		while(g_mutex != goal) {}
	}
	__syncthreads();
}
#endif
