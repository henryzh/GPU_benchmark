#include "hip/hip_runtime.h"
// Parallel bitonic sort using CUDA
// Xuhao Chen, IMPACT group, UIUC
 
#include <stdlib.h>
#include <stdio.h>
#include "parboil.h"
#include "common.h"
#include "barrier.cu"

//#define TEST
//#define DEBUG
//#define PRINT
#define TIMING
//#define SHM
#define SHARED_SIZE_LIMIT 1024

#ifdef TEST
#define THREADS 8
#define BLOCKS 4
#else
#define THREADS 512
#define BLOCKS 32
#endif
#define NUM_VALS 2*THREADS*BLOCKS
struct pb_TimerSet timers;

#define CUERR { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
  printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
  exit(0); }}

void choose_best_device() {
  int num_devices, device;
  hipGetDeviceCount(&num_devices);
  if (num_devices > 1) {
    int max_sm = 0, max_id = 0;
    for (device = 0; device < num_devices; device++) {
      hipDeviceProp_t properties;
      hipGetDeviceProperties(&properties, device);
      if (max_sm < properties.multiProcessorCount) {
        max_sm = properties.multiProcessorCount;
        max_id = device;
      }
    }
    hipSetDevice(max_id);
  }
}

void choose_device() {
  int num_devices, device;
  hipGetDeviceCount(&num_devices);
  if (num_devices > 1) {
    int min_sm = 0, min_id = 0;
    for (device = 0; device < num_devices; device++) {
      hipDeviceProp_t properties;
      hipGetDeviceProperties(&properties, device);
      if (min_sm < properties.multiProcessorCount) {
        min_sm = properties.multiProcessorCount;
        min_id = device;
      }
    }
    hipSetDevice(min_id);
  }
}

int compare (const void * a, const void * b) {
  return ( *(float*)a > *(float*)b );
}

void print_elapsed(clock_t start, clock_t stop) {
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs\n", elapsed);
}

float random_float() {
  return (float)rand()/(float)RAND_MAX;
}

void array_print(float *arr, int length)  {
  int i;
  for (i = 0; i < length; ++i) {
    printf("%1.3f ",  arr[i]);
  }
  printf("\n");
}

void array_fill(float *arr, int length) {
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = length-i;//random_float();
  }
}

void array_copy(float *dst, float *src, int length) {
  int i;
  for (i=0; i<length; ++i) {
    dst[i] = src[i];
  }
}

bool array_compare(float *ref, float *res, int length) {
  int i;
  for (i=0; i<length; ++i) {
    if(ref[i]!=res[i])
      return false;
  }
  return true;
}

#ifdef DEBUG
__device__ void gpu_array_print(float *arr, int length)  {
  int i;
  for (i = 0; i < length; ++i) {
    printf("%1.3f ",  arr[i]);
  }
  printf("\n");
}
#endif
/*
__device__ void comparator_volatile(volatile float &A, volatile float &B, uint dir) {
    float t;
    if ((A > B) == dir) {
        t = A;
        A = B;
        B = t;
    }
}
//*/
__device__ inline void comparator(float &A, float &B, uint dir) {
    float t;
    if ((A > B) == dir) {
        t = A;
        A = B;
        B = t;
    }
}

#ifdef NAIVE
__global__ void bitonicSortNaive(float *src, int stride, int size) {
  uint tid = threadIdx.x + blockDim.x * blockIdx.x;
  uint dir = (tid & (size / 2)) == 0;
  uint pos = 2*tid - (tid & (stride - 1));
  comparator(src[pos], src[pos+stride], dir);
}
#endif

#ifdef LOCK_FREE
__global__ void bitonicSortLockfree(float *src, int length, int *in, int *out) {
  int goalVal = 0;
//  uint barrier_count = 0;

  uint tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint size=2; size<=length; size<<=1) {
    for(uint stride=size>>1; stride>0; stride=stride>>1) {
      uint dir = (tid & (size / 2)) == 0;
      uint pos = 2*tid - (tid & (stride - 1));
      comparator(src[pos], src[pos+stride], dir);
      if(stride>THREADS || (stride==1 && size>=THREADS)) {
//        barrier_count ++;
        __threadfence();
        goalVal ++;
        __syncblocks_lockfree(goalVal, in, out);
      }
      else {
//        comparator_volatile(src[pos], src[pos+stride], dir);
        __syncthreads();
      }
    } // end for stride
  } // end for size
//  if(tid==0)printf("barrier_count=%d\n", barrier_count);
}
#endif

#ifdef ATOMIC
__global__ void bitonicSortAtomic(float *src, int length) {
#ifndef HW_BARRIER
  uint numBlocks = gridDim.x * gridDim.y * gridDim.z;
  uint goalVal = 0;
#endif
//  uint barrier_count = 0;

  uint tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint size=2; size<=length; size<<=1) {
    for(uint stride=size>>1; stride>0; stride=stride>>1) {
      uint dir = (tid & (size / 2)) == 0;
      uint pos = 2*tid - (tid & (stride - 1));
//      comparator_volatile(src[pos], src[pos+stride], dir);
      comparator(src[pos], src[pos+stride], dir);
      if(stride>THREADS || (stride==1 && size>=THREADS)) {
//        barrier_count ++;
        __threadfence();
#ifndef HW_BARRIER
        goalVal += numBlocks;
        __syncblocks_atomic(goalVal);
#endif
      }
      else
        __syncthreads();
    } // end for stride
  } // end for size
//  if(tid==0)printf("barrier_count=%d\n", barrier_count);
}
#endif

__global__ void bitonicSortSmall(float *src, int length, uint dir) {
//    gpu_array_print(src, length);
    int tx = threadIdx.x;
//    int bx = blockIdx.x;
    __shared__ float src_shared[SHARED_SIZE_LIMIT];
    src_shared[tx] = src[tx];
    src_shared[tx + (length / 2)] = src[tx + (length / 2)];

    for (uint size = 2; size < length; size <<= 1) {
        uint ddd = dir ^ ((tx & (size / 2)) != 0);//direction: ascending or descending
        for (uint stride = size/2; stride > 0; stride >>= 1) {
            __syncthreads();
            uint pos = 2 * tx - (tx & (stride - 1));
            comparator(src_shared[pos], src_shared[pos + stride], ddd);
        }
    }
    {
        for (uint stride = length/2; stride > 0; stride >>= 1) {
            __syncthreads();
            uint pos = 2 * tx - (tx & (stride - 1));
            comparator(src_shared[pos + 0], src_shared[pos + stride], dir);
        }
    }
    __syncthreads();
    src[tx] = src_shared[tx];
    src[tx + (length/2)] = src_shared[tx + (length/2)];
}

#ifdef SHM
__global__ void bitonicSortShared(float *src) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int index = blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    __shared__ float src_shared[SHARED_SIZE_LIMIT];
    src_shared[tx] = src[index];
    src_shared[tx + (SHARED_SIZE_LIMIT/2)] = src[index + (SHARED_SIZE_LIMIT/2)];

    for (uint size = 2; size < SHARED_SIZE_LIMIT; size <<= 1) {
        uint ddd = (tx & (size / 2)) == 1;//direction: ascending or descending
        for (uint stride = size/2; stride > 0; stride >>= 1) {
            __syncthreads();
            uint pos = 2 * tx - (tx & (stride - 1));
            comparator(src_shared[pos], src_shared[pos + stride], ddd);
        }
    }
    uint ddd = (bx&1);
//    uint ddd = ((bx&1)==0);
    {
        for (uint stride = SHARED_SIZE_LIMIT/2; stride > 0; stride >>= 1) {
            __syncthreads();
            uint pos = 2 * tx - (tx & (stride - 1));
            comparator(src_shared[pos + 0], src_shared[pos + stride], ddd);
        }
    }
    __syncthreads();
    src[index] = src_shared[tx];
    src[index+(SHARED_SIZE_LIMIT/2)] = src_shared[tx+(SHARED_SIZE_LIMIT/2)];
}

__global__ void bitonicMergeGlobal(float *src, int length, int size, int stride, uint dir) {
  uint tid = threadIdx.x + blockDim.x * blockIdx.x;
  uint comparatorI = tid & (length/2 - 1);
  uint ddd = dir ^ ((comparatorI & (size / 2)) != 0);
  unsigned int pos = 2*tid - (tid & (stride - 1));
//  printf("bx=%d, tx=%d, tid=%d, stride=%d, pos=%d, ddd=%d, size=%d\n", blockIdx.x, threadIdx.x, tid, stride, pos, ddd, size);
  comparator(src[pos], src[pos+stride], ddd);
}
//Map to single instructions on G8x / G9x / G100
#define UMUL(a, b) __umul24((a), (b))
#define UMAD(a, b, c) ( UMUL((a), (b)) + (c) )
__global__ void bitonicMergeShared(float *src, int length, int size, uint dir) {
    __shared__ float src_shm[SHARED_SIZE_LIMIT];
    int index = blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    src_shm[threadIdx.x] = src[index];
    src_shm[threadIdx.x + (SHARED_SIZE_LIMIT/2)] = src[index+(SHARED_SIZE_LIMIT/2)];

    uint comparatorI = UMAD(blockIdx.x, blockDim.x, threadIdx.x) & ((length / 2) - 1);
    uint ddd = dir ^ ((comparatorI & (size / 2)) != 0);
    for (uint stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1) {
        __syncthreads();
        uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
        comparator(src_shm[pos], src_shm[pos + stride], ddd);
    }
    __syncthreads();
    src[index] = src_shm[threadIdx.x];
    src[index+(SHARED_SIZE_LIMIT / 2)] = src_shm[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}
#endif

// Inplace bitonic sort using CUDA
void bitonic_sort(float *values) {
#ifdef DEBUG
  int iteration = 0;
#endif

  float *dev_values;
  size_t size = NUM_VALS * sizeof(float);
  hipMalloc((void**) &dev_values, size);
  CUERR
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);
  CUERR

#ifdef LOCK_FREE
  int *in, *out;
  hipMalloc(&in, BLOCKS*sizeof(int));
  CUERR
  hipMalloc(&out, BLOCKS*sizeof(int));
  CUERR
#endif

  dim3 blocks(BLOCKS,1);
  dim3 threads(THREADS,1);
  hipDeviceSynchronize();

#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);
#endif

  if (NUM_VALS <= SHARED_SIZE_LIMIT) {
    uint blockCount = 1;
    uint threadCount = NUM_VALS / 2;
    printf("[BENCH] Small size, only one block\n");
    printf("[BENCH] blockCount=%d, threadCount=%d\n", blockCount, threadCount);
    bitonicSortSmall<<<blockCount, threadCount>>>(dev_values, NUM_VALS, 1);
    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    CUERR
    hipFree(dev_values);
    CUERR
    return;
  }
#ifndef SHM
  printf("[BENCH] blockCount=%d, threadCount=%d\n", BLOCKS, THREADS);
#endif

#ifdef NAIVE
  int j, k;
  for (k = 2; k <= NUM_VALS; k <<= 1) {
    for (j=k>>1; j>0; j=j>>1) {
//      printf("k=%d, j=%d\n", k, j);
      bitonicSortNaive<<<blocks, threads>>>(dev_values, j, k);
#ifdef DEBUG
      iteration ++;
#endif
    }
  }
  CUERR
#endif

#ifdef LOCK_FREE
  bitonicSortLockfree<<<blocks, threads>>>(dev_values, NUM_VALS, in, out);
  CUERR
#endif

#ifdef ATOMIC
  bitonicSortAtomic<<<blocks, threads>>>(dev_values, NUM_VALS);
  CUERR
#endif

#ifdef SHM
  uint blockCount = NUM_VALS / SHARED_SIZE_LIMIT;
  uint threadCount = SHARED_SIZE_LIMIT / 2;
  printf("blockCount=%d, threadCount=%d, SHARED_SIZE_LIMIT=%d\n", blockCount, threadCount, SHARED_SIZE_LIMIT);
  bitonicSortShared<<<blockCount, threadCount>>>(dev_values);
//  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
//  printf("Inter1 array:\n");
//  array_print(values, NUM_VALS);
  CUERR
  
  for(uint size = 2 * SHARED_SIZE_LIMIT; size <= NUM_VALS; size <<= 1)
    for(unsigned stride = size / 2; stride > 0; stride >>= 1)
      if(stride >= SHARED_SIZE_LIMIT) {
//        printf("bitonicMergeGlobal, blockCount=%d, threadCount=%d\n", NUM_VALS/threadCount, threadCount/2);
        bitonicMergeGlobal<<<NUM_VALS/threadCount, threadCount/2>>>(dev_values, NUM_VALS, size, stride, 1);
//  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
//  printf("Inter2 array:\n");
//  array_print(values, NUM_VALS);
      }
      else {
        bitonicMergeShared<<<blockCount, threadCount>>>(dev_values, NUM_VALS, size, 1);
        break;
      }
  CUERR
#endif

  hipDeviceSynchronize();
#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_COPY);
#endif

#ifdef DEBUG
  printf("iteration=%d\n", iteration);
#endif

  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  CUERR
  hipFree(dev_values);
#ifdef LOCK_FREE
  hipFree(in);
  CUERR
  hipFree(out);
  CUERR
#endif
  CUERR
}

////////////////////////////////////
//           MAIN function        //
////////////////////////////////////
int main(void) {
  printf("[BENCH] Bitonic Sort %d elements\n", NUM_VALS);
  printf("[BENCH] Xuhao Chen <cxh@illinois.edu>\n");
#ifdef SHM
  printf("[BENCH] Shared memory version\n");
#endif
#ifdef NAIVE
  printf("[BENCH] Naive version\n");
#endif
#ifdef LOCK_FREE
  printf("[BENCH] Lock-free Barrier\n");
#endif
#ifndef HW_BARRIER
#ifdef ATOMIC
  printf("[BENCH] Atomic Barrier\n");
#endif
#endif
#ifdef HW_BARRIER
  printf("[BENCH] Hardware Barrier\n");
#endif

#ifdef TIMING
  pb_InitializeTimerSet(&timers);
#endif
  float *values = (float*) malloc( NUM_VALS * sizeof(float));
  float *ref = (float*) malloc( NUM_VALS * sizeof(float));
  array_fill(values, NUM_VALS);
  array_copy(ref, values, NUM_VALS);

  qsort(ref, NUM_VALS, sizeof(float), compare);

#ifdef PRINT
  printf("[BENCH] Input array:\n");
  array_print(values, NUM_VALS);
#endif
#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_COPY);
#endif

  bitonic_sort(values);

#ifdef TIMING
  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);
#endif

  if(array_compare(ref, values, NUM_VALS))
    printf("[BENCH] Pass\n");
  else
    printf("[BENCH] Mismatch\n");

#ifdef PRINT
  printf("[BENCH] Output array:\n");
  array_print(values, NUM_VALS);
//  printf("Referance array3:\n");
//  array_print(ref, NUM_VALS);
#endif
  free(values);
  free(ref);
}
