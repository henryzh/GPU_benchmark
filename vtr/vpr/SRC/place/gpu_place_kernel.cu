#include "hip/hip_runtime.h"
#ifndef _GPU_PLACE_KERNEL_H_
#define _GPU_PLACE_KERNEL_H_

#include <stdio.h>
#include <string.h>
#include "vpr_types.h"
#include <assert.h>
#include "my_cutil.h"

//#define COPY_EVERYTHING_DEVICE_TO_HOST                // copy back everything from device to host, even read only data
//#define GPU_VERBOSE

#define NO_KERNEL_MALLOC                                // For CUDA 3.1 which does not support dynamic memory allocation
#define MAX_PINS_PER_BLOCK 200                          // Used if NO_KERNEL_BLOCK is enabled

#define WARP_SIZE 32

#define d_max(a,b) (((a) > (b))? (a) : (b))
#define d_min(a,b) ((a) > (b)? (b) : (a))

// Copied from VPR place.c
__device__ static const float cross_count[50] = {  /* [0..49] */
    1.0, 1.0, 1.0, 1.0828, 1.1536, 1.2206, 1.2823, 1.3385, 1.3991, 1.4493,
    1.4974, 1.5455, 1.5937, 1.6418, 1.6899, 1.7304, 1.7709, 1.8114, 1.8519,
    1.8924,
    1.9288, 1.9652, 2.0015, 2.0379, 2.0743, 2.1061, 2.1379, 2.1698, 2.2016,
    2.2334,
    2.2646, 2.2958, 2.3271, 2.3583, 2.3895, 2.4187, 2.4479, 2.4772, 2.5064,
    2.5356,
    2.5610, 2.5864, 2.6117, 2.6371, 2.6625, 2.6887, 2.7148, 2.7410, 2.7671,
    2.7933
};

// Struct to hold data for a grid of single type of cells
struct s_g_grid_t {
    int type;
    int num_cells;                // number of cells in this grid
    int* usage;                    // array holding usage for each cell
    int* blocks;                // array holding block index for each cell, array layout is
                                //             [z_index][cell #] (where cell# is in column major order, i.e. 1,1=0; 1,2=1; ...
                                //            the maximum z_index is given by type.capacity
    int* x;                     // store the real x,y location, needed because we skip columns of different types
    int* y;

    int* lock;                   // locks for grid cells

    int nx;                     // # of columns in this grid
    int ny;                     // # of rows in this grid, should be the same as universal ny
};
typedef struct s_g_grid_t g_grid_t;

// Struct to hold mapping from g_grid index back to grid[x][y]
struct s_g_grid_map_t {
    int num_cells;
    int* x;
    int* y;
};
typedef struct s_g_grid_map_t g_grid_map_t;


// Struct to hold block information
// The nets for each block are stored in a separate contiguous array.
// Since the number of nets (pins) per block depends on the type of block,
// we need to store an index into the shared block_nets array.
struct s_g_block_t {
    int type;
    int index_start_net;        // stores the start index for this block's net-list
    int x;
    int y;
    int z;
};
typedef struct s_g_block_t g_block_t;

// Struct to hold net information
// The blocks for each net are stored in a separate shared array
// Since the number of blocks per net are dynamically determined, we need to
// store an index specifying where this net's block array starts
struct s_g_net_t {
    int num_blocks;
    int index_start_block;        // stores the start index for this net's block-list
    //int *node_block;
    boolean is_global;
    boolean is_const_gen;
};
typedef struct s_g_net_t g_net_t;

// Struct to hold all the type information
struct s_g_type_t {
    int index;
    int num_cells;
    int capacity;
    int num_pins;
};
typedef struct s_g_type_t g_type_t;


////////////////////////////////////////
// Declarations of local functions
////////////////////////////////////////
__global__ void gpu_place_kernel(
        g_type_t* d_types, g_grid_t* d_grid, g_block_t* d_block, int* d_block_nets, g_net_t* d_net, float* d_net_cost, int* d_net_blocks,
        int num_blocks, int num_nets, int global_nx, int global_ny,
        unsigned* seeds,
        float** d_chanx_place_cost_fac, float** d_chany_place_cost_fac,
        float t, float rlim, int n_moves_per_warp, int* d_success_sum
        );

void gpu_transform_data_from_vpr (
        int num_types, struct s_type_descriptor * type_descriptors,
        g_type_t** g_type_p,
        struct s_grid_tile** grid, int nx, int ny,
        g_grid_t** g_grid_p, g_grid_map_t* g_grid_map,
        t_block* block, int num_blocks,
        g_block_t** g_block_p, int** g_block_nets_p, int* net_list_size,
        struct s_net* clb_net, int num_nets,
        g_net_t** g_net_p, int** g_net_blocks_p, int* block_list_size
        );

void gpu_transform_data_to_vpr (
        g_type_t* g_type, int num_types,
        struct s_type_descriptor * type_descriptors,
        g_grid_t* g_grid, g_grid_map_t* g_grid_map,
        struct s_grid_tile** grid,
        g_block_t* g_block, int* g_block_nets, int num_blocks,
        t_block* block,
        g_net_t* g_net, int* g_net_blocks, int num_nets,
        struct s_net* clb_net
        );

void gpu_copy_data_to_device (
        g_type_t* g_type, g_grid_t* g_grid, g_block_t* g_block, int* g_block_nets, g_net_t* g_net, float* g_net_cost, int* g_net_blocks,
        g_type_t** d_type_p, g_grid_t** d_grid_p, g_block_t** d_block_p, int** d_block_nets_p, g_net_t** d_net_p, float** d_net_cost_p, int** d_net_blocks_p,
        int num_types, int num_blocks, int num_nets, int net_list_size, int block_list_size,
        float **g_chanx, float **g_chany, float*** d_chanx, float*** d_chany,
        int nx, int ny
        );

void gpu_copy_data_to_host (
        g_type_t* d_type, g_grid_t* d_grid, g_block_t* d_block, int* d_block_nets, g_net_t* d_net, float* d_net_cost, int* d_net_blocks,
        g_type_t* g_type, g_grid_t* g_grid, g_block_t* g_block, int* g_block_nets, g_net_t* g_net, float* g_net_cost, int* g_net_blocks,
        int num_types, int num_blocks, int num_nets, int net_list_size, int block_list_size
        );
////////////////////////////////////////


/*
 * Helper functions for kernel
 */
struct g_bb_t{
    int m_nx_start;
    int m_nx_end;
    int m_ny_start;
    int m_ny_end;
    __device__ g_bb_t(int nx_start, int nx_end, int ny_start, int ny_end)
    : m_nx_start(nx_start), m_nx_end(nx_end), m_ny_start(ny_start), m_ny_end(ny_end) {}
};  // bounding box struct

// LCG psuedo random number generator
__device__ unsigned rand(unsigned* old) {
    *old = (1664525 * *old + 1013904223);
    return *old;
}

// Randomly select two swap canditates
// nx/ny values are the stored locations, not real locations
// bb_from/to give swap-from and swap-to regions
__device__ void select_swap_candidates(
        g_grid_t* d_grid,
        unsigned* seed, float rlim,
        int* nx_from, int* ny_from, int* nx_to, int* ny_to,
        g_bb_t bb_from
        )
{
    // Select from candidate from the given region
    *nx_from = bb_from.m_nx_start + ( rand(seed) % (bb_from.m_nx_end - bb_from.m_nx_start + 1) );
    *ny_from = bb_from.m_ny_start + ( rand(seed) % (bb_from.m_ny_end - bb_from.m_ny_start + 1) );

    // Select to candidate in the rlim region
    const int rlx = d_min(d_grid->nx-1, rlim);
    const int rly = d_min(d_grid->ny-1, rlim);
    const int min_x = d_max(0, *nx_from - rlx);
    const int max_x = d_min(d_grid->nx-1, *nx_from + rlx);
    const int min_y = d_max(0, *ny_from - rly);
    const int max_y = d_min(d_grid->ny-1, *ny_from + rly);

    *nx_to = min_x + ( rand(seed) % (max_x - min_x + 1) );
    *ny_to = min_y + ( rand(seed) % (max_y - min_y + 1) );
}

// Return true is the swap canditates are valid
// Swap is valid is at least one of the canditates is non-empty and there are different candidates
__device__ bool is_swap_valid(
        int nx_from, int ny_from, int nx_to, int ny_to,
        g_grid_t* d_grid, g_block_t* d_block
        )
{
    if(nx_from == nx_to && ny_from == ny_to)
        return false;

    int cell_from_index = nx_from * d_grid->ny + ny_from;
    int cell_to_index = nx_to * d_grid->ny + ny_to;
    int block_from_usage = d_grid->usage[cell_from_index];
    int block_to_usage = d_grid->usage[cell_to_index];

    return (block_from_usage>=1 || block_to_usage>=1);
}

// Do a swap
// Input nx/ny values are the stored locations, not real locations
// Locations must have been validated before being passed into this function
__device__ void do_swap(
        int nx_from, int ny_from, int nx_to, int ny_to,
        g_grid_t* d_grid, g_block_t* d_block
        )
{
    // TODO: handle z-locations
    int cell_from_index = nx_from * d_grid->ny + ny_from;
    int cell_to_index = nx_to * d_grid->ny + ny_to;

    // Undocumented hack: use volatile keyword to enforce coherence for shared writeable locations
    volatile int* blocks = d_grid->blocks;
    volatile int* usage = d_grid->usage;
    int block_from_id = blocks[cell_from_index];
    int block_to_id = blocks[cell_to_index];

    if(block_from_id >= 0) {
        volatile int* block_from_x = &d_block[block_from_id].x;
        volatile int* block_from_y = &d_block[block_from_id].y;
        *block_from_x = d_grid->x[cell_to_index];
        *block_from_y = d_grid->y[cell_to_index];
        usage[cell_from_index]--;
        usage[cell_to_index]++;
    }
    if(block_to_id >= 0) {
        volatile int* block_to_x = &d_block[block_to_id].x;
        volatile int* block_to_y = &d_block[block_to_id].y;
        *block_to_x = d_grid->x[cell_from_index];
        *block_to_y = d_grid->y[cell_from_index];
        usage[cell_from_index]++;
        usage[cell_to_index]--;
    }

    // Update grid
    blocks[cell_from_index] = block_to_id;
    blocks[cell_to_index] = block_from_id;

    /*
    printf("do_swap from:%d,%d to:%d:%d (from:%d,%d to:%d:%d)\n",
            d_grid->x[cell_to_index], d_grid->y[cell_to_index], d_grid->x[cell_from_index], d_grid->y[cell_from_index],
            nx_from, ny_from, nx_to, ny_to
          );
    */
}

// Compute the cost of a net given a bounding box
// Copied from VPR place.c
__device__ float get_net_bb_cost(
        g_net_t* net, g_bb_t bb_p,
        float** d_chanx_place_cost_fac, float** d_chany_place_cost_fac
)
{
    /* Finds the cost due to one net by looking at its coordinate bounding  *
     * box.                                                                 */
    float ncost, crossing;

    /* Get the expected "crossing count" of a net, based on its number *
     * of pins.  Extrapolate for very large nets.                      */
    if(net->num_blocks > 50)
    {
        crossing = 2.7933 + 0.02616 * ((net->num_blocks) - 50);
        /*    crossing = 3.0;    Old value  */
    }
    else
    {
        crossing = cross_count[(net->num_blocks) - 1];
    }

    ncost = (bb_p.m_nx_end - bb_p.m_nx_start + 1) * crossing *
    d_chanx_place_cost_fac[bb_p.m_ny_end][bb_p.m_ny_start - 1];

    ncost += (bb_p.m_ny_end - bb_p.m_ny_start + 1) * crossing *
    d_chany_place_cost_fac[bb_p.m_nx_end][bb_p.m_nx_start - 1];

    return (ncost);
}

// Function to compute the delta_cost for all nets in a given block
// Net ids and new costs are stored in the provided buffer
// valid_nets indicates the head pointer into the provided buffer
__device__ float get_delta_cost_of_block(
        int warp_tid,
        int nx_cell, int ny_cell,
        int* bb_nx_start, int* bb_nx_end, int* bb_ny_start, int* bb_ny_end,
        g_grid_t* d_grid, g_block_t* d_block, int* d_block_nets, g_net_t* d_net, float* d_net_cost, int* d_net_blocks,
        int num_pins,
        int global_nx, int global_ny,
        float** d_chanx_place_cost_fac, float** d_chany_place_cost_fac,
        float* temp_net_costs, int* temp_net_ids, int* valid_nets           // <--- output
        )
{
    // Loop through all the nets for both blocks and compute the new cost of each net
    // Don't build a unique set of nets like VPR does as we don't want to do searches
    // Potential problems:
    //      * redundant computation of shared nets (where the cost doesn't change)
    //      * nets occurring multiple times per block are double counted in cost
    // Also, we want to temporarily store each net's recomputed cost so that we can
    // update it if the move is accepted in the end. I use dynamically allocated global
    // memory here because shared memory usage would scale with num_pins, workload dependent = bad

    // TODO: handle z-locations
    const int cell_index = nx_cell * d_grid->ny + ny_cell;
    const int block_id = d_grid->blocks[cell_index];

    float delta_cost = 0;                     // aggregate delta cost

    // From block
    if(block_id >= 0) {
        const int net_list_start = d_block[block_id].index_start_net;
        /*
        if(warp_tid == 0)
            printf("warp_tid=%d net_list_start=%d num_pins=%d block_id=%d\n",
                    warp_tid, net_list_start, num_pins, block_id);
        */
        for(unsigned n=0; n<num_pins; n++) {
            const int inet = d_block_nets[net_list_start+n];                        // get the number of net
            if(inet >= 0 and !d_net[inet].is_global) {
                // Valid net that we need to compute the cost for
                g_net_t* net = &d_net[inet];                                        // pointer to net for convenience

                // Each thread in warp independently computes its own bounding box
                // We combine the bounding boxes using atomicMin and atomicMax
                const int num_blocks_in_net = net->num_blocks;                       // number of blocks net connects to
                const int block_list_start = net->index_start_block;
                const int n_iterations = (num_blocks_in_net / WARP_SIZE)
                                           + (num_blocks_in_net%WARP_SIZE?1:0);     // number of iterations that we need to do to cover all blocks

                // bounding box local to each thread
                // initialize it such that it doesn't affect min and max operations
                int* local_bb_nx_start = &bb_nx_start[warp_tid];
                int* local_bb_nx_end = &bb_nx_end[warp_tid];
                int* local_bb_ny_start = &bb_ny_start[warp_tid];
                int* local_bb_ny_end = &bb_ny_end[warp_tid];
                *local_bb_nx_start = global_nx+1;
                *local_bb_nx_end = 0;
                *local_bb_ny_start = global_ny+1;
                *local_bb_ny_end = 0;

                for(int b=0; b<n_iterations; b++) {
                    const int block_list_offset = b*WARP_SIZE + warp_tid;
                    if(block_list_offset < num_blocks_in_net) {                     // make sure we don't overflow in the block list
                        const int local_block_num = d_net_blocks[block_list_start+block_list_offset];
                        // Get the block and add it to local bounding box
                        g_block_t* local_block = &d_block[local_block_num];
                        *local_bb_nx_start = d_min(*local_bb_nx_start, local_block->x);
                        *local_bb_nx_end = d_max(*local_bb_nx_end, local_block->x);
                        *local_bb_ny_start = d_min(*local_bb_ny_start, local_block->y);
                        *local_bb_ny_end = d_max(*local_bb_ny_end, local_block->y);
                    }
                }

                // Combine all the bounding boxes using a reduction operation
                // The final result will be at the start of each shared memory array
                int total_warp_threads = WARP_SIZE;
                while(total_warp_threads > 1) {
                    int half_point = (total_warp_threads >> 1);     // divide by two
                    if(warp_tid < half_point) {
                        bb_nx_start[warp_tid] = d_min(bb_nx_start[warp_tid], bb_nx_start[warp_tid+half_point]);
                        bb_nx_end[warp_tid] = d_max(bb_nx_end[warp_tid], bb_nx_end[warp_tid+half_point]);
                        bb_ny_start[warp_tid] = d_min(bb_ny_start[warp_tid], bb_ny_start[warp_tid+half_point]);
                        bb_ny_end[warp_tid] = d_max(bb_ny_end[warp_tid], bb_ny_end[warp_tid+half_point]);
                    }
                    total_warp_threads = total_warp_threads >> 1;
                }

                // Now first thread needs to calculate the new cost and update the delta cost
                if(warp_tid == 0) {
                    // VPR does a final clipping of the bounding box (to remove IO tracks?)
                    const int bb_final_nx_start = d_max(d_min(*bb_nx_start, global_nx), 1);
                    const int bb_final_nx_end = d_max(d_min(*bb_nx_end, global_nx), 1);
                    const int bb_final_ny_start = d_max(d_min(*bb_ny_start, global_ny), 1);
                    const int bb_final_ny_end = d_max(d_min(*bb_ny_end, global_ny), 1);
                    // Store the net cost and id in a buffer
                    temp_net_costs[*valid_nets] = get_net_bb_cost(net, g_bb_t(bb_final_nx_start,bb_final_nx_end,bb_final_ny_start,bb_final_ny_end),
                                                                 d_chanx_place_cost_fac, d_chany_place_cost_fac);
                    temp_net_ids[*valid_nets] = inet;
                    // Compute the delta cost
                    delta_cost += temp_net_costs[*valid_nets] - d_net_cost[inet];
                    /*
                    printf("warp_tid=%d net_id=%5d \tbb_nx:%2d-%2d   bb_ny:%2d-%2d  cost=%f\n", warp_tid, inet,
                            bb_final_nx_start, bb_final_nx_end, bb_final_ny_start, bb_final_ny_end, temp_net_costs[*valid_nets]
                           );
                    */
                }

                *valid_nets += 1;
            }
        }

        return delta_cost;
    } else {
        return 0.0;
    }
}


// Check if move should be accepted
__device__ bool accept_swap(float delta_cost, float t, unsigned* seed) {
    // Always accept good moves
    if(delta_cost <= 0)
        return true;

    // Never accept bad moves at t=0
    if(t<=0)
        return false;

    // Accept bad moves with some probability
    int rand_int = rand(seed);
    float fnum = ((float) rand_int / powf(2,32)) + 0.5;      // float between 0 and 1
    float prob_fac = exp(-delta_cost / t);
    //printf("fnum=%f\n", fnum);
    return prob_fac > fnum;
}



// Dynamic shared memory array
extern __shared__ int dynamicSMEM[];

/*
 * Kernel
 */
__global__ void gpu_place_kernel(
        g_type_t* d_types, g_grid_t* d_grid, g_block_t* d_block, int* d_block_nets, g_net_t* d_net, float* d_net_cost, int* d_net_blocks,
        int num_blocks, int num_nets, int global_nx, int global_ny,
        unsigned* seeds,
        float** d_chanx_place_cost_fac, float** d_chany_place_cost_fac,
        float t, float rlim, int n_moves_per_warp, int* d_success_sum,
        float* d_nomalloc_net_costs, int* d_nomalloc_net_ids
        )
{
    // access thread id
    const int block_tid = threadIdx.x;
    const int block_wid = block_tid / WARP_SIZE;
    const int warp_tid = block_tid % WARP_SIZE;
    const int global_tid = blockIdx.y * (gridDim.x*blockDim.x) +
                           blockIdx.x * blockDim.x +
                           threadIdx.x;
    const int global_wid = global_tid / WARP_SIZE;
    //const int global_bid = blockIdx.y*gridDim.x + blockIdx.x;
    //const int num_thread_blocks = gridDim.x;
    //const int num_threads_per_block = blockDim.x;
    const int n_warps_per_block = blockDim.x/WARP_SIZE + (blockDim.x%WARP_SIZE?1:0);

    //const int nx = d_grid->nx;
    //const int ny = d_grid->ny;

    // Get the seed
    unsigned* seed = &seeds[global_tid];

    // Set up the array that first thread in warp uses to communicate locked cells to rest of the warp
    int* nx_from_array = (int*)dynamicSMEM;
    int* ny_from_array = (int*)&dynamicSMEM[n_warps_per_block];
    int* nx_to_array = (int*)&dynamicSMEM[n_warps_per_block*2];
    int* ny_to_array = (int*)&dynamicSMEM[n_warps_per_block*3];
    int* nx_from = &nx_from_array[block_wid];
    int* ny_from = &ny_from_array[block_wid];
    int* nx_to = &nx_to_array[block_wid];
    int* ny_to = &ny_to_array[block_wid];

    // Set up the array that warps use to combine bounding boxes
    const int bb_smem_start_index = n_warps_per_block*4;
    const int bb_smem_array_size = n_warps_per_block*WARP_SIZE;
    int* bb_nx_start_array = (int*)&dynamicSMEM[bb_smem_start_index];
    int* bb_nx_end_array = (int*)&dynamicSMEM[bb_smem_start_index+bb_smem_array_size*1];
    int* bb_ny_start_array = (int*)&dynamicSMEM[bb_smem_start_index+bb_smem_array_size*2];
    int* bb_ny_end_array = (int*)&dynamicSMEM[bb_smem_start_index+bb_smem_array_size*3];
    int* bb_nx_start = &bb_nx_start_array[block_wid*WARP_SIZE];
    int* bb_nx_end = &bb_nx_end_array[block_wid*WARP_SIZE];
    int* bb_ny_start = &bb_ny_start_array[block_wid*WARP_SIZE];
    int* bb_ny_end = &bb_ny_end_array[block_wid*WARP_SIZE];

    // Dynamically allocate the temporary buffers used for storing temporary net costs
    // Need 2, one for costs and one for net ids. Each is 2*num_pin because of two blocks
    // Only first thread in each warp reads/writes the buffers
    int num_pins = d_types[d_grid->type].num_pins;
    float* temp_net_costs;
    int* temp_net_ids;
    if(warp_tid == 0) {
#ifdef NO_KERNEL_MALLOC
        temp_net_costs = &d_nomalloc_net_costs[global_wid*2*MAX_PINS_PER_BLOCK];
        temp_net_ids = &d_nomalloc_net_ids[global_wid*2*MAX_PINS_PER_BLOCK];
#else
        temp_net_costs = (float*) malloc(2*num_pins*sizeof(float));
        temp_net_ids = (int*) malloc(2*num_pins*sizeof(int));
#endif
    }

    int local_success_sum = 0;                      // Keep track of local success rate

    // Do the swap tries
    for(unsigned i=0; i<n_moves_per_warp; i++) {

        if(warp_tid == 0) {     // only the first thread per warp acquires the lock

            bool lock_acquired = false;

            while(!lock_acquired) {
                // Select two valid locations to swap
                do {
                select_swap_candidates(
                        d_grid,
                        seed, rlim,
                        nx_from, ny_from, nx_to, ny_to,
                        g_bb_t(0,d_grid->nx-1,0,d_grid->ny-1)
                        );
                } while(!is_swap_valid(*nx_from, *ny_from, *nx_to, *ny_to, d_grid, d_block ));


                // Try to acquire locks on both cells and swap
                // If we can't acquire locks, select new candidates instead of retrying the lock
                // we don't want to wait a long time for the lock to be released
                // Note: since only one thread per warp is accessing the locks, we don't have to worry about SIMD stack related issues
                const int cell_from_index = *nx_from * d_grid->ny + *ny_from;
                const int cell_to_index = *nx_to * d_grid->ny + *ny_to;
                int* lock1 = &d_grid->lock[cell_from_index];
                int* lock2 = &d_grid->lock[cell_to_index];
                if(atomicCAS(lock1, 0, 1) == 0) {
                    if(atomicCAS(lock2, 0, 1) == 0) {

                        lock_acquired = true;
                    } else {
                        *lock1 = 0; // release the first lock
                    }
                }
            } // while not lock acquired

        } // first thread per warp only

        //__threadfence_block();  // make sure shared memory values are visible

        //  Do the swap
        if(warp_tid == 0) {
            do_swap(*nx_from, *ny_from, *nx_to, *ny_to,
                    d_grid, d_block);
        }
        //__threadfence_block();  // make sure the swap is visible to all threads in the warp


        int valid_nets = 0;
        float delta_cost = 0.0;

        // Get the delta cost from both the blocks
        // Only the first thread in the warp will get the valid result
        delta_cost += get_delta_cost_of_block(
                            warp_tid,
                            *nx_from, *ny_from,
                            bb_nx_start, bb_nx_end, bb_ny_start, bb_ny_end,
                            d_grid, d_block, d_block_nets, d_net, d_net_cost, d_net_blocks,
                            num_pins,
                            global_nx, global_ny,
                            d_chanx_place_cost_fac, d_chany_place_cost_fac,
                            temp_net_costs, temp_net_ids, &valid_nets
                       );
        delta_cost += get_delta_cost_of_block(
                            warp_tid,
                            *nx_to, *ny_to,
                            bb_nx_start, bb_nx_end, bb_ny_start, bb_ny_end,
                            d_grid, d_block, d_block_nets, d_net, d_net_cost, d_net_blocks,
                            num_pins,
                            global_nx, global_ny,
                            d_chanx_place_cost_fac, d_chany_place_cost_fac,
                            temp_net_costs, temp_net_ids, &valid_nets
                       );

        // If the swap is to be accepted, write out the new net costs
        // If not, call do_swap again to reverse the swap
        if(warp_tid == 0) {
            //printf("global_tid=%d delta_cost=%f\n", global_tid, delta_cost);
            if( accept_swap(delta_cost, t, seed) ) {
                // Swap accepted, write out the new costs
                for(int n=0; n<valid_nets; n++) {
                    const int inet = temp_net_ids[n];
                    d_net_cost[inet] = temp_net_costs[n];
                    //printf("global_tid=%d inet=%5d cost=%f\n", global_tid, inet, temp_net_costs[n]);
                }

                local_success_sum += 1;             // increment local success counter
            } else {
                // Swap rejected, call do_swap to undo it
                do_swap(*nx_from, *ny_from, *nx_to, *ny_to,
                                    d_grid, d_block);
                //__threadfence_block();  // make sure the swap is visible to all threads in the warp
            }
        }

        // Use threadfence to ensure writes are visible to all threads
        // Hardware doesn't require this before shared data is already marked volatile, but simulator needs this
        __threadfence();

        // Release the locks
        if(warp_tid == 0) {
            const int cell_from_index = *nx_from * d_grid->ny + *ny_from;
            const int cell_to_index = *nx_to * d_grid->ny + *ny_to;
            int* lock1 = &d_grid->lock[cell_from_index];
            int* lock2 = &d_grid->lock[cell_to_index];

            *lock1 = 0;
            *lock2 = 0;
        }

    }   // another swap

    // End of block processing
    if(warp_tid == 0) {
        // Atomically add the local success rate
        atomicAdd(d_success_sum, local_success_sum);

        // Free the dynamically allocated buffers
#ifdef NO_KERNEL_MALLOC
#else
        free(temp_net_costs);
        free(temp_net_ids);
#endif
    }

}

/*
 * Interface function between vpr and cuda kernel
 * Copy data structs received from vpr into required internal format
 */

// Host pointers
g_grid_t* g_grid_CLBs = NULL;           // Grid of CLBs
g_block_t* g_block = NULL;              // AOS holding block information
int* g_block_nets = NULL;               // Shared array of nets, each chunk corresponds to one block
g_net_t* g_net = NULL;                  // AOS holding net information
int* g_net_blocks = NULL;               // Shared array of blocks, each chunk corresponds to one net
g_type_t* g_types = NULL;
g_grid_map_t g_grid_map_CLBs;           // Host-only, used for mapping VPR's grid cells to our own grid cells

int net_list_size;                      // total size of shared net-list (used by blocks)
int block_list_size;                    // total size of shared block-list (used by nets)

// Device pointers
g_grid_t* d_grid_CLBs = NULL;
g_block_t* d_block = NULL;
int* d_block_nets = NULL;
g_net_t* d_net = NULL;
int* d_net_blocks = NULL;
float* d_net_cost = NULL;
g_type_t* d_types = NULL;

__constant__ float **d_chanx_place_cost_fac = NULL;
__constant__ float **d_chany_place_cost_fac = NULL;  // net cost factors

unsigned* d_seeds = NULL;
bool seeds_initialized = false;  // we want to initialize the seeds only once

extern "C"
void gpu_place_launch_kernel (
        int nx, int ny,
        int num_blocks, int num_nets,
        float t, float rlim, int* move_lim, int* success_sum
        )
{
    // Set up the grid
    const int n_warps_per_block = 16;        // 16 warps = 512 threads per block
    //const int n_x_nodes_per_block = 256;    // each thread block responsible for n_x x n_y sized grid of nodes
    //const int n_y_nodes_per_block = 256;

    const int grid_dim_x = 16;
    const int grid_dim_y = 1;

    const int n_threads_per_block = n_warps_per_block * WARP_SIZE;
    const int n_total_warps = n_warps_per_block * grid_dim_x * grid_dim_y;

    // Divide up move_lim among all warps, and report back how many moves we actually tried
    printf("\tmove_lim_requested=%d\n",*move_lim);
    // hack to override moves per warp
    // TODO: make this an option
    const int n_moves_per_warp = 1;     //*move_lim / n_total_warps;
    *move_lim = n_moves_per_warp * n_total_warps;


    printf("\tn_warps_per_block=%d\n",n_warps_per_block);
    printf("\tn_threads_per_block=%d\n",n_threads_per_block);
    printf("\tgrid_dim_x=%d\n",grid_dim_x);
    printf("\tgrid_dim_y=%d\n",grid_dim_y);
    printf("\tn_total_warps=%d\n",n_total_warps);
    printf("\tg_grid_CLBs->nx=%d\n",g_grid_CLBs->nx);
    printf("\tg_grid_CLBs->ny=%d\n",g_grid_CLBs->ny);

    printf("\tn_moves_per_warp=%d\n",n_moves_per_warp);


    dim3 dimBlock(n_threads_per_block, 1, 1);
    dim3 dimGrid(grid_dim_x, grid_dim_y, 1);
    int smemSize = 0;
    smemSize += sizeof(int)*n_warps_per_block*4;            // this chunk is for passing nx/ny_from/to values among threads in a warp after acquiring a lock
    smemSize += sizeof(int)*n_warps_per_block*4*WARP_SIZE;  // this chunk is for computing the combined bounding box for each warp, each thread needs 4 words (xmin, xmax, ymin, ymax)

    // Set up the seeds - we do it here because we need to know how many threads there are
    const int n_total_threads = n_threads_per_block*grid_dim_x*grid_dim_y;
    if(!seeds_initialized) {
        int* h_seed = (int*) malloc(n_total_threads * sizeof(int));
        for(int t=0; t<n_total_threads; t++) {
            h_seed[t] = t;
        }
        cutilSafeCall( hipMalloc( (void**) &d_seeds, n_total_threads*sizeof(unsigned) ) );
        cutilSafeCall( hipMemcpy( d_seeds, h_seed, n_total_threads*sizeof(unsigned), hipMemcpyHostToDevice) );
        seeds_initialized = true;
    }

    // Set up device space and copy over:
    //      success_sum
    //      bb_cost
    int* d_success_sum;
    cutilSafeCall( hipMalloc( (void**) &d_success_sum, sizeof(int) ) );
    cutilSafeCall( hipMemcpy( d_success_sum, success_sum, sizeof(int), hipMemcpyHostToDevice) );
    float* d_bb_cost;
    cutilSafeCall( hipMalloc( (void**) &d_bb_cost, sizeof(int) ) );

    // Setup up preallocated device memory if dynamic allocation is disabled
    float* d_nomalloc_net_costs;
    int* d_nomalloc_net_ids;
#ifdef NO_KERNEL_MALLOC
    cutilSafeCall( hipMalloc( (void**) &d_nomalloc_net_costs, n_total_warps*2*MAX_PINS_PER_BLOCK*sizeof(float) ) );
    cutilSafeCall( hipMalloc( (void**) &d_nomalloc_net_ids, n_total_warps*2*MAX_PINS_PER_BLOCK*sizeof(int) ) );
#endif

    // LAUNCH KERNEL HERE
#ifdef GPU_VERBOSE
    printf("GPU: RUNNING KERNEL...");
#endif
    gpu_place_kernel<<< dimGrid, dimBlock, smemSize >>>(
            d_types, d_grid_CLBs, d_block, d_block_nets, d_net, d_net_cost, d_net_blocks,
            num_blocks, num_nets, nx, ny,
            d_seeds,
            d_chanx_place_cost_fac, d_chany_place_cost_fac,
            t, rlim, n_moves_per_warp, d_success_sum,
            d_nomalloc_net_costs, d_nomalloc_net_ids
            );
#ifdef GPU_VERBOSE
    printf(" done\n");
#endif

    // Copy back
    //      success_sum
    //      bb_cost
    cutilSafeCall( hipMemcpy( success_sum, d_success_sum, sizeof(int), hipMemcpyDeviceToHost) );
    hipFree(d_success_sum);

#ifdef NO_KERNEL_MALLOC
    hipFree(d_nomalloc_net_costs);
    hipFree(d_nomalloc_net_ids);
#endif

#ifdef GPU_VERBOSE
    printf("success_sum=%d\n", *success_sum);
#endif

}


// This function updates the net costs on gpu
// This is needed because VPR may recompute the net costs
// This assumes that d_net_cost has been already allocated
extern "C"
void gpu_update_net_costs_on_gpu(
        float* net_cost, int num_nets
        )
{
    // Copy data to device
    cutilSafeCall( hipMemcpy( d_net_cost, net_cost, num_nets*sizeof(float), hipMemcpyHostToDevice) );
}

// This function is to copy only the net_costs from gpu back to VPR
// This is used to recompute the cost on CPU
// Note that net_costs will be inaccurate due to races
extern "C"
void gpu_update_net_costs_on_vpr(
        float* net_cost, int num_nets
        )
{
    // Copy data to host
    cutilSafeCall( hipMemcpy( net_cost, d_net_cost, num_nets*sizeof(float), hipMemcpyDeviceToHost) );
}

extern "C"
void gpu_place_copy_data_to_gpu(
        int num_types, struct s_type_descriptor * type_descriptors,
        struct s_grid_tile** grid, int nx, int ny,
        t_block* block, int num_blocks,
        struct s_net* clb_net, float* net_cost, int num_nets,
        float **chanx_place_cost_fac, float **chany_place_cost_fac
        )
{
    // Copy data from VPR into internal format
    gpu_transform_data_from_vpr(num_types, type_descriptors,
                                &g_types,
                                grid, nx, ny,
                                &g_grid_CLBs, &g_grid_map_CLBs,
                                block, num_blocks,
                                &g_block, &g_block_nets, &net_list_size,
                                clb_net, num_nets,
                                &g_net, &g_net_blocks, &block_list_size
                                );


    gpu_copy_data_to_device(g_types, g_grid_CLBs, g_block, g_block_nets, g_net, net_cost, g_net_blocks,
                            &d_types, &d_grid_CLBs, &d_block, &d_block_nets, &d_net, &d_net_cost, &d_net_blocks,
                            num_types, num_blocks, num_nets, net_list_size, block_list_size,
                            chanx_place_cost_fac, chany_place_cost_fac, &d_chanx_place_cost_fac, &d_chany_place_cost_fac,
                            nx, ny);
}

extern "C"
void gpu_place_copy_data_to_vpr(
        int num_types, struct s_type_descriptor * type_descriptors,
        struct s_grid_tile** grid, int nx, int ny,
        t_block* block, int num_blocks,
        struct s_net* clb_net, float* net_cost, int num_nets
        )
{
    gpu_copy_data_to_host(d_types, d_grid_CLBs, d_block, d_block_nets, d_net, d_net_cost, d_net_blocks,
                          g_types, g_grid_CLBs, g_block, g_block_nets, g_net, net_cost, g_net_blocks,
                          num_types, num_blocks, num_nets, net_list_size, block_list_size);

    // Copy data back to vpr
    gpu_transform_data_to_vpr(g_types, num_types,
                              type_descriptors,
                              g_grid_CLBs, &g_grid_map_CLBs,
                              grid,
                              g_block, g_block_nets, num_blocks,
                              block,
                              g_net, g_net_blocks, num_nets,
                              clb_net
                              );
}


/*
 * Copy vpr data into required representation
 *
 * Assumptions:
 *     1. IO blocks are on the outside of grid
 *     2. Each column can only contain blocks on 1 type, plus empty blocks
 *     3. CLB columns don't have empty blocks (at least not on the periphery)
 */
void gpu_transform_data_from_vpr (
        int num_types, struct s_type_descriptor * type_descriptors,
        g_type_t** g_type_p,
        struct s_grid_tile** grid, int nx, int ny,
        g_grid_t** g_grid_p, g_grid_map_t* g_grid_map,
        t_block* block, int num_blocks,
        g_block_t** g_block_p, int** g_block_nets_p, int* net_list_size,
        struct s_net* clb_net, int num_nets,
        g_net_t** g_net_p, int** g_net_blocks_p, int* block_list_size
        )
{

#ifdef GPU_VERBOSE
    printf("GPU: Copy data from VPR to interface\n");
#endif

    // Copy all the types
    *g_type_p = (g_type_t*) malloc(num_types*sizeof(g_type_t));
    g_type_t* g_type = *g_type_p;
    for(int i=0; i<num_types; i++) {
        g_type[i].index = i;
        g_type[i].num_cells = type_descriptors[i].num_instances_type;
        g_type[i].capacity = type_descriptors[i].capacity;
        g_type[i].num_pins = type_descriptors[i].num_pins;
    }

    // Copy all the CLBs into grid
    // TODO: For now, use the string to find clbs

    // Find CLB info and allocate space for copying
    *g_grid_p = (g_grid_t*) malloc(sizeof(g_grid_t));
    g_grid_t* g_grid = *g_grid_p;
    for(int i = 0; i < num_types; i++)
    {
        if(strcmp(type_descriptors[i].name,"clb")==0) {
            // Found clb, allocate members of g_grid
            g_grid->type = g_type[i].index;
            g_grid->num_cells = g_type[g_grid->type].num_cells;
            g_grid->usage = (int*) malloc(g_grid->num_cells * sizeof(int));
            g_grid->blocks = (int*) malloc(g_grid->num_cells * g_type[g_grid->type].capacity * sizeof(int));
            g_grid->x = (int*) malloc(g_grid->num_cells * sizeof(int));
            g_grid->y = (int*) malloc(g_grid->num_cells * sizeof(int));
            g_grid->lock = (int*) calloc(g_grid->num_cells, sizeof(int));

            g_grid_map->num_cells = g_grid->num_cells;
            g_grid_map->x = (int*) malloc(g_grid->num_cells * sizeof(int));
            g_grid_map->y = (int*) malloc(g_grid->num_cells * sizeof(int));
        }
    }
    assert(g_grid->num_cells > 0);

    // Copy the CLBs in
    int cells_copied = 0;
    g_grid->ny = ny;            // # of rows are the same
    g_grid->nx = 0;
    for(int x=1; x<=nx; x++) {
        if(strcmp(grid[x][1].type->name,"clb")==0) {
            // Column belongs to this CLB
            g_grid->nx += 1;
            for(int y=1; y<=ny; y++) {
                // Copy usage
                g_grid->usage[cells_copied] = grid[x][y].usage;

                // Copy each block id
                for(int z=0; z<g_type[g_grid->type].capacity; z++)
                    g_grid->blocks[z*g_grid->num_cells + cells_copied] = grid[x][y].blocks[z];

                // Record the x,y we copied from
                g_grid->x[cells_copied] = x;
                g_grid->y[cells_copied] = y;
                g_grid_map->x[cells_copied] = x;
                g_grid_map->y[cells_copied] = y;

                cells_copied++;
            }
        }
    }
    assert(cells_copied == g_grid->num_cells);
    assert(g_grid->nx*g_grid->ny == cells_copied);


    // Copy the blocks in
    // Since we have a single array for the blocks' net-list, let's count the total number of nets all the blocks need to store
    *net_list_size = 0;
    for(int b=0; b<num_blocks; b++) {
        *net_list_size += block[b].type->num_pins;
    }

    *g_block_p = (g_block_t*) malloc(num_blocks * sizeof(g_block_t));
    g_block_t* g_block = *g_block_p;
    *g_block_nets_p = (int*) malloc(*net_list_size * sizeof(int));
    int* g_block_nets = *g_block_nets_p;

    int block_net_list_index = 0;
    for(int b=0; b<num_blocks; b++) {
        g_block[b].x = block[b].x;
        g_block[b].y = block[b].y;
        g_block[b].z = block[b].z;
        g_block[b].type = block[b].type->index;
        g_block[b].index_start_net = block_net_list_index;
        for(int n=0; n<g_type[g_block[b].type].num_pins; n++) {
            assert(block_net_list_index < *net_list_size);
            g_block_nets[block_net_list_index] = block[b].nets[n];
            block_net_list_index++;
        }
    }
    assert(block_net_list_index==*net_list_size);

    // Copy nets in
    // Since we have a single array for the nets' block-list, let's count the total number of blocks all the nets need to store
    *block_list_size = 0;
    for(int n=0; n<num_nets; n++)
        // align each net's block_list to WARP_SIZE to ensure coalescing
        *block_list_size += (clb_net[n].num_sinks+1) + (WARP_SIZE - ((clb_net[n].num_sinks+1)%WARP_SIZE));

    // Allocate and fill the net structs and net_block list
    *g_net_p = (g_net_t*) malloc(num_nets * sizeof(g_net_t));
    *g_net_blocks_p = (int*) malloc(*block_list_size * sizeof(int));
    g_net_t* g_net = *g_net_p;
    int* g_net_blocks = *g_net_blocks_p;

    int net_block_list_index = 0;
    for(int n=0; n<num_nets; n++) {
        g_net[n].num_blocks = (clb_net[n].num_sinks+1);
        g_net[n].is_global = clb_net[n].is_global;
        g_net[n].is_const_gen = clb_net[n].is_const_gen;
        g_net[n].index_start_block = net_block_list_index;
        for(int b=0; b<g_net[n].num_blocks; b++) {
            g_net_blocks[net_block_list_index+b] = clb_net[n].node_block[b];
        }
        net_block_list_index += (clb_net[n].num_sinks+1) + (WARP_SIZE - ((clb_net[n].num_sinks+1)%WARP_SIZE));
    }
    assert(net_block_list_index == *block_list_size);
}

/*
 * Copy kernel modified data back to vpr
 * The grid is copied from g_grid to grid using the g_grid_map mapping
 * Blocks g_block and nets g_net storage is simply freed
 */
void gpu_transform_data_to_vpr (
        g_type_t* g_type, int num_types,
        struct s_type_descriptor * type_descriptors,
        g_grid_t* g_grid, g_grid_map_t* g_grid_map,
        struct s_grid_tile** grid,
        g_block_t* g_block, int* g_block_nets, int num_blocks,
        t_block* block,
        g_net_t* g_net, int* g_net_blocks, int num_nets,
        struct s_net* clb_net
        )
{

#ifdef GPU_VERBOSE
    printf("GPU: Copy data from interface to VPR\n");
#endif

    // Copy type data back to VPR
    // Read only data, check instead of copy
    for(int i=0; i<num_types; i++) {
        assert(type_descriptors[i].num_instances_type == g_type[i].num_cells);
        assert(type_descriptors[i].capacity == g_type[i].capacity);
        assert(type_descriptors[i].num_pins == g_type[i].num_pins);
        assert(type_descriptors[i].index == g_type[i].index);
    }

    // Copy grid data back to VPR
    for(int i=0; i<g_grid->num_cells; i++) {
        int x = g_grid_map->x[i];
        int y = g_grid_map->y[i];
        assert(g_grid->x[i] == x);
        assert(g_grid->y[i] == y);
        grid[x][y].usage = g_grid->usage[i];
        for(int z=0; z<g_type[g_grid->type].capacity; z++)
            grid[x][y].blocks[z] = g_grid->blocks[z*g_grid->num_cells + i];
    }

    // Copy block data back to VPR
    for(int b=0; b<num_blocks; b++) {
        block[b].x = g_block[b].x;
        block[b].y = g_block[b].y;
        block[b].z = g_block[b].z;
        assert( block[b].type->index == g_type[g_block[b].type].index);
        int block_net_index = g_block[b].index_start_net;
        for(int n=0; n<g_type[g_block[b].type].num_pins; n++) {
            // Read only data, check instead of copy
            assert(block[b].nets[n] == g_block_nets[block_net_index + n]);
        }
    }

    // Copy net data back to VPR
    // Read-only data, check instead of copy
    for(int n=0; n<num_nets; n++) {
        assert(clb_net[n].num_sinks+1 == g_net[n].num_blocks);
        assert(clb_net[n].is_global == g_net[n].is_global);
        assert(clb_net[n].is_const_gen == g_net[n].is_const_gen);
        int net_block_list_index = g_net[n].index_start_block;
        for(int b=0; b<g_net[n].num_blocks; b++) {
            assert(clb_net[n].node_block[b] == g_net_blocks[net_block_list_index + b]);
        }
    }
}

/*
 * Copy host data to device
 */
void gpu_copy_data_to_device (
        g_type_t* g_type, g_grid_t* g_grid, g_block_t* g_block, int* g_block_nets, g_net_t* g_net, float* g_net_cost, int* g_net_blocks,
        g_type_t** d_type_p, g_grid_t** d_grid_p, g_block_t** d_block_p, int** d_block_nets_p, g_net_t** d_net_p, float** d_net_cost_p, int** d_net_blocks_p,
        int num_types, int num_blocks, int num_nets, int net_list_size, int block_list_size,
        float **g_chanx, float **g_chany, float*** d_chanx, float*** d_chany,
        int nx, int ny
        )
{
#ifdef GPU_VERBOSE
    printf("GPU: Copy data from host to device\n");
#endif

    // Types
    // Simple AOS
    cutilSafeCall( hipMalloc( (void**) d_type_p, num_types*sizeof(g_type_t)) );
    cutilSafeCall( hipMemcpy( *d_type_p, g_type, num_types*sizeof(g_type_t), hipMemcpyHostToDevice) );
    //printf("\t\t ...done copying types\n");


    // Grid - allocate and copy grid struct
    // Since this is a SOA, we have to do the following:
    //        1. Make a temporary copy of the host grid struct on the host
    //        2. Allocate all arrays on the device and update the grid copy to point to device arrays
    //        3. Copy arrays from host to device
    //        4. Allocate grid struct on device and copy host copy to device
    g_grid_t h_grid = *g_grid;        // 1. temporary copy of grid struct on host
    // 2. Allocate on device and update host grid copy
    cutilSafeCall( hipMalloc( (void**) &(h_grid.usage), g_grid->num_cells * sizeof(int)) );
    cutilSafeCall( hipMalloc( (void**) &(h_grid.x), g_grid->num_cells * sizeof(int)) );
    cutilSafeCall( hipMalloc( (void**) &(h_grid.y), g_grid->num_cells * sizeof(int)) );
    cutilSafeCall( hipMalloc( (void**) &(h_grid.blocks), g_grid->num_cells * g_type[g_grid->type].capacity * sizeof(int)) );
    cutilSafeCall( hipMalloc( (void**) &(h_grid.lock), g_grid->num_cells * sizeof(int)) );
    // 3. Copy arrays from host to device
    cutilSafeCall( hipMemcpy( h_grid.usage, g_grid->usage, g_grid->num_cells * sizeof(int), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy( h_grid.x, g_grid->x, g_grid->num_cells * sizeof(int), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy( h_grid.y, g_grid->y, g_grid->num_cells * sizeof(int), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy( h_grid.blocks, g_grid->blocks, g_grid->num_cells * g_type[g_grid->type].capacity * sizeof(int), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy( h_grid.lock, g_grid->lock, g_grid->num_cells * sizeof(int), hipMemcpyHostToDevice) );
    // 4. Allocate grid on device and copy from host to device
    cutilSafeCall( hipMalloc( (void**) d_grid_p, sizeof(g_grid_t)) );
    cutilSafeCall( hipMemcpy( *d_grid_p, &h_grid, sizeof(g_grid_t), hipMemcpyHostToDevice) );
    //printf("\t\t ...done copying grid\n");

    // Blocks and net-list
    // Simple AOS and array
    cutilSafeCall( hipMalloc( (void**) d_block_p, num_blocks*sizeof(g_block_t)) );
    cutilSafeCall( hipMemcpy( *d_block_p, g_block, num_blocks*sizeof(g_block_t), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMalloc( (void**) d_block_nets_p, net_list_size*sizeof(int)) );
    cutilSafeCall( hipMemcpy( *d_block_nets_p, g_block_nets, net_list_size*sizeof(int), hipMemcpyHostToDevice) );
    //printf("\t\t ...done copying blocks\n");

    // Nets and block-list
    // Simple AOS and array
    cutilSafeCall( hipMalloc( (void**) d_net_p, num_nets*sizeof(g_net_t)) );
    cutilSafeCall( hipMemcpy( *d_net_p, g_net, num_nets*sizeof(g_net_t), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMalloc( (void**) d_net_blocks_p, block_list_size*sizeof(int)) );
    cutilSafeCall( hipMemcpy( *d_net_blocks_p, g_net_blocks, block_list_size*sizeof(int), hipMemcpyHostToDevice) );
    //printf("\t\t ...done copying nets\n");
    // Net costs
    cutilSafeCall( hipMalloc( (void**) d_net_cost_p, num_nets*sizeof(float)) );
    cutilSafeCall( hipMemcpy( *d_net_cost_p, g_net_cost, num_nets*sizeof(float), hipMemcpyHostToDevice) );



    // chan_x/y arrays are tricky, they are triangular arrays
    cutilSafeCall( hipMalloc( (void**) d_chanx, (ny+1)*sizeof(float *)) );
    float** h_chanx = (float**) malloc((ny+1)*sizeof(float *));
    for(int i = 0; i <= ny; i++) {
        float* temp;
        cutilSafeCall( hipMalloc( (void**) &temp, (i+1)*sizeof(float)) );
        cutilSafeCall( hipMemcpy( temp, g_chanx[i], (i+1)*sizeof(float), hipMemcpyHostToDevice) );
        h_chanx[i] = temp;
    }
    cutilSafeCall( hipMemcpy( *d_chanx, h_chanx, (ny+1)*sizeof(float *), hipMemcpyHostToDevice) );
    free(h_chanx);

    cutilSafeCall( hipMalloc( (void**) d_chany, (nx+1)*sizeof(float *)) );
    float** h_chany = (float**) malloc((nx+1)*sizeof(float *));
    for(int i = 0; i <= nx; i++) {
        float* temp;
        cutilSafeCall( hipMalloc( (void**) &temp, (i+1)*sizeof(float)) );
        cutilSafeCall( hipMemcpy( temp, g_chany[i], (i+1)*sizeof(float), hipMemcpyHostToDevice) );
        h_chany[i] = temp;
    }
    cutilSafeCall( hipMemcpy( *d_chany, h_chany, (nx+1)*sizeof(float *), hipMemcpyHostToDevice) );
    free(h_chany);

}

/*
 * Copy data from device back to host (into internal structures)
 */
void gpu_copy_data_to_host (
        g_type_t* d_type, g_grid_t* d_grid, g_block_t* d_block, int* d_block_nets, g_net_t* d_net, float* d_net_cost, int* d_net_blocks,
        g_type_t* g_type, g_grid_t* g_grid, g_block_t* g_block, int* g_block_nets, g_net_t* g_net, float* g_net_cost, int* g_net_blocks,
        int num_types, int num_blocks, int num_nets, int net_list_size, int block_list_size
        )
{
    //printf("GPU: Copy data from device to host\n");

    // Types
    #ifdef COPY_EVERYTHING_DEVICE_TO_HOST
        cutilSafeCall( hipMemcpy( g_type, d_type, num_types*sizeof(g_type_t), hipMemcpyDeviceToHost) );
        //printf("\t\t ...done copying types\n");
    #endif

    // Grid
    g_grid_t h_grid;
    cutilSafeCall( hipMemcpy( &h_grid, d_grid, sizeof(g_grid_t), hipMemcpyDeviceToHost) );
    #ifdef COPY_EVERYTHING_DEVICE_TO_HOST
        g_grid->num_cells = h_grid.num_cells;
        cutilSafeCall( hipMemcpy( g_grid->x, h_grid.x, g_grid->num_cells*sizeof(int), hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( g_grid->y, h_grid.y, g_grid->num_cells*sizeof(int), hipMemcpyDeviceToHost) );
    #endif
    cutilSafeCall( hipMemcpy( g_grid->usage, h_grid.usage, g_grid->num_cells*sizeof(int), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy( g_grid->blocks, h_grid.blocks, g_grid->num_cells*g_type[g_grid->type].capacity*sizeof(int), hipMemcpyDeviceToHost) );
    //printf("\t\t ...done copying grid\n");

    // Block
    cutilSafeCall( hipMemcpy( g_block, d_block, num_blocks*sizeof(g_block_t), hipMemcpyDeviceToHost) );
    #ifdef COPY_EVERYTHING_DEVICE_TO_HOST
        cutilSafeCall( hipMemcpy( g_block_nets, d_block_nets, net_list_size*sizeof(int), hipMemcpyDeviceToHost) );
    #endif
        //printf("\t\t ...done copying blocks\n");

    // Nets
    #ifdef COPY_EVERYTHING_DEVICE_TO_HOST
        cutilSafeCall( hipMemcpy( g_net, d_net, num_nets*sizeof(g_net_t), hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy( g_net_blocks, d_net_blocks, block_list_size*sizeof(int), hipMemcpyDeviceToHost) );
        //printf("\t\t ...done copying nets\n");
    #endif
    // Net costs
    cutilSafeCall( hipMemcpy( g_net_cost, d_net_cost, num_nets*sizeof(float), hipMemcpyDeviceToHost) );
}

// Free both the internal representation (g_*) and device allocated (d_*) data
extern "C"
void gpu_free_data() {

#ifdef GPU_VERBOSE
    printf("gpu_free_data\n");
#endif

    // Free internally allocated memory
    // Free every malloc'ed memory
    free(g_block);
    free(g_block_nets);

    free(g_net);
    free(g_net_blocks);

    free(g_grid_CLBs->usage);
    free(g_grid_CLBs->blocks);
    free(g_grid_CLBs->x);
    free(g_grid_CLBs->y);
    free(g_grid_map_CLBs.x);
    free(g_grid_map_CLBs.y);
    free(g_grid_CLBs);

    // Free memory on device
    // Grid
    g_grid_t h_grid_CLBs;
    cutilSafeCall( hipMemcpy( &h_grid_CLBs, d_grid_CLBs, sizeof(g_grid_t), hipMemcpyDeviceToHost) );
    hipFree(h_grid_CLBs.x);
    hipFree(h_grid_CLBs.y);
    hipFree(h_grid_CLBs.usage);
    hipFree(h_grid_CLBs.blocks);

    hipFree(d_types);
    hipFree(d_grid_CLBs);
    hipFree(d_block);
    hipFree(d_block_nets);
    hipFree(d_net);
    hipFree(d_net_cost);
    hipFree(d_net_blocks);
    // TODO: free the chanx/y arrays
}

#endif // #ifndef _GPU_PLACE_KERNEL_H_
