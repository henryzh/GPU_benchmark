#include "hip/hip_runtime.h"
#include "common.h"
#ifdef LOCKFREE
__device__ inline void __syncblocks_lockfree(int goalVal, uint tx, uint bx, uint numBlocks, volatile int *arrayIn, volatile int *arrayOut) {
	if(tx == 0) {
		arrayIn[bx] = goalVal;
	}
	if(bx == 1) {
		if(tx < numBlocks) {
			while (arrayIn[tx] != goalVal) {}
		}
		__syncthreads();
		if(tx < numBlocks) {
			arrayOut[tx] = goalVal;
		}
	}
	if(tx == 0) {
		while(arrayOut[bx] != goalVal) {}
	}
	__syncthreads();
}
#endif

#ifdef ATOMIC
__device__ volatile int mutex = 0;
__device__ inline void __syncblocks_atomic(int goal, uint tx) {
	if (tx == 0) {
		atomicAdd((int *)&mutex, 1);
		while(mutex != goal) {}
	}
	__syncthreads();
}
#endif
