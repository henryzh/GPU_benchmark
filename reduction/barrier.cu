#include "hip/hip_runtime.h"
#include "common.h"
#ifdef LOCKFREE
__device__ inline void __syncblocks_lockfree(int goalVal, volatile int *arrayIn, volatile int *arrayOut) {
//	int tx = threadIdx.x;
	int tx = threadIdx.z*(blockDim.x*blockDim.y) + threadIdx.y*(blockDim.x) + threadIdx.x;
	int numBlocks = gridDim.x;// * gridDim.y;
	int bid = blockIdx.x;// * gridDim.y + blockIdx.y;
	if(tx == 0) {
		arrayIn[bid] = goalVal;
	}
	if(bid == 1) {
		if(tx < numBlocks) {
			while (arrayIn[tx] != goalVal) {}
		}
		__syncthreads();
		if(tx < numBlocks) {
			arrayOut[tx] = goalVal;
		}
	}
	if(tx == 0) {
		while(arrayOut[bid] != goalVal) {}
	}
	__syncthreads();
}
#endif

#ifdef ATOMIC
__device__ volatile int mutex = 0;
__device__ inline void __syncblocks_atomic(int goal) {
	int tx = threadIdx.z*(blockDim.x*blockDim.y) + threadIdx.y*(blockDim.x) + threadIdx.x;
	if (tx == 0) {
		atomicAdd((int *)&mutex, 1);
		while(mutex != goal) {}
	}
	__syncthreads();
}
#endif
