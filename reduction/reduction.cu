#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"
#include "barrier.cu"
#define BLOCK_SIZE 256
#define GRID_SIZE 64
//__device__ unsigned count = 0;
//__shared__ bool isLastBlockDone;

__device__ float calculatePartialSum(const float* input, unsigned N) {
    unsigned tx = threadIdx.x;
    unsigned bs = blockDim.x;
    unsigned shift = bs*blockIdx.x;
    unsigned tid = shift+tx;
    if (tid>=N) return 0;
    __shared__ float partialSum[2*BLOCK_SIZE];
    unsigned start = 2*shift;

    // each thread loads two array elements
    partialSum[tx] = input[start + tx];
    partialSum[bs + tx] = input[start + bs + tx];

    // calculate partial sum
    for (unsigned stride = bs; stride >=1; stride >>=1) {
        __syncthreads();
        if (tx < stride) partialSum[tx] += partialSum[tx + stride];
    }
    return partialSum[0];
}

__device__ float calculateTotalSum(float* input) {
    unsigned tx = threadIdx.x;
    unsigned bs = GRID_SIZE;
    if (tx>=GRID_SIZE/2) return 0;
    __shared__ float partialSum[GRID_SIZE];
    partialSum[tx] = input[tx];
    partialSum[bs/2 + tx] = input[bs/2 + tx];
    for (unsigned stride = bs/2; stride >=1; stride >>=1) {
        __syncthreads();
        if (tx < stride) partialSum[tx] += partialSum[tx + stride];
    }
    return partialSum[0];
}

#ifdef LOCKFREE
__global__ void sum(const float* array, unsigned N, float* result, volatile int* arrayIn, volatile int* arrayOut) {
#else
__global__ void sum(const float* array, unsigned N, float* result) {
#endif
///*
    // Each block sums a subset of the input array
    float partialSum = calculatePartialSum(array, N);
    if (threadIdx.x == 0) {
//        printf("partialSum[%d] = %f\n", blockIdx.x, partialSum);
        // Thread 0 of each block stores the partial sum
        // to global memory
        result[blockIdx.x] = partialSum;

        // Thread 0 makes sure that the threads of the
        // last block will read its correct partial sum
        __threadfence();

        // Thread 0 of each block signals that it is done
//        unsigned value = atomicInc(&count, gridDim.x);

        // Thread 0 of each block determines if its block is
        // the last block to be done
//        isLastBlockDone = (value == (gridDim.x - 1));
    }
    // Synchronize to make sure that each thread reads
    // the correct value of isLastBlockDone
//    __syncthreads();

#ifdef LOCKFREE
    __syncblocks_lockfree(1, arrayIn, arrayOut);
#endif
#ifdef ATOMIC
    __syncblocks_atomic(gridDim.x);
#endif
#ifdef HW_BARRIER
    __threadfence();
#endif

//    if (isLastBlockDone) {
    if (blockIdx.x == 0) {
        // The last block sums the partial sums
        // stored in result[0 .. gridDim.x-1]
        float totalSum = calculateTotalSum(result);
        if (threadIdx.x == 0) {
            // Thread 0 of last block stores total sum
            // to global memory and resets count so that
            // next kernel call works properly
            result[0] = totalSum;
//            printf("totalSum = %f\n", result[0]);
//            count = 0;
        }
    }
//*/
//    if (threadIdx.x == 0)  result[blockIdx.x] = blockIdx.x;
}

int main(int argc, char* argv[]) {
    unsigned n = 2 * BLOCK_SIZE * GRID_SIZE;
    if(argc>1)
        n = atoi(argv[1]);
    if(GRID_SIZE > BLOCK_SIZE) {
        printf("ERROR: GRID_SIZE > BLOCK_SIZE not allowed\n");
        exit(0);
    }
    printf("[BENCH] CUDA Sum, n = %d\n", n);
    printf("[BENCH] Xuhao Chen <cxh@illinois.edu>\n");
    printf("[BENCH] Block Size: %d\n", BLOCK_SIZE);
    printf("[BENCH] Number of Blocks: %d\n", GRID_SIZE);
#ifdef LOCKFREE
    printf("[BENCH] Lock Free Barrier\n");
#endif
#ifdef ATOMIC
    printf("[BENCH] Atomic Barrier\n");
#endif
#ifdef HW_BARRIER
    printf("[BENCH] Hardware Barrier\n");
#endif
    float *h_input;
    float *h_result;
    h_input = (float*)malloc(n*sizeof(float));
    h_result = (float*)calloc(GRID_SIZE, sizeof(float));
    int i;
    for(i = 0; i < n; i++) {
        h_input[i] = ((float) rand() / (RAND_MAX));
//        h_input[i] = 1.0f;
    }
    float *d_input;
    float *d_result;
    CUDA_SAFE_CALL(hipMalloc(&d_input, n*sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc(&d_result, GRID_SIZE*sizeof(float)));
    CUDA_SAFE_CALL(hipMemcpy(d_input, h_input, n*sizeof(float), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemset(d_result, 0, GRID_SIZE*sizeof(float)));

#ifdef LOCKFREE
   int* d_arrayIn;
   int* d_arrayOut;
   CUDA_SAFE_CALL(hipMalloc((void**) &d_arrayIn, sizeof(int)*GRID_SIZE));
   CUDA_SAFE_CALL(hipMalloc((void**) &d_arrayOut, sizeof(int)*GRID_SIZE));
   CUDA_SAFE_CALL(hipMemset((void*) d_arrayIn, 0, sizeof(int)*GRID_SIZE));
   CUDA_SAFE_CALL(hipMemset((void*) d_arrayOut, 0, sizeof(int)*GRID_SIZE));
#endif

    int blockSize = BLOCK_SIZE;
    int gridSize = GRID_SIZE;

    hipDeviceSynchronize();
#ifdef LOCKFREE
    sum<<<gridSize, blockSize>>>(d_input, n, d_result, d_arrayIn, d_arrayOut);
#else
    sum<<<gridSize, blockSize>>>(d_input, n, d_result);
#endif
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Kernel Launch Failed!")

    CUDA_SAFE_CALL(hipMemcpy(h_result, d_result, GRID_SIZE*sizeof(float), hipMemcpyDeviceToHost));
    printf("[BENCH] Final Result: %f\n", h_result[0]);
/*
    for(i = 0; i < GRID_SIZE; i++) {
        printf("h_result[%d] = %f\n", i, h_result[i]);
    }
*/
    float ref_result = 0.0f;
    for(i = 0; i < n; i++) {
        ref_result += h_input[i];
    }
    float error = abs(ref_result - h_result[0]);
    if(error < 0.1)
        printf("[BENCH] Result Correct, error = %f\n", error);
    else
        printf("[BENCH] ERROR (Ref Result = %f)!\n", ref_result);
    CUDA_SAFE_CALL(hipFree(d_input));
    CUDA_SAFE_CALL(hipFree(d_result));
    free(h_input);
    return 0;
}
