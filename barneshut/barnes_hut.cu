#include "hip/hip_runtime.h"
//CUDA BarnesHut v1.1: Simulation of the gravitational forces
//in a galactic cluster using the Barnes-Hut n-body algorithm
//Copyright (c) 2010 The University of Texas at Austin
//Author: Dr. Martin Burtscher

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

// thread count
#define THREADS0 512
#define THREADS1 512
#define THREADS2 288
#define THREADS3 256
#define THREADS4 512
#define THREADS5 384
#define THREADS6 512

// block count = factor * SMs
#define FACTOR0 2
#define FACTOR1 1
#define FACTOR2 2
#define FACTOR3 1
#define FACTOR4 1
#define FACTOR5 2
#define FACTOR6 1

#define WARPSIZE 32
#define MAXDEPTH 26

// input generation
#define MULT 1103515245
#define ADD 12345
#define MASK 0x7FFFFFFF
#define TWOTO31 2147483648.0

#define CUDA_SAFE_CALL_NO_SYNC(call) {                                       \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }

#define CUDA_SAFE_CALL(call) CUDA_SAFE_CALL_NO_SYNC(call);

#define CUT_CHECK_ERROR(errorMessage) {                                      \
    hipDeviceSynchronize();                                                 \
    hipError_t err = hipGetLastError();                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "[CUDA ERROR] %s\n", errorMessage);                  \
        fprintf(stderr, "[CUDA ERROR] in file '%s' in line %i : %s.\n",      \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
  }

/*
static void CudaTest(char *msg) {
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(-1);
  }
}
//*/

static int A = 1;
static int B = 0;
static int randx = 1;
static int lastrand;

static void drndset(int seed) {
   A = 1;
   B = 0;
   randx = (A * seed + B) & MASK;
   A = (MULT * A) & MASK;
   B = (MULT * B + ADD) & MASK;
}

static double drnd() {
   lastrand = randx;
   randx = (A * randx + B) & MASK;
   return (double)lastrand / TWOTO31;
}

// childd is aliased with velxd, velyd, velzd, accxd, accyd, acczd, and sortd but they never use the same memory locations
__constant__ volatile int nnodesd, nbodiesd;
__constant__ volatile float dtimed, dthfd, epssqd, itolsqd;

//__constant__ volatile int *errd;
__device__ volatile int errd = 0;
/*
__device__ volatile int *sortd, *childd, *countd, *startd;
__device__ volatile float *massd, *posxd, *posyd, *poszd, *velxd, *velyd, *velzd, *accxd, *accyd, *acczd;
__device__ volatile float *maxxd, *maxyd, *maxzd, *minxd, *minyd, *minzd;
*/
__device__ volatile int stepd = -1;
__device__ volatile int bottomd;
__device__ volatile int maxdepthd = 1;
__device__ volatile unsigned int blkcntd = 0;
__device__ volatile float radiusd;

// initialize memory
__global__ void InitializationKernel(volatile float *accxd, volatile float *accyd, volatile float *acczd) {
  int i, inc;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  inc = blockDim.x * gridDim.x;
  if (i == 0) {
    printf("[DEBUG] nbodiesd = %d\n", nbodiesd);
    printf("[DEBUG] nnodesd = %d\n", nnodesd);
    printf("[DEBUG] stepd = %d\n", stepd);
    printf("[DEBUG] maxdepthd = %d\n", maxdepthd);
    printf("[DEBUG] blkcntd = %d\n", blkcntd);
    printf("[DEBUG] errd = %d\n", errd);
  }
  for (; i < nbodiesd; i += inc) {
//    accxd[i] = 0.0f;
//    accyd[i] = 0.0f;
//    acczd[i] = 0.0f;
  }
  if (i == 0) {
    printf("[DEBUG] accxd[0] = %d\n", accxd[0]);
    printf("[DEBUG] accyd[0] = %d\n", accyd[0]);
    printf("[DEBUG] acczd[0] = %d\n", acczd[0]);
  }
}

// compute center and radius
__global__ void BoundingBoxKernel(volatile float *posxd, volatile float *posyd, volatile float *poszd, volatile float *maxxd, volatile float *maxyd, volatile float *maxzd, volatile float *minxd, volatile float *minyd, volatile float *minzd, volatile float *massd, volatile int *startd, volatile int *childd) {
  register int i, j, inc;
  register float tmp;
  __shared__ volatile float minx[THREADS1], miny[THREADS1], minz[THREADS1];
  __shared__ volatile float maxx[THREADS1], maxy[THREADS1], maxz[THREADS1];

  i = threadIdx.x;
  if (i == 0) {
    minx[0] = posxd[0];
    miny[0] = posyd[0];
    minz[0] = poszd[0];
  }
  __syncthreads();

  // initialize with valid data (in case #bodies < #threads)
  minx[i] = maxx[i] = minx[0];
  miny[i] = maxy[i] = miny[0];
  minz[i] = maxz[i] = minz[0];

  inc = blockDim.x * gridDim.x;
  j = i + blockIdx.x * blockDim.x;

  // scan bodies
  while (j < nbodiesd) {
    tmp = posxd[j];
    minx[i] = min(minx[i], tmp);
    maxx[i] = max(maxx[i], tmp);

    tmp = posyd[j];
    miny[i] = min(miny[i], tmp);
    maxy[i] = max(maxy[i], tmp);

    tmp = poszd[j];
    minz[i] = min(minz[i], tmp);
    maxz[i] = max(maxz[i], tmp);

    j += inc;  // move on to next body
  }

  // reduction in shared memory
  j = blockDim.x >> 1;
  while (j > 0) {
    __syncthreads();
    if (i < j) {
      minx[i] = min(minx[i], minx[i+j]);
      miny[i] = min(miny[i], miny[i+j]);
      minz[i] = min(minz[i], minz[i+j]);

      maxx[i] = max(maxx[i], maxx[i+j]);
      maxy[i] = max(maxy[i], maxy[i+j]);
      maxz[i] = max(maxz[i], maxz[i+j]);
    }
    j >>= 1;
  }

  if (i == 0) {
    // write block result to global memory
    j = blockIdx.x;
    minxd[j] = minx[0];
    minyd[j] = miny[0];
    minzd[j] = minz[0];

    maxxd[j] = maxx[0];
    maxyd[j] = maxy[0];
    maxzd[j] = maxz[0];
    __threadfence();

    inc = gridDim.x - 1;
    if (inc == atomicInc((unsigned int *)&blkcntd, inc)) {
      // I'm the last block, so combine all block results
      for (j = 0; j <= inc; j++) {
        minx[0] = min(minx[0], minxd[j]);
        miny[0] = min(miny[0], minyd[j]);
        minz[0] = min(minz[0], minzd[j]);

        maxx[0] = max(maxx[0], maxxd[j]);
        maxy[0] = max(maxy[0], maxyd[j]);
        maxz[0] = max(maxz[0], maxzd[j]);
      }

      // compute radius
      tmp = max(maxx[0] - minx[0], maxy[0] - miny[0]);
      radiusd = max(tmp, maxz[0] - minz[0]) * 0.5f;

      // create root node
      j = nnodesd;
      massd[j] = -1.0f;
      startd[j] = 0;
      posxd[j] = (minx[0] + maxx[0]) * 0.5f;
      posyd[j] = (miny[0] + maxy[0]) * 0.5f;
      poszd[j] = (minz[0] + maxz[0]) * 0.5f;
#pragma unroll 8
      for (i = 0; i < 8; i++) childd[j*8+i] = -1;

      bottomd = j;
      stepd++;
    }
  }
}

// build tree
__global__ void TreeBuildingKernel(volatile float *posxd, volatile float *posyd, volatile float *poszd, volatile float *massd, volatile int *startd, volatile int *childd) {
  register int i, j, k, depth, localmaxdepth, skip, inc;
  register float x, y, z, r;
  register float px, py, pz;
  register int ch, n, cell, locked, patch;
  __shared__ float radius, rootx, rooty, rootz;

  i = threadIdx.x;
  if (i == 0) {
    // cache root data
    radius = radiusd;
    rootx = posxd[nnodesd];
    rooty = posyd[nnodesd];
    rootz = poszd[nnodesd];
  }
  __syncthreads();

  localmaxdepth = 1;
  skip = 1;
  inc = blockDim.x * gridDim.x;
  i += blockIdx.x * blockDim.x;

  // iterate over all bodies assigned to thread
  while (i < nbodiesd) {
    if (skip != 0) {
      // new body, so start traversing at root
      skip = 0;
      px = posxd[i];
      py = posyd[i];
      pz = poszd[i];
      n = nnodesd;
      depth = 1;
      r = radius;
      j = 0;
      // determine which child to follow
      if (rootx < px) j = 1;
      if (rooty < py) j += 2;
      if (rootz < pz) j += 4;
    }

    ch = childd[n*8+j];
    // follow path to leaf cell
    while (ch >= nbodiesd) {
      n = ch;
      depth++;
      r *= 0.5f;
      j = 0;
      // determine which child to follow
      if (posxd[n] < px) j = 1;
      if (posyd[n] < py) j += 2;
      if (poszd[n] < pz) j += 4;
      ch = childd[n*8+j];
    }

    if (ch != -2) {  // skip if child pointer is locked and try again later
      locked = n*8+j;
      if (ch == atomicCAS((int *)&childd[locked], ch, -2)) {  // try to lock
        if (ch == -1) {
          // if null, just insert the new body
          childd[locked] = i;
        } else {  // there already is a body in this position
          patch = -1;
          // create new cell(s) and insert the old and new body
          do {
            depth++;

            cell = atomicSub((int *)&bottomd, 1) - 1;
            if (cell <= nbodiesd) {
              errd = 1;
//              *errd = 1;
              bottomd = nnodesd;
            }
            patch = max(patch, cell);

            x = (j & 1) * r;
            y = ((j >> 1) & 1) * r;
            z = ((j >> 2) & 1) * r;
            r *= 0.5f;

            massd[cell] = -1.0f;
            startd[cell] = -1;
            x = posxd[cell] = posxd[n] - r + x;
            y = posyd[cell] = posyd[n] - r + y;
            z = poszd[cell] = poszd[n] - r + z;
#pragma unroll 8
            for (k = 0; k < 8; k++) childd[cell*8+k] = -1;

            if (patch != cell) { 
              childd[n*8+j] = cell;
            }

            j = 0;
            if (x < posxd[ch]) j = 1;
            if (y < posyd[ch]) j += 2;
            if (z < poszd[ch]) j += 4;
            childd[cell*8+j] = ch;

            n = cell;
            j = 0;
            if (x < px) j = 1;
            if (y < py) j += 2;
            if (z < pz) j += 4;

            ch = childd[n*8+j];
            // repeat until the two bodies are different children
          } while (ch >= 0);// end of do-while

          childd[n*8+j] = i;
          __threadfence();
          childd[locked] = patch;
        }// end else

        localmaxdepth = max(depth, localmaxdepth);
        i += inc;  // move on to next body
        skip = 1;
      } // end if (ch == atomicCAS...)
    } // end if (ch != -2)
    __syncthreads();
  } // end while (i < nbodiesd)
  atomicMax((int *)&maxdepthd, localmaxdepth);
}

// compute center of mass
__global__ void SummarizationKernel(volatile float *posxd, volatile float *posyd, volatile float *poszd, volatile float *massd, volatile int *childd, volatile int *countd) {
  register int i, j, k, ch, inc, missing, cnt;
  register float m, cm, px, py, pz;
  __shared__ volatile int bottom, child[THREADS3 * 8];

  if (0 == threadIdx.x) {
    bottom = bottomd;
  }
  __syncthreads();

  inc = blockDim.x * gridDim.x;
  k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
  if (k < bottom) k += inc;

  missing = 0;
  // iterate over all cells assigned to thread
  while (k <= nnodesd) {
    if (missing == 0) {
      // new cell, so initialize
      cm = 0.0f;
      px = 0.0f;
      py = 0.0f;
      pz = 0.0f;
      cnt = 0;
      j = 0;
#pragma unroll 8
      for (i = 0; i < 8; i++) {
        ch = childd[k*8+i];
        if (ch >= 0) {
          if (i != j) {
            // move children to front (needed later for speed)
            childd[k*8+i] = -1;
            childd[k*8+j] = ch;
          }
          child[missing*THREADS3+threadIdx.x] = ch;  // cache missing children
          m = massd[ch];
          missing++;
          if (m >= 0.0f) {
            // child is ready
            missing--;
            if (ch >= nbodiesd) {  // count bodies (needed later)
              cnt += countd[ch] - 1;
            }
            // add child's contribution
            cm += m;
            px += posxd[ch] * m;
            py += posyd[ch] * m;
            pz += poszd[ch] * m;
          }
          j++;
        }
      }
      cnt += j;
    }

    if (missing != 0) {
      do {
        // poll missing child
        ch = child[(missing-1)*THREADS3+threadIdx.x];
        m = massd[ch];
        if (m >= 0.0f) {
          // child is now ready
          missing--;
          if (ch >= nbodiesd) {
            // count bodies (needed later)
            cnt += countd[ch] - 1;
          }
          // add child's contribution
          cm += m;
          px += posxd[ch] * m;
          py += posyd[ch] * m;
          pz += poszd[ch] * m;
        }
        // repeat until we are done or child is not ready
      } while ((m >= 0.0f) && (missing != 0));
    }

    if (missing == 0) {
      // all children are ready, so store computed information
      countd[k] = cnt;
      m = 1.0f / cm;
      posxd[k] = px * m;
      posyd[k] = py * m;
      poszd[k] = pz * m;
      __threadfence();
      massd[k] = cm;
      k += inc;  // move on to next cell
    }
  }
}

// sort bodies
__global__ void SortKernel(volatile int *startd, volatile int *childd, volatile int *countd, volatile int *sortd) {
  register int i, k, ch, dec, start, bottom;
  __shared__ int bottoms;

  if (0 == threadIdx.x) {
    bottoms = bottomd;
  }
  __syncthreads();
  bottom = bottoms;

  dec = blockDim.x * gridDim.x;
  k = nnodesd + 1 - dec + threadIdx.x + blockIdx.x * blockDim.x;

  // iterate over all cells assigned to thread
  while (k >= bottom) {
    start = startd[k];
    if (start >= 0) {
#pragma unroll 8
      for (i = 0; i < 8; i++) {
        ch = childd[k*8+i];
        if (ch >= nbodiesd) {
          // child is a cell
          startd[ch] = start;  // set start ID of child
          start += countd[ch];  // add #bodies in subtree
        } else if (ch >= 0) {
          // child is a body
          sortd[start] = ch;  // record body in sorted array
          start++;
        }
      }
      k -= dec;  // move on to next cell
    }
  }
}

// compute force
__global__ void ForceCalculationKernel(volatile float *posxd, volatile float *posyd, volatile float *poszd, volatile float *accxd, volatile float *accyd, volatile float *acczd, volatile float *velxd, volatile float *velyd, volatile float *velzd, volatile float *massd, volatile int *childd, volatile int *sortd) {
  register int i, j, k, n, depth, base, sbase, diff;
  register float px, py, pz, ax, ay, az, dx, dy, dz, tmp;
  __shared__ int step, maxdepth;
  __shared__ int ch[THREADS5/WARPSIZE];
  __shared__ volatile int pos[MAXDEPTH * THREADS5/WARPSIZE], node[MAXDEPTH * THREADS5/WARPSIZE];
  __shared__ volatile float dq[MAXDEPTH * THREADS5/WARPSIZE];
  __shared__ volatile float nx[THREADS5/WARPSIZE], ny[THREADS5/WARPSIZE], nz[THREADS5/WARPSIZE], nm[THREADS5/WARPSIZE];

  if (0 == threadIdx.x) {
    step = stepd;
    maxdepth = maxdepthd;
    tmp = radiusd;
    // precompute values that depend only on tree level
    dq[0] = tmp * tmp * itolsqd;
    for (i = 1; i < maxdepth; i++) {
      dq[i] = dq[i - 1] * 0.25f;
    }

    if (maxdepth > MAXDEPTH) {
      errd = maxdepth;
//      *errd = maxdepth;
    }
  }
  __syncthreads();

  if (maxdepth <= MAXDEPTH) {
    // figure out first thread in each warp
    base = threadIdx.x / WARPSIZE;
    sbase = base * WARPSIZE;
    j = base * MAXDEPTH;

    diff = threadIdx.x - sbase;
    // make multiple copies to avoid index calculations later
    if (diff < MAXDEPTH) {
      dq[diff+j] = dq[diff];
    }
    __syncthreads();

    // iterate over all bodies assigned to thread
    for (k = threadIdx.x + blockIdx.x * blockDim.x; k < nbodiesd; k += blockDim.x * gridDim.x) {
      i = sortd[k];  // get permuted index
      // cache position info
      px = posxd[i];
      py = posyd[i];
      pz = poszd[i];

      ax = 0.0f;
      ay = 0.0f;
      az = 0.0f;

      // initialize iteration stack, i.e., push root node onto stack
      depth = j;
      if (sbase == threadIdx.x) {
        node[j] = nnodesd;
        pos[j] = 0;
      }
      __threadfence_block();

      while (depth >= j) {
        // stack is not empty
        while (pos[depth] < 8) {
          // node on top of stack has more children to process
          if (sbase == threadIdx.x) {
            // I'm the first thread in the warp
            n = childd[node[depth]*8+pos[depth]];  // load child pointer
            pos[depth]++;
            ch[base] = n;  // cache child pointer
            if (n >= 0) {
              // cache position and mass
              nx[base] = posxd[n];
              ny[base] = posyd[n];
              nz[base] = poszd[n];
              nm[base] = massd[n];
            }
          }
          __threadfence_block();
          // all threads retrieve cached data
          n = ch[base];
          if (n >= 0) {
            dx = nx[base] - px;
            dy = ny[base] - py;
            dz = nz[base] - pz;
            tmp = dx*dx + dy*dy + dz*dz;  // compute distance squared
            if ((n < nbodiesd) || __all(tmp >= dq[depth])) {  // check if all threads agree that cell is far enough away (or is a body)
              if (n != i) {
                tmp = rsqrtf(tmp + epssqd);  // compute distance
                tmp = nm[base] * tmp * tmp * tmp;
                ax += dx * tmp;
                ay += dy * tmp;
                az += dz * tmp;
              }
            } else {
              // push cell onto stack
              depth++;
              if (sbase == threadIdx.x) {
                node[depth] = n;
                pos[depth] = 0;
              }
              __threadfence_block();
            }
          } else {
            depth = max(j, depth - 1);  // early out because all remaining children are also zero
          }
        }
        depth--;  // done with this level
      }

      if (step > 0) {
        velxd[i] += (ax - accxd[i]) * dthfd;
        velyd[i] += (ay - accyd[i]) * dthfd;
        velzd[i] += (az - acczd[i]) * dthfd;
      }

      // save computed acceleration
      accxd[i] = ax;
      accyd[i] = ay;
      acczd[i] = az;
    }
  }
}

// advance bodies
__global__ void IntegrationKernel(volatile float *posxd, volatile float *posyd, volatile float *poszd, volatile float *accxd, volatile float *accyd, volatile float *acczd, volatile float *velxd, volatile float *velyd, volatile float *velzd) {
  register int i, inc;
  register float dvelx, dvely, dvelz;
  register float velhx, velhy, velhz;

  inc = blockDim.x * gridDim.x;
  // iterate over all bodies assigned to thread
  for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc) {
    // integrate
    dvelx = accxd[i] * dthfd;
    dvely = accyd[i] * dthfd;
    dvelz = acczd[i] * dthfd;

    velhx = velxd[i] + dvelx;
    velhy = velyd[i] + dvely;
    velhz = velzd[i] + dvelz;

    posxd[i] += velhx * dtimed;
    posyd[i] += velhy * dtimed;
    poszd[i] += velhz * dtimed;

    velxd[i] = velhx + dvelx;
    velyd[i] = velhy + dvely;
    velzd[i] = velhz + dvelz;
  }
}

/*
static void debugDump() {
   // host copy for fast debug purpose
   int bottomh, maxdepthh, nnodesh, nbodiesh;
   int *childh; 
   float *posxh, *posyh, *poszh;

   hipMemcpyFromSymbol(&bottomh, HIP_SYMBOL(bottomd), sizeof(int));
   hipMemcpyFromSymbol(&nnodesh, HIP_SYMBOL(nnodesd), sizeof(int));
   hipMemcpyFromSymbol(&nbodiesh, HIP_SYMBOL(nbodiesd), sizeof(int));
   hipMemcpyFromSymbol(&maxdepthh, HIP_SYMBOL(maxdepthd), sizeof(int));
   fprintf(stdout, "buttomd = %d\n", bottomh); 
   fprintf(stdout, "nnodesd = %d\n", nnodesh); 
   fprintf(stdout, "nbodiesd = %d\n", nbodiesh); 
   fprintf(stdout, "maxdepthd = %d\n", maxdepthh); 
   childh = (int*) malloc(sizeof(int) * (nnodesh+1) * 8); 
   posxh = (float*) malloc(sizeof(float) * (nnodesh+1)); 
   posyh = (float*) malloc(sizeof(float) * (nnodesh+1)); 
   poszh = (float*) malloc(sizeof(float) * (nnodesh+1)); 
   int *childl;
   int *posxl, *posyl, *poszl; 
   hipMemcpyFromSymbol(&childl, HIP_SYMBOL(childd), sizeof(int*));
   CudaTest("DebugDump obtain childd\n"); 
   hipMemcpyFromSymbol(&posxl, HIP_SYMBOL(posxd), sizeof(int*));
   hipMemcpyFromSymbol(&posyl, HIP_SYMBOL(posyd), sizeof(int*));
   hipMemcpyFromSymbol(&poszl, HIP_SYMBOL(poszd), sizeof(int*));
   hipMemcpy(childh, childl, sizeof(int) * (nnodesh+1) * 8, hipMemcpyDeviceToHost);
   hipMemcpy(posxh, posxl, sizeof(float) * (nnodesh+1), hipMemcpyDeviceToHost);
   hipMemcpy(posyh, posyl, sizeof(float) * (nnodesh+1), hipMemcpyDeviceToHost);
   hipMemcpy(poszh, poszl, sizeof(float) * (nnodesh+1), hipMemcpyDeviceToHost);
   CudaTest("DebugDump copyback\n"); 
   
   // depth first traversal of tree
   fprintf(stdout, "octree in DFS traversal:\n"); 
   int *stack = (int*) malloc(sizeof(int) * (nnodesh+1) * 8); 
   int *treedepth = (int*) malloc(sizeof(int) * (nnodesh+1) * 8); 
   stack[0] = nnodesh; 
   treedepth[0] = 0; 
   int stack_top = 0; 
   while (stack_top >= 0) {
      int node = stack[stack_top]; 
      int depth = treedepth[stack_top]; 
      stack_top--; 

      int nchild = 0; 
      for (int c = 0; c < 8; c++) {
         int childnode = (childh[8 * node + c]); 
         if (childnode >= 0 && childnode < nbodiesh) {
            // print leaf node 
            fprintf(stdout, "   %d %2d %d \n", childnode, depth + 1, c); 
            // fprintf(stdout, "   %d %2d %d:%d %p\n", childnode, depth + 1, node, c, &(childl[8 * node + c])); 
         } else if (childnode != -1) {
            nchild += 1; 
            stack_top++; 
            stack[stack_top] = childnode; 
            treedepth[stack_top] = depth + 1; 
            // fprintf(stdout, " b %d %2d %d:%d %p\n", childnode, depth + 1, node, c, &(childl[8 * node + c])); 
            // if (depth <= 1) 
            //    fprintf(stdout, "treenode: %d %d [%f %f %f]\n", 
            //            childnode, depth + 1, posxh[childnode] ,posyh[childnode], poszh[childnode]); 
         }
      }
   }
   fprintf(stdout, "\n");
   exit(0);
}
*/

int main(int argc, char *argv[]) {
  register int i, run, blocks;
  register int nnodes, nbodies, step, timesteps;
  int error;
  register float dtime, dthf, epssq, itolsq;
  float *mass;
  float *posx, *posy, *posz;
  float *velx, *vely, *velz;
  int *errl;
  int *sortl, *childl, *countl, *startl;
  float *massl;
  float *posxl, *posyl, *poszl;
  float *velxl, *velyl, *velzl;
  float *accxl, *accyl, *acczl;
  float *maxxl, *maxyl, *maxzl;
  float *minxl, *minyl, *minzl;
  register double rsc, vsc, r, v, x, y, z, sq, scale;

  printf("[BENCH] CUDA BarnesHut\n");
  if (argc != 3) {
    fprintf(stderr, "\n");
    fprintf(stderr, "arguments: number_of_bodies number_of_timesteps\n");
    exit(-1);
  }
/*
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    fprintf(stderr, "There is no device supporting CUDA\n");
    exit(-1);
  }
*/
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
/*
  if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {
    fprintf(stderr, "There is no CUDA capable device\n");
    exit(-1);
  }
  if ((deviceProp.major < 1) || ((deviceProp.major == 1) && (deviceProp.minor < 2))) {
    fprintf(stderr, "Need at least compute capability 1.2\n");
    exit(-1);
  }
  if (deviceProp.warpSize != WARPSIZE) {
    fprintf(stderr, "Warp size must be %d\n", deviceProp.warpSize);
    exit(-1);
  }
  if (deviceProp.warpSize != WARPSIZE) {
    fprintf(stderr, "Warp size must be %d\n", deviceProp.warpSize);
    exit(-1);
  }
*/
  blocks = deviceProp.multiProcessorCount;
  printf("[BENCH] blocks = %d\n", blocks);
/*
  if ((WARPSIZE <= 0) || (WARPSIZE & (WARPSIZE-1) != 0)) {
    fprintf(stderr, "Warp size must be greater than zero and a power of two\n");
    exit(-1);
  }
  if (WARPSIZE < MAXDEPTH) {
    fprintf(stderr, "Warp size must be greater than or equal to MAXDEPTH\n");
    exit(-1);
  }
  if ((THREADS0 <= 0) || ((THREADS0 & (WARPSIZE-1)) != 0)) {
    fprintf(stderr, "THREADS0 must be greater than zero and an integer multiple of the warp size\n");
    exit(-1);
  }
  if ((THREADS1 <= 0) || ((THREADS1 & (WARPSIZE-1)) != 0) || ((THREADS1 & (THREADS1-1)) != 0)) {
    fprintf(stderr, "THREADS1 must be greater than zero, an integer multiple of the warp size, and a power of two\n");
    exit(-1);
  }
  if ((THREADS2 <= 0) || ((THREADS2 & (WARPSIZE-1)) != 0)) {
    fprintf(stderr, "THREADS2 must be greater than zero and an integer multiple of the warp size\n");
    exit(-1);
  }
  if ((THREADS3 <= 0) || ((THREADS3 & (WARPSIZE-1)) != 0)) {
    fprintf(stderr, "THREADS3 must be greater than zero and an integer multiple of the warp size\n");
    exit(-1);
  }
  if ((THREADS4 <= 0) || ((THREADS4 & (WARPSIZE-1)) != 0)) {
    fprintf(stderr, "THREADS4 must be greater than zero and an integer multiple of the warp size\n");
    exit(-1);
  }
  if ((THREADS5 <= 0) || ((THREADS5 & (WARPSIZE-1)) != 0)) {
    fprintf(stderr, "THREADS5 must be greater than zero and an integer multiple of the warp size\n");
    exit(-1);
  }
  if ((THREADS6 <= 0) || ((THREADS6 & (WARPSIZE-1)) != 0)) {
    fprintf(stderr, "THREADS6 must be greater than zero and an integer multiple of the warp size\n");
    exit(-1);
  }
*/

  hipGetLastError();  // reset error value
  for (run = 0; run < 1; run++) {
//    for (i = 0; i < 7; i++) timing[i] = 0.0f;
    nbodies = atoi(argv[1]);
    if (nbodies < 1) {
      fprintf(stderr, "nbodies is too small: %d\n", nbodies);
      exit(-1);
    }
    if (nbodies > (1 << 30)) {
      fprintf(stderr, "nbodies is too large: %d\n", nbodies);
      exit(-1);
    }
    nnodes = nbodies * 2;
    if (nnodes < 1024*blocks) nnodes = 1024*blocks;
    while ((nnodes & (WARPSIZE-1)) != 0) nnodes++;
    nnodes--;
    timesteps = atoi(argv[2]);
    dtime = 0.025;
    dthf = dtime * 0.5f;
    epssq = 0.05 * 0.05;
    itolsq = 1.0f / (0.5 * 0.5);

    // allocate memory
    if (run == 0) {
      fprintf(stderr, "[BENCH] nodes = %d\n", nnodes+1);
      fprintf(stderr, "[BENCH] configuration: %d bodies, %d time steps\n", nbodies, timesteps);

      mass = (float *)malloc(sizeof(float) * nbodies);
      if (mass == NULL) {fprintf(stderr, "cannot allocate mass\n");  exit(-1);}
      posx = (float *)malloc(sizeof(float) * nbodies);
      if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
      posy = (float *)malloc(sizeof(float) * nbodies);
      if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
      posz = (float *)malloc(sizeof(float) * nbodies);
      if (posz == NULL) {fprintf(stderr, "cannot allocate posz\n");  exit(-1);}
      velx = (float *)malloc(sizeof(float) * nbodies);
      if (velx == NULL) {fprintf(stderr, "cannot allocate velx\n");  exit(-1);}
      vely = (float *)malloc(sizeof(float) * nbodies);
      if (vely == NULL) {fprintf(stderr, "cannot allocate vely\n");  exit(-1);}
      velz = (float *)malloc(sizeof(float) * nbodies);
      if (velz == NULL) {fprintf(stderr, "cannot allocate velz\n");  exit(-1);}

      int int_size = (nnodes+1)*sizeof(int);
      int flt_size = (nnodes+1)*sizeof(float);
      CUDA_SAFE_CALL(hipMalloc((void **)&errl, sizeof(int)));
      CUDA_SAFE_CALL(hipMalloc((void **)&childl, 8*int_size));
      CUDA_SAFE_CALL(hipMalloc((void **)&massl, flt_size));
      CUDA_SAFE_CALL(hipMalloc((void **)&posxl, flt_size));
      CUDA_SAFE_CALL(hipMalloc((void **)&posyl, flt_size));
      CUDA_SAFE_CALL(hipMalloc((void **)&poszl, flt_size));
      CUDA_SAFE_CALL(hipMalloc((void **)&countl, int_size));
      CUDA_SAFE_CALL(hipMalloc((void **)&startl, int_size));

      hipMemset(childl, 0, 8*int_size);

      // alias arrays
      int inc = (nbodies + WARPSIZE - 1) & (-WARPSIZE);
      velxl = (float *)&childl[0*inc];
      velyl = (float *)&childl[1*inc];
      velzl = (float *)&childl[2*inc];
      accxl = (float *)&childl[3*inc];
      accyl = (float *)&childl[4*inc];
      acczl = (float *)&childl[5*inc];
      sortl = (int *)&childl[6*inc];

      CUDA_SAFE_CALL(hipMalloc((void **)&maxxl, sizeof(float) * blocks));
      CUDA_SAFE_CALL(hipMalloc((void **)&maxyl, sizeof(float) * blocks));
      CUDA_SAFE_CALL(hipMalloc((void **)&maxzl, sizeof(float) * blocks));
      CUDA_SAFE_CALL(hipMalloc((void **)&minxl, sizeof(float) * blocks));
      CUDA_SAFE_CALL(hipMalloc((void **)&minyl, sizeof(float) * blocks));
      CUDA_SAFE_CALL(hipMalloc((void **)&minzl, sizeof(float) * blocks));

      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(nnodesd), &nnodes, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(nbodiesd), &nbodies, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dtimed), &dtime, sizeof(float)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dthfd), &dthf, sizeof(float)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(epssqd), &epssq, sizeof(float)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(itolsqd), &itolsq, sizeof(float)));
//      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(errd), &errl, sizeof(int)));
/*
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sortd), &sortl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(countd), &countl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(startd), &startl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(childd), &childl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(massd), &massl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(posxd), &posxl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(posyd), &posyl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(poszd), &poszl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(velxd), &velxl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(velyd), &velyl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(velzd), &velzl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(accxd), &accxl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(accyd), &accyl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(acczd), &acczl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(maxxd), &maxxl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(maxyd), &maxyl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(maxzd), &maxzl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(minxd), &minxl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(minyd), &minyl, sizeof(int)));
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(minzd), &minzl, sizeof(int)));
*/
    }
    // generate input
    drndset(7);
    rsc = (3 * M_PI) / 16;
    vsc = sqrt(1.0 / rsc);
    for (i = 0; i < nbodies; i++) {
      mass[i] = 1.0 / nbodies;
      r = 1.0 / sqrt(pow(drnd()*0.999, -2.0/3.0) - 1);
      do {
        x = drnd()*2.0 - 1.0;
        y = drnd()*2.0 - 1.0;
        z = drnd()*2.0 - 1.0;
        sq = x*x + y*y + z*z;
      } while (sq > 1.0);
      scale = rsc * r / sqrt(sq);
      posx[i] = x * scale;
      posy[i] = y * scale;
      posz[i] = z * scale;

      do {
        x = drnd();
        y = drnd() * 0.1;
      } while (y > x*x * pow(1 - x*x, 3.5));
      v = x * sqrt(2.0 / sqrt(1 + r*r));
      do {
        x = drnd()*2.0 - 1.0;
        y = drnd()*2.0 - 1.0;
        z = drnd()*2.0 - 1.0;
        sq = x*x + y*y + z*z;
      } while (sq > 1.0);
      scale = vsc * v / sqrt(sq);
      velx[i] = x * scale;
      vely[i] = y * scale;
      velz[i] = z * scale;
    }

    CUDA_SAFE_CALL(hipMemcpy(massl, mass, sizeof(float) * nbodies, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(posxl, posx, sizeof(float) * nbodies, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(posyl, posy, sizeof(float) * nbodies, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(poszl, posz, sizeof(float) * nbodies, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(velxl, velx, sizeof(float) * nbodies, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(velyl, vely, sizeof(float) * nbodies, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(velzl, velz, sizeof(float) * nbodies, hipMemcpyHostToDevice));

    printf("[BENCH] lauch GPU kernels\n");
//    InitializationKernel<<<blocks*FACTOR0, THREADS0>>>();
    CUT_CHECK_ERROR("InitializationKernel execution failed")
//    CudaTest("InitializationKernel ERROR");
    for (step = 0; step < timesteps; step++) {
      BoundingBoxKernel<<<blocks*FACTOR1, THREADS1>>>(posxl, posyl, poszl, maxxl, maxyl, maxzl, minxl, minyl, minzl, massl, startl, childl);
      CUT_CHECK_ERROR("BoundingBoxKernel execution failed")
      TreeBuildingKernel<<<blocks*FACTOR2, THREADS2>>>(posxl, posyl, poszl, massl, startl, childl);
      CUT_CHECK_ERROR("TreeBuildingKernel execution failed")
      SummarizationKernel<<<blocks*FACTOR3, THREADS3>>>(posxl, posyl, poszl, massl, childl, countl);
      CUT_CHECK_ERROR("SummarizationKernel execution failed")
      SortKernel<<<blocks*FACTOR4, 512>>>(startl, childl, countl, sortl);
      CUT_CHECK_ERROR("SortKernel execution failed")
      ForceCalculationKernel<<<blocks*FACTOR5, THREADS5>>>(posxl, posyl, poszl, accxl, accyl, acczl, velxl, velyl, velzl, massl, childl, sortl);
      CUT_CHECK_ERROR("ForceCalculationKernel execution failed")
      IntegrationKernel<<<blocks*FACTOR6, THREADS6>>>(posxl, posyl, poszl, accxl, accyl, acczl, velxl, velyl, velzl);
      CUT_CHECK_ERROR("IntegrationKernel execution failed")
    }
    printf("[BENCH] GPU kernels finished\n");

    // transfer result back to CPU
    CUDA_SAFE_CALL(hipMemcpy(&error, errl, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(posx, posxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(posy, posyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(posz, poszl, sizeof(float) * nbodies, hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(velx, velxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(vely, velyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(velz, velzl, sizeof(float) * nbodies, hipMemcpyDeviceToHost));
  }

  // print output
  FILE * fp = fopen("out.txt", "w");
  for (i = 0; i < nbodies; i++) {
    fprintf(fp, "%.2e %.2e %.2e\n", posx[i], posy[i], posz[i]);
  }
  fclose(fp);
  return 0;
}
