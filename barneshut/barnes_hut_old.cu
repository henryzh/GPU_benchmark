#include "hip/hip_runtime.h"
/*
CUDA BarnesHut v1.1: Simulation of the gravitational forces
in a galactic cluster using the Barnes-Hut n-body algorithm
Copyright (c) 2010 The University of Texas at Austin
Author: Dr. Martin Burtscher
*/


#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>


// thread count
#define THREADS0 512
#define THREADS1 512
#define THREADS2 288
#define THREADS3 256
#define THREADS4 512
#define THREADS5 384
#define THREADS6 512

// block count = factor * SMs
#define FACTOR0 2
#define FACTOR1 1
#define FACTOR2 2
#define FACTOR3 1
#define FACTOR4 1
#define FACTOR5 2
#define FACTOR6 1

#define WARPSIZE 32
#define MAXDEPTH 26


/************************************************************************************/

// input generation

#define MULT 1103515245
#define ADD 12345
#define MASK 0x7FFFFFFF
#define TWOTO31 2147483648.0

static int A = 1;
static int B = 0;
static int randx = 1;
static int lastrand;


static void drndset(int seed)
{
   A = 1;
   B = 0;
   randx = (A * seed + B) & MASK;
   A = (MULT * A) & MASK;
   B = (MULT * B + ADD) & MASK;
}


static double drnd()
{
   lastrand = randx;
   randx = (A * randx + B) & MASK;
   return (double)lastrand / TWOTO31;
}


/************************************************************************************/

// childd is aliased with velxd, velyd, velzd, accxd, accyd, acczd, and sortd but they never use the same memory locations
__constant__ volatile int nnodesd, nbodiesd, *errd, *sortd, *childd, *countd, *startd;
__constant__ volatile float dtimed, dthfd, epssqd, itolsqd;
__constant__ volatile float *massd, *posxd, *posyd, *poszd, *velxd, *velyd, *velzd, *accxd, *accyd, *acczd;
__constant__ volatile float *maxxd, *maxyd, *maxzd, *minxd, *minyd, *minzd;

__device__ volatile int stepd, bottomd, maxdepthd;
__device__ volatile unsigned int blkcntd;
__device__ volatile float radiusd;


/************************************************************************************/
/*** initialize memory **************************************************************/
/************************************************************************************/

__global__ void InitializationKernel() {
  int i, inc;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i == 0) {
    *errd = 0;
    stepd = -1;
    maxdepthd = 1;
    blkcntd = 0;
  }
  inc = blockDim.x * gridDim.x;
  for (; i < nbodiesd; i += inc) {
    accxd[i] = 0.0f;
    accyd[i] = 0.0f;
    acczd[i] = 0.0f;
  }
}


/******************************************************************************/
/*** compute center and radius ************************************************/
/******************************************************************************/
__global__ void BoundingBoxKernel() {
  register int i, j, inc;
  register float tmp;
  __shared__ volatile float minx[THREADS1], miny[THREADS1], minz[THREADS1];
  __shared__ volatile float maxx[THREADS1], maxy[THREADS1], maxz[THREADS1];

  i = threadIdx.x;
  if (i == 0) {
    minx[0] = posxd[0];
    miny[0] = posyd[0];
    minz[0] = poszd[0];
  }
  __syncthreads();

  // initialize with valid data (in case #bodies < #threads)
  minx[i] = maxx[i] = minx[0];
  miny[i] = maxy[i] = miny[0];
  minz[i] = maxz[i] = minz[0];

  inc = blockDim.x * gridDim.x;
  j = i + blockIdx.x * blockDim.x;

  // scan bodies
  while (j < nbodiesd) {
    tmp = posxd[j];
    minx[i] = min(minx[i], tmp);
    maxx[i] = max(maxx[i], tmp);

    tmp = posyd[j];
    miny[i] = min(miny[i], tmp);
    maxy[i] = max(maxy[i], tmp);

    tmp = poszd[j];
    minz[i] = min(minz[i], tmp);
    maxz[i] = max(maxz[i], tmp);

    j += inc;  // move on to next body
  }

  // reduction in shared memory
  j = blockDim.x >> 1;
  while (j > 0) {
    __syncthreads();
    if (i < j) {
      minx[i] = min(minx[i], minx[i+j]);
      miny[i] = min(miny[i], miny[i+j]);
      minz[i] = min(minz[i], minz[i+j]);

      maxx[i] = max(maxx[i], maxx[i+j]);
      maxy[i] = max(maxy[i], maxy[i+j]);
      maxz[i] = max(maxz[i], maxz[i+j]);
    }
    j >>= 1;
  }

  if (i == 0) {
    // write block result to global memory
    j = blockIdx.x;
    minxd[j] = minx[0];
    minyd[j] = miny[0];
    minzd[j] = minz[0];

    maxxd[j] = maxx[0];
    maxyd[j] = maxy[0];
    maxzd[j] = maxz[0];
    __threadfence();

    inc = gridDim.x - 1;
    if (inc == atomicInc((unsigned int *)&blkcntd, inc)) {
      // I'm the last block, so combine all block results
      for (j = 0; j <= inc; j++) {
        minx[0] = min(minx[0], minxd[j]);
        miny[0] = min(miny[0], minyd[j]);
        minz[0] = min(minz[0], minzd[j]);

        maxx[0] = max(maxx[0], maxxd[j]);
        maxy[0] = max(maxy[0], maxyd[j]);
        maxz[0] = max(maxz[0], maxzd[j]);
      }

      // compute radius
      tmp = max(maxx[0] - minx[0], maxy[0] - miny[0]);
      radiusd = max(tmp, maxz[0] - minz[0]) * 0.5f;

      // create root node
      j = nnodesd;
      massd[j] = -1.0f;
      startd[j] = 0;
      posxd[j] = (minx[0] + maxx[0]) * 0.5f;
      posyd[j] = (miny[0] + maxy[0]) * 0.5f;
      poszd[j] = (minz[0] + maxz[0]) * 0.5f;
#pragma unroll 8
      for (i = 0; i < 8; i++) childd[j*8+i] = -1;

      bottomd = j;
      stepd++;
    }
  }
}


/******************************************************************************/
/*** build tree ***************************************************************/
/******************************************************************************/
__global__ void TreeBuildingKernel() {
  register int i, j, k, depth, localmaxdepth, skip, inc;
  register float x, y, z, r;
  register float px, py, pz;
  register int ch, n, cell, locked, patch;
  __shared__ float radius, rootx, rooty, rootz;

  i = threadIdx.x;
  if (i == 0) {
    // cache root data
    radius = radiusd;
    rootx = posxd[nnodesd];
    rooty = posyd[nnodesd];
    rootz = poszd[nnodesd];
  }
  __syncthreads();

  localmaxdepth = 1;
  skip = 1;
  inc = blockDim.x * gridDim.x;
  i += blockIdx.x * blockDim.x;

  // iterate over all bodies assigned to thread
  while (i < nbodiesd) {
    if (skip != 0) {
      // new body, so start traversing at root
      skip = 0;
      px = posxd[i];
      py = posyd[i];
      pz = poszd[i];
      n = nnodesd;
      depth = 1;
      r = radius;
      j = 0;
      // determine which child to follow
      if (rootx < px) j = 1;
      if (rooty < py) j += 2;
      if (rootz < pz) j += 4;
    }

    ch = childd[n*8+j];
    // follow path to leaf cell
    while (ch >= nbodiesd) {
      n = ch;
      depth++;
      r *= 0.5f;
      j = 0;
      // determine which child to follow
      if (posxd[n] < px) j = 1;
      if (posyd[n] < py) j += 2;
      if (poszd[n] < pz) j += 4;
      ch = childd[n*8+j];
    }

    if (ch != -2) {  // skip if child pointer is locked and try again later
      locked = n*8+j;
      if (ch == atomicCAS((int *)&childd[locked], ch, -2)) {  // try to lock
        if (ch == -1) {
          // if null, just insert the new body
          childd[locked] = i;
        } else {  // there already is a body in this position
          patch = -1;
          // create new cell(s) and insert the old and new body
          do {
            depth++;

            cell = atomicSub((int *)&bottomd, 1) - 1;
            if (cell <= nbodiesd) {
              *errd = 1;
              bottomd = nnodesd;
            }
            patch = max(patch, cell);

            x = (j & 1) * r;
            y = ((j >> 1) & 1) * r;
            z = ((j >> 2) & 1) * r;
            r *= 0.5f;

            massd[cell] = -1.0f;
            startd[cell] = -1;
            x = posxd[cell] = posxd[n] - r + x;
            y = posyd[cell] = posyd[n] - r + y;
            z = poszd[cell] = poszd[n] - r + z;
#pragma unroll 8
            for (k = 0; k < 8; k++) childd[cell*8+k] = -1;

            if (patch != cell) { 
              childd[n*8+j] = cell;
            }

            j = 0;
            if (x < posxd[ch]) j = 1;
            if (y < posyd[ch]) j += 2;
            if (z < poszd[ch]) j += 4;
            childd[cell*8+j] = ch;

            n = cell;
            j = 0;
            if (x < px) j = 1;
            if (y < py) j += 2;
            if (z < pz) j += 4;

            ch = childd[n*8+j];
            // repeat until the two bodies are different children
          } while (ch >= 0);
          childd[n*8+j] = i;
          __threadfence();
          childd[locked] = patch;
        }

        localmaxdepth = max(depth, localmaxdepth);
        i += inc;  // move on to next body
        skip = 1;
      }
    }
    __syncthreads();
  }
  atomicMax((int *)&maxdepthd, localmaxdepth);
}


/******************************************************************************/
/*** compute center of mass ***************************************************/
/******************************************************************************/
__global__ void SummarizationKernel() {
  register int i, j, k, ch, inc, missing, cnt;
  register float m, cm, px, py, pz;
  __shared__ volatile int bottom, child[THREADS3 * 8];

  if (0 == threadIdx.x) {
    bottom = bottomd;
  }
  __syncthreads();

  inc = blockDim.x * gridDim.x;
  k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
  if (k < bottom) k += inc;

  missing = 0;
  // iterate over all cells assigned to thread
  while (k <= nnodesd) {
    if (missing == 0) {
      // new cell, so initialize
      cm = 0.0f;
      px = 0.0f;
      py = 0.0f;
      pz = 0.0f;
      cnt = 0;
      j = 0;
#pragma unroll 8
      for (i = 0; i < 8; i++) {
        ch = childd[k*8+i];
        if (ch >= 0) {
          if (i != j) {
            // move children to front (needed later for speed)
            childd[k*8+i] = -1;
            childd[k*8+j] = ch;
          }
          child[missing*THREADS3+threadIdx.x] = ch;  // cache missing children
          m = massd[ch];
          missing++;
          if (m >= 0.0f) {
            // child is ready
            missing--;
            if (ch >= nbodiesd) {  // count bodies (needed later)
              cnt += countd[ch] - 1;
            }
            // add child's contribution
            cm += m;
            px += posxd[ch] * m;
            py += posyd[ch] * m;
            pz += poszd[ch] * m;
          }
          j++;
        }
      }
      cnt += j;
    }

    if (missing != 0) {
      do {
        // poll missing child
        ch = child[(missing-1)*THREADS3+threadIdx.x];
        m = massd[ch];
        if (m >= 0.0f) {
          // child is now ready
          missing--;
          if (ch >= nbodiesd) {
            // count bodies (needed later)
            cnt += countd[ch] - 1;
          }
          // add child's contribution
          cm += m;
          px += posxd[ch] * m;
          py += posyd[ch] * m;
          pz += poszd[ch] * m;
        }
        // repeat until we are done or child is not ready
      } while ((m >= 0.0f) && (missing != 0));
    }

    if (missing == 0) {
      // all children are ready, so store computed information
      countd[k] = cnt;
      m = 1.0f / cm;
      posxd[k] = px * m;
      posyd[k] = py * m;
      poszd[k] = pz * m;
      __threadfence();
      massd[k] = cm;
      k += inc;  // move on to next cell
    }
  }
}


/******************************************************************************/
/*** sort bodies **************************************************************/
/******************************************************************************/
__global__ void SortKernel() {
  register int i, k, ch, dec, start, bottom;
  __shared__ int bottoms;

  if (0 == threadIdx.x) {
    bottoms = bottomd;
  }
  __syncthreads();
  bottom = bottoms;

  dec = blockDim.x * gridDim.x;
  k = nnodesd + 1 - dec + threadIdx.x + blockIdx.x * blockDim.x;

  // iterate over all cells assigned to thread
  while (k >= bottom) {
    start = startd[k];
    if (start >= 0) {
#pragma unroll 8
      for (i = 0; i < 8; i++) {
        ch = childd[k*8+i];
        if (ch >= nbodiesd) {
          // child is a cell
          startd[ch] = start;  // set start ID of child
          start += countd[ch];  // add #bodies in subtree
        } else if (ch >= 0) {
          // child is a body
          sortd[start] = ch;  // record body in sorted array
          start++;
        }
      }
      k -= dec;  // move on to next cell
    }
  }
}


/******************************************************************************/
/*** compute force ************************************************************/
/******************************************************************************/
__global__ void ForceCalculationKernel() {
  register int i, j, k, n, depth, base, sbase, diff;
  register float px, py, pz, ax, ay, az, dx, dy, dz, tmp;
  __shared__ int step, maxdepth;
  __shared__ int ch[THREADS5/WARPSIZE];
  __shared__ volatile int pos[MAXDEPTH * THREADS5/WARPSIZE], node[MAXDEPTH * THREADS5/WARPSIZE];
  __shared__ volatile float dq[MAXDEPTH * THREADS5/WARPSIZE];
  __shared__ volatile float nx[THREADS5/WARPSIZE], ny[THREADS5/WARPSIZE], nz[THREADS5/WARPSIZE], nm[THREADS5/WARPSIZE];

  if (0 == threadIdx.x) {
    step = stepd;
    maxdepth = maxdepthd;
    tmp = radiusd;
    // precompute values that depend only on tree level
    dq[0] = tmp * tmp * itolsqd;
    for (i = 1; i < maxdepth; i++) {
      dq[i] = dq[i - 1] * 0.25f;
    }

    if (maxdepth > MAXDEPTH) {
      *errd = maxdepth;
    }
  }
  __syncthreads();

  if (maxdepth <= MAXDEPTH) {
    // figure out first thread in each warp
    base = threadIdx.x / WARPSIZE;
    sbase = base * WARPSIZE;
    j = base * MAXDEPTH;

    diff = threadIdx.x - sbase;
    // make multiple copies to avoid index calculations later
    if (diff < MAXDEPTH) {
      dq[diff+j] = dq[diff];
    }
    __syncthreads();

    // iterate over all bodies assigned to thread
    for (k = threadIdx.x + blockIdx.x * blockDim.x; k < nbodiesd; k += blockDim.x * gridDim.x) {
      i = sortd[k];  // get permuted index
      // cache position info
      px = posxd[i];
      py = posyd[i];
      pz = poszd[i];

      ax = 0.0f;
      ay = 0.0f;
      az = 0.0f;

      // initialize iteration stack, i.e., push root node onto stack
      depth = j;
      if (sbase == threadIdx.x) {
        node[j] = nnodesd;
        pos[j] = 0;
      }
      __threadfence_block();

      while (depth >= j) {
        // stack is not empty
        while (pos[depth] < 8) {
          // node on top of stack has more children to process
          if (sbase == threadIdx.x) {
            // I'm the first thread in the warp
            n = childd[node[depth]*8+pos[depth]];  // load child pointer
            pos[depth]++;
            ch[base] = n;  // cache child pointer
            if (n >= 0) {
              // cache position and mass
              nx[base] = posxd[n];
              ny[base] = posyd[n];
              nz[base] = poszd[n];
              nm[base] = massd[n];
            }
          }
          __threadfence_block();
          // all threads retrieve cached data
          n = ch[base];
          if (n >= 0) {
            dx = nx[base] - px;
            dy = ny[base] - py;
            dz = nz[base] - pz;
            tmp = dx*dx + dy*dy + dz*dz;  // compute distance squared
            if ((n < nbodiesd) || __all(tmp >= dq[depth])) {  // check if all threads agree that cell is far enough away (or is a body)
              if (n != i) {
                tmp = rsqrtf(tmp + epssqd);  // compute distance
                tmp = nm[base] * tmp * tmp * tmp;
                ax += dx * tmp;
                ay += dy * tmp;
                az += dz * tmp;
              }
            } else {
              // push cell onto stack
              depth++;
              if (sbase == threadIdx.x) {
                node[depth] = n;
                pos[depth] = 0;
              }
              __threadfence_block();
            }
          } else {
            depth = max(j, depth - 1);  // early out because all remaining children are also zero
          }
        }
        depth--;  // done with this level
      }

      if (step > 0) {
        velxd[i] += (ax - accxd[i]) * dthfd;
        velyd[i] += (ay - accyd[i]) * dthfd;
        velzd[i] += (az - acczd[i]) * dthfd;
      }

      // save computed acceleration
      accxd[i] = ax;
      accyd[i] = ay;
      acczd[i] = az;
    }
  }
}


/******************************************************************************/
/*** advance bodies ***********************************************************/
/******************************************************************************/
__global__ void IntegrationKernel() {
  register int i, inc;
  register float dvelx, dvely, dvelz;
  register float velhx, velhy, velhz;

  inc = blockDim.x * gridDim.x;
  // iterate over all bodies assigned to thread
  for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc) {
    // integrate
    dvelx = accxd[i] * dthfd;
    dvely = accyd[i] * dthfd;
    dvelz = acczd[i] * dthfd;

    velhx = velxd[i] + dvelx;
    velhy = velyd[i] + dvely;
    velhz = velzd[i] + dvelz;

    posxd[i] += velhx * dtimed;
    posyd[i] += velhy * dtimed;
    poszd[i] += velhz * dtimed;

    velxd[i] = velhx + dvelx;
    velyd[i] = velhy + dvely;
    velzd[i] = velhz + dvelz;
  }
}


/******************************************************************************/
static void CudaTest(char *msg) {
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(-1);
  }
}

static void debugDump() {
   // host copy for fast debug purpose
   int bottomh, maxdepthh, nnodesh, nbodiesh;
   int *childh; 
   float *posxh, *posyh, *poszh;

   hipMemcpyFromSymbol(&bottomh, HIP_SYMBOL(bottomd), sizeof(int));
   hipMemcpyFromSymbol(&nnodesh, HIP_SYMBOL(nnodesd), sizeof(int));
   hipMemcpyFromSymbol(&nbodiesh, HIP_SYMBOL(nbodiesd), sizeof(int));
   hipMemcpyFromSymbol(&maxdepthh, HIP_SYMBOL(maxdepthd), sizeof(int));
   fprintf(stdout, "buttomd = %d\n", bottomh); 
   fprintf(stdout, "nnodesd = %d\n", nnodesh); 
   fprintf(stdout, "nbodiesd = %d\n", nbodiesh); 
   fprintf(stdout, "maxdepthd = %d\n", maxdepthh); 
   childh = (int*) malloc(sizeof(int) * (nnodesh+1) * 8); 
   posxh = (float*) malloc(sizeof(float) * (nnodesh+1)); 
   posyh = (float*) malloc(sizeof(float) * (nnodesh+1)); 
   poszh = (float*) malloc(sizeof(float) * (nnodesh+1)); 
   int *childl;
   int *posxl, *posyl, *poszl; 
   hipMemcpyFromSymbol(&childl, HIP_SYMBOL(childd), sizeof(int*));
   CudaTest("DebugDump obtain childd\n"); 
   hipMemcpyFromSymbol(&posxl, HIP_SYMBOL(posxd), sizeof(int*));
   hipMemcpyFromSymbol(&posyl, HIP_SYMBOL(posyd), sizeof(int*));
   hipMemcpyFromSymbol(&poszl, HIP_SYMBOL(poszd), sizeof(int*));
   hipMemcpy(childh, childl, sizeof(int) * (nnodesh+1) * 8, hipMemcpyDeviceToHost);
   hipMemcpy(posxh, posxl, sizeof(float) * (nnodesh+1), hipMemcpyDeviceToHost);
   hipMemcpy(posyh, posyl, sizeof(float) * (nnodesh+1), hipMemcpyDeviceToHost);
   hipMemcpy(poszh, poszl, sizeof(float) * (nnodesh+1), hipMemcpyDeviceToHost);
   CudaTest("DebugDump copyback\n"); 
   
   // depth first traversal of tree
   fprintf(stdout, "octree in DFS traversal:\n"); 
   int *stack = (int*) malloc(sizeof(int) * (nnodesh+1) * 8); 
   int *treedepth = (int*) malloc(sizeof(int) * (nnodesh+1) * 8); 
   stack[0] = nnodesh; 
   treedepth[0] = 0; 
   int stack_top = 0; 
   while (stack_top >= 0) {
      int node = stack[stack_top]; 
      int depth = treedepth[stack_top]; 
      stack_top--; 

      int nchild = 0; 
      for (int c = 0; c < 8; c++) {
         int childnode = (childh[8 * node + c]); 
         if (childnode >= 0 && childnode < nbodiesh) {
            // print leaf node 
            fprintf(stdout, "   %d %2d %d \n", childnode, depth + 1, c); 
            // fprintf(stdout, "   %d %2d %d:%d %p\n", childnode, depth + 1, node, c, &(childl[8 * node + c])); 
         } else if (childnode != -1) {
            nchild += 1; 
            stack_top++; 
            stack[stack_top] = childnode; 
            treedepth[stack_top] = depth + 1; 
            // fprintf(stdout, " b %d %2d %d:%d %p\n", childnode, depth + 1, node, c, &(childl[8 * node + c])); 
            // if (depth <= 1) 
            //    fprintf(stdout, "treenode: %d %d [%f %f %f]\n", 
            //            childnode, depth + 1, posxh[childnode] ,posyh[childnode], poszh[childnode]); 
         }
      }
   }
   fprintf(stdout, "\n");
   exit(0);
}

/******************************************************************************/
int main(int argc, char *argv[]) {
  register int i, run, blocks;
  register int nnodes, nbodies, step, timesteps;
  //register int runtime, mintime;
  int error;
  register float dtime, dthf, epssq, itolsq;
  //float time, timing[7];
  //clock_t starttime, endtime;
  //hipEvent_t start, stop;
  float *mass, *posx, *posy, *posz, *velx, *vely, *velz;
  int *errl, *sortl, *childl, *countl, *startl;
  float *massl;
  float *posxl, *posyl, *poszl;
  float *velxl, *velyl, *velzl;
  float *accxl, *accyl, *acczl;
  float *maxxl, *maxyl, *maxzl;
  float *minxl, *minyl, *minzl;
  register double rsc, vsc, r, v, x, y, z, sq, scale;

  // perform some checks
  printf("[BENCH] CUDA BarnesHut\n");
  if (argc != 3) {
    fprintf(stderr, "\n");
    fprintf(stderr, "arguments: number_of_bodies number_of_timesteps\n");
    exit(-1);
  }

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    fprintf(stderr, "There is no device supporting CUDA\n");
    exit(-1);
  }
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {
    fprintf(stderr, "There is no CUDA capable device\n");
    exit(-1);
  }
  if ((deviceProp.major < 1) || ((deviceProp.major == 1) && (deviceProp.minor < 2))) {
    fprintf(stderr, "Need at least compute capability 1.2\n");
    exit(-1);
  }
  if (deviceProp.warpSize != WARPSIZE) {
    fprintf(stderr, "Warp size must be %d\n", deviceProp.warpSize);
    exit(-1);
  }
  if (deviceProp.warpSize != WARPSIZE) {
    fprintf(stderr, "Warp size must be %d\n", deviceProp.warpSize);
    exit(-1);
  }

  blocks = deviceProp.multiProcessorCount;
  printf("numBlocks = %d\n", blocks);

  if ((WARPSIZE <= 0) || (WARPSIZE & (WARPSIZE-1) != 0)) {
    fprintf(stderr, "Warp size must be greater than zero and a power of two\n");
    exit(-1);
  }
  if (WARPSIZE < MAXDEPTH) {
    fprintf(stderr, "Warp size must be greater than or equal to MAXDEPTH\n");
    exit(-1);
  }
  if ((THREADS0 <= 0) || ((THREADS0 & (WARPSIZE-1)) != 0)) {
    fprintf(stderr, "THREADS0 must be greater than zero and an integer multiple of the warp size\n");
    exit(-1);
  }
  if ((THREADS1 <= 0) || ((THREADS1 & (WARPSIZE-1)) != 0) || ((THREADS1 & (THREADS1-1)) != 0)) {
    fprintf(stderr, "THREADS1 must be greater than zero, an integer multiple of the warp size, and a power of two\n");
    exit(-1);
  }
  if ((THREADS2 <= 0) || ((THREADS2 & (WARPSIZE-1)) != 0)) {
    fprintf(stderr, "THREADS2 must be greater than zero and an integer multiple of the warp size\n");
    exit(-1);
  }
  if ((THREADS3 <= 0) || ((THREADS3 & (WARPSIZE-1)) != 0)) {
    fprintf(stderr, "THREADS3 must be greater than zero and an integer multiple of the warp size\n");
    exit(-1);
  }
  if ((THREADS4 <= 0) || ((THREADS4 & (WARPSIZE-1)) != 0)) {
    fprintf(stderr, "THREADS4 must be greater than zero and an integer multiple of the warp size\n");
    exit(-1);
  }
  if ((THREADS5 <= 0) || ((THREADS5 & (WARPSIZE-1)) != 0)) {  /* must be a multiple of the warp size */
    fprintf(stderr, "THREADS5 must be greater than zero and an integer multiple of the warp size\n");
    exit(-1);
  }
  if ((THREADS6 <= 0) || ((THREADS6 & (WARPSIZE-1)) != 0)) {
    fprintf(stderr, "THREADS6 must be greater than zero and an integer multiple of the warp size\n");
    exit(-1);
  }

  hipGetLastError();  // reset error value
  for (run = 0; run < 1; run++) {
    //for (i = 0; i < 7; i++) timing[i] = 0.0f;

    nbodies = atoi(argv[1]);
    if (nbodies < 1) {
      fprintf(stderr, "nbodies is too small: %d\n", nbodies);
      exit(-1);
    }
    if (nbodies > (1 << 30)) {
      fprintf(stderr, "nbodies is too large: %d\n", nbodies);
      exit(-1);
    }
    nnodes = nbodies * 2;
    if (nnodes < 1024*blocks) nnodes = 1024*blocks;
    while ((nnodes & (WARPSIZE-1)) != 0) nnodes++;
    nnodes--;

    timesteps = atoi(argv[2]);
    dtime = 0.025;  dthf = dtime * 0.5f;
    epssq = 0.05 * 0.05;
    itolsq = 1.0f / (0.5 * 0.5);

    // allocate memory
    if (run == 0) {
      fprintf(stderr, "nodes = %d\n", nnodes+1);
      fprintf(stderr, "configuration: %d bodies, %d time steps\n", nbodies, timesteps);

      mass = (float *)malloc(sizeof(float) * nbodies);
      if (mass == NULL) {fprintf(stderr, "cannot allocate mass\n");  exit(-1);}
      posx = (float *)malloc(sizeof(float) * nbodies);
      if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
      posy = (float *)malloc(sizeof(float) * nbodies);
      if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
      posz = (float *)malloc(sizeof(float) * nbodies);
      if (posz == NULL) {fprintf(stderr, "cannot allocate posz\n");  exit(-1);}
      velx = (float *)malloc(sizeof(float) * nbodies);
      if (velx == NULL) {fprintf(stderr, "cannot allocate velx\n");  exit(-1);}
      vely = (float *)malloc(sizeof(float) * nbodies);
      if (vely == NULL) {fprintf(stderr, "cannot allocate vely\n");  exit(-1);}
      velz = (float *)malloc(sizeof(float) * nbodies);
      if (velz == NULL) {fprintf(stderr, "cannot allocate velz\n");  exit(-1);}

      if (hipSuccess != hipMalloc((void **)&errl, sizeof(int))) fprintf(stderr, "could not allocate errd\n");  CudaTest("couldn't allocate errd");
      if (hipSuccess != hipMalloc((void **)&childl, sizeof(int) * (nnodes+1) * 8)) fprintf(stderr, "could not allocate childd\n");  CudaTest("couldn't allocate childd");
      if (hipSuccess != hipMalloc((void **)&massl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate massd\n");  CudaTest("couldn't allocate massd");
      if (hipSuccess != hipMalloc((void **)&posxl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate posxd\n");  CudaTest("couldn't allocate posxd");
      if (hipSuccess != hipMalloc((void **)&posyl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate posyd\n");  CudaTest("couldn't allocate posyd");
      if (hipSuccess != hipMalloc((void **)&poszl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate poszd\n");  CudaTest("couldn't allocate poszd");
      if (hipSuccess != hipMalloc((void **)&countl, sizeof(int) * (nnodes+1))) fprintf(stderr, "could not allocate countd\n");  CudaTest("couldn't allocate countd");
      if (hipSuccess != hipMalloc((void **)&startl, sizeof(int) * (nnodes+1))) fprintf(stderr, "could not allocate startd\n");  CudaTest("couldn't allocate startd");

      // alias arrays
      int inc = (nbodies + WARPSIZE - 1) & (-WARPSIZE);
      velxl = (float *)&childl[0*inc];
      velyl = (float *)&childl[1*inc];
      velzl = (float *)&childl[2*inc];
      accxl = (float *)&childl[3*inc];
      accyl = (float *)&childl[4*inc];
      acczl = (float *)&childl[5*inc];
      sortl = (int *)&childl[6*inc];

      if (hipSuccess != hipMalloc((void **)&maxxl, sizeof(float) * blocks)) fprintf(stderr, "could not allocate maxxd\n");  CudaTest("couldn't allocate maxxd");
      if (hipSuccess != hipMalloc((void **)&maxyl, sizeof(float) * blocks)) fprintf(stderr, "could not allocate maxyd\n");  CudaTest("couldn't allocate maxyd");
      if (hipSuccess != hipMalloc((void **)&maxzl, sizeof(float) * blocks)) fprintf(stderr, "could not allocate maxzd\n");  CudaTest("couldn't allocate maxzd");
      if (hipSuccess != hipMalloc((void **)&minxl, sizeof(float) * blocks)) fprintf(stderr, "could not allocate minxd\n");  CudaTest("couldn't allocate minxd");
      if (hipSuccess != hipMalloc((void **)&minyl, sizeof(float) * blocks)) fprintf(stderr, "could not allocate minyd\n");  CudaTest("couldn't allocate minyd");
      if (hipSuccess != hipMalloc((void **)&minzl, sizeof(float) * blocks)) fprintf(stderr, "could not allocate minzd\n");  CudaTest("couldn't allocate minzd");

      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(nnodesd), &nnodes, sizeof(int))) fprintf(stderr, "copying of nnodes to device failed\n");  CudaTest("nnode copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(nbodiesd), &nbodies, sizeof(int))) fprintf(stderr, "copying of nbodies to device failed\n");  CudaTest("nbody copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(errd), &errl, sizeof(int))) fprintf(stderr, "copying of err to device failed\n");  CudaTest("err copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dtimed), &dtime, sizeof(float))) fprintf(stderr, "copying of dtime to device failed\n");  CudaTest("dtime copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dthfd), &dthf, sizeof(float))) fprintf(stderr, "copying of dthf to device failed\n");  CudaTest("dthf copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(epssqd), &epssq, sizeof(float))) fprintf(stderr, "copying of epssq to device failed\n");  CudaTest("epssq copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(itolsqd), &itolsq, sizeof(float))) fprintf(stderr, "copying of itolsq to device failed\n");  CudaTest("itolsq copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(sortd), &sortl, sizeof(int))) fprintf(stderr, "copying of sortl to device failed\n");  CudaTest("sortl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(countd), &countl, sizeof(int))) fprintf(stderr, "copying of countl to device failed\n");  CudaTest("countl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(startd), &startl, sizeof(int))) fprintf(stderr, "copying of startl to device failed\n");  CudaTest("startl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(childd), &childl, sizeof(int))) fprintf(stderr, "copying of childl to device failed\n");  CudaTest("childl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(massd), &massl, sizeof(int))) fprintf(stderr, "copying of massl to device failed\n");  CudaTest("massl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(posxd), &posxl, sizeof(int))) fprintf(stderr, "copying of posxl to device failed\n");  CudaTest("posxl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(posyd), &posyl, sizeof(int))) fprintf(stderr, "copying of posyl to device failed\n");  CudaTest("posyl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(poszd), &poszl, sizeof(int))) fprintf(stderr, "copying of poszl to device failed\n");  CudaTest("poszl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(velxd), &velxl, sizeof(int))) fprintf(stderr, "copying of velxl to device failed\n");  CudaTest("velxl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(velyd), &velyl, sizeof(int))) fprintf(stderr, "copying of velyl to device failed\n");  CudaTest("velyl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(velzd), &velzl, sizeof(int))) fprintf(stderr, "copying of velzl to device failed\n");  CudaTest("velzl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(accxd), &accxl, sizeof(int))) fprintf(stderr, "copying of accxl to device failed\n");  CudaTest("accxl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(accyd), &accyl, sizeof(int))) fprintf(stderr, "copying of accyl to device failed\n");  CudaTest("accyl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(acczd), &acczl, sizeof(int))) fprintf(stderr, "copying of acczl to device failed\n");  CudaTest("acczl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(maxxd), &maxxl, sizeof(int))) fprintf(stderr, "copying of maxxl to device failed\n");  CudaTest("maxxl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(maxyd), &maxyl, sizeof(int))) fprintf(stderr, "copying of maxyl to device failed\n");  CudaTest("maxyl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(maxzd), &maxzl, sizeof(int))) fprintf(stderr, "copying of maxzl to device failed\n");  CudaTest("maxzl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(minxd), &minxl, sizeof(int))) fprintf(stderr, "copying of minxl to device failed\n");  CudaTest("minxl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(minyd), &minyl, sizeof(int))) fprintf(stderr, "copying of minyl to device failed\n");  CudaTest("minyl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(minzd), &minzl, sizeof(int))) fprintf(stderr, "copying of minzl to device failed\n");  CudaTest("minzl copy to device failed");
    }

    // generate input
    drndset(7);
    rsc = (3 * M_PI) / 16;
    vsc = sqrt(1.0 / rsc);
    for (i = 0; i < nbodies; i++) {
      mass[i] = 1.0 / nbodies;
      r = 1.0 / sqrt(pow(drnd()*0.999, -2.0/3.0) - 1);
      do {
        x = drnd()*2.0 - 1.0;
        y = drnd()*2.0 - 1.0;
        z = drnd()*2.0 - 1.0;
        sq = x*x + y*y + z*z;
      } while (sq > 1.0);
      scale = rsc * r / sqrt(sq);
      posx[i] = x * scale;
      posy[i] = y * scale;
      posz[i] = z * scale;

      do {
        x = drnd();
        y = drnd() * 0.1;
      } while (y > x*x * pow(1 - x*x, 3.5));
      v = x * sqrt(2.0 / sqrt(1 + r*r));
      do {
        x = drnd()*2.0 - 1.0;
        y = drnd()*2.0 - 1.0;
        z = drnd()*2.0 - 1.0;
        sq = x*x + y*y + z*z;
      } while (sq > 1.0);
      scale = vsc * v / sqrt(sq);
      velx[i] = x * scale;
      vely[i] = y * scale;
      velz[i] = z * scale;
    }

    if (hipSuccess != hipMemcpy(massl, mass, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of mass to device failed\n");  CudaTest("mass copy to device failed");
    if (hipSuccess != hipMemcpy(posxl, posx, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of posx to device failed\n");  CudaTest("posx copy to device failed");
    if (hipSuccess != hipMemcpy(posyl, posy, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of posy to device failed\n");  CudaTest("posy copy to device failed");
    if (hipSuccess != hipMemcpy(poszl, posz, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of posz to device failed\n");  CudaTest("posz copy to device failed");
    if (hipSuccess != hipMemcpy(velxl, velx, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of velx to device failed\n");  CudaTest("velx copy to device failed");
    if (hipSuccess != hipMemcpy(velyl, vely, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of vely to device failed\n");  CudaTest("vely copy to device failed");
    if (hipSuccess != hipMemcpy(velzl, velz, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of velz to device failed\n");  CudaTest("velz copy to device failed");

    CudaTest("[ERROR] Before kernel launch");
    printf("[BENCH] Lauch GPU kernels\n");

    //hipEventCreate(&start);  hipEventCreate(&stop);  
    //starttime = clock();
    //hipEventRecord(start, 0);
    InitializationKernel<<<blocks*FACTOR0, THREADS0>>>();
    CudaTest("[ERROR] Kernel <Initialization> launch failed");
    //hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
    //timing[0] += time;
    for (step = 0; step < timesteps; step++) {
      //hipEventRecord(start, 0);
      BoundingBoxKernel<<<blocks*FACTOR1, THREADS1>>>();
      CudaTest("[ERROR] Kernel <BoundingBox> launch failed");
      //hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      //timing[1] += time;
      //CudaTest("kernel 1 launch failed");
      //hipEventRecord(start, 0);
      TreeBuildingKernel<<<blocks*FACTOR2, THREADS2>>>();
      CudaTest("[ERROR] Kernel <TreeBuilding> launch failed");
      //hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      //timing[2] += time;
      //debugDump(); 
      //CudaTest("kernel 2 launch failed");
      //hipEventRecord(start, 0);
      SummarizationKernel<<<blocks*FACTOR3, THREADS3>>>();
      CudaTest("[ERROR] Kernel <Summarization> launch failed");
      //hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      //timing[3] += time;
      //CudaTest("kernel 3 launch failed");
      //hipEventRecord(start, 0);
      SortKernel<<<blocks*FACTOR4, 512>>>();
      CudaTest("[ERROR] Kernel <Sort> launch failed");
      //hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      //timing[4] += time;
      //CudaTest("kernel 4 launch failed");
      //hipEventRecord(start, 0);
      ForceCalculationKernel<<<blocks*FACTOR5, THREADS5>>>();
      CudaTest("[ERROR] Kernel <ForceCalculation> launch failed");
      //hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      //timing[5] += time;
      //CudaTest("kernel 5 launch failed");
      //hipEventRecord(start, 0);
      IntegrationKernel<<<blocks*FACTOR6, THREADS6>>>();
      CudaTest("[ERROR] Kernel <Intergration> launch failed");
      //hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      //timing[6] += time;
      //CudaTest("kernel 6 launch failed");
    }
    //endtime = clock();
    CudaTest("[ERROR] Kernel launch failed");
    //hipEventDestroy(start);  hipEventDestroy(stop);

    // transfer result back to CPU
    if (hipSuccess != hipMemcpy(&error, errl, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "copying of err from device failed\n");  CudaTest("err copy from device failed");
    if (hipSuccess != hipMemcpy(posx, posxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of posx from device failed\n");  CudaTest("posx copy from device failed");
    if (hipSuccess != hipMemcpy(posy, posyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of posy from device failed\n");  CudaTest("posy copy from device failed");
    if (hipSuccess != hipMemcpy(posz, poszl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of posz from device failed\n");  CudaTest("posz copy from device failed");
    if (hipSuccess != hipMemcpy(velx, velxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of velx from device failed\n");  CudaTest("velx copy from device failed");
    if (hipSuccess != hipMemcpy(vely, velyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of vely from device failed\n");  CudaTest("vely copy from device failed");
    if (hipSuccess != hipMemcpy(velz, velzl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of velz from device failed\n");  CudaTest("velz copy from device failed");

/*
    runtime = (int) (1000.0f * (endtime - starttime) / CLOCKS_PER_SEC);
    fprintf(stderr, "runtime: %d ms  (", runtime);
    time = 0;
    for (i = 1; i < 7; i++) {
      fprintf(stderr, " %.1f ", timing[i]);
      time += timing[i];
    }
    if (error == 0) {
      fprintf(stderr, ") = %.1f\n", time);
    } else {
      fprintf(stderr, ") = %.1f FAILED %d\n", time, error);
    }

    if ((run == 0) || (mintime > runtime)) mintime = runtime;
*/
  }

//  fprintf(stderr, "mintime: %d ms\n", mintime);

  // print output
  for (i = 0; i < nbodies; i++) {
    printf("%.2e %.2e %.2e\n", posx[i], posy[i], posz[i]);
  }
  return 0;
}
